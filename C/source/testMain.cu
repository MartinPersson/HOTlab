/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "hologram.h"

int main(int argc, char *argv[])
{
	HLG_initailize();
	HLG_process();
	HLG_cleanup();
	return 0;
}