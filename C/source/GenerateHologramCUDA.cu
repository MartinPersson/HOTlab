#include "hip/hip_runtime.h"
/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

// Activates a number of custom debug macros
#define M_CUDA_DEBUG

// Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

// Number of spots/traps/depth planes - even 64 is pushing it
#define MAX_SPOTS 64

#define BLOCK_SIZE 256

#define SLM_SIZE 512

// Use bitwise modulo operations if the SLM size is a power of 2
#if (((SLM_SIZE - 1) & (SLM_SIZE)) == 0)
#define SLMPOW2
#endif

// Forward declarations
__global__ void ApplyCorrections(unsigned char *g_pSLM_uc, unsigned char *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f);
__global__ void LensesAndPrisms(unsigned char *g_SLMuc, unsigned char *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f);
__global__ void calculateIobtained(unsigned char *g_pSLM_uc, float *g_Iobtained);
__global__ void PropagateToSLM_Fresnel(float *g_spotRe_f, float *g_spotIm_f, float *g_pSLM2pi, float *g_weights, int iteration, float *g_pSLMstart, float *g_amps, bool getpSLM255, unsigned char *g_pSLM255_uc, unsigned char *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f);
__global__ void PropagateToSpotPositions_Fresnel(float *g_pSLM2pi, float *g_spotRe_f, float *g_spotIm_f);
__global__ void XYtoIndex();
__global__ void f2uc(unsigned char *uc, float *f, int N_pixels, unsigned char *g_LUT, int use_linLUT, int data_w);
__global__ void uc2f(float *f, unsigned char *uc, int N);
inline int computeAndCopySpotData(float *h_I, float *x, float *y, float *z, int N_spots, int method);

// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, char *file);
inline void mCheckError(int line, char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataUC(unsigned char *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

// Global declaration
float *d_x, *d_y, *d_z, *d_I;         //trap coordinates and intensity in GPU memory
float *d_pSLM_f;                //the optimized pSpot pattern, float [-pi, pi]
float *d_weights, *d_Iobtained, *d_desiredAmp;    //used h_weights and calculated amplitudes for each spot and each iteration
float *d_pSLMstart_f;             //Initial pSpot pattern [-pi, pi]
float *d_spotRe_f, *d_spotIm_f;
float *d_AberrationCorr_f = NULL;
float *d_LUTPolCoeff_f = NULL;
float SLMsizef = (float) SLM_SIZE;
int N_PolLUTCoeff = 0;
int n_blocks_Phi, memsize_SLM_f, memsize_SLMuc, memsize_spotsf, data_w, N_pixels;
float h_desiredAmp[MAX_SPOTS];
unsigned char *d_pSLM_uc;           //The optimized pSpot pattern, unsigned char, the one sent to the SLM [0, 255]
unsigned char *h_LUT_uc;
unsigned char *d_LUT_uc = NULL;
bool ApplyLUT_b = false, UseAberrationCorr_b = false, UsePolLUT_b = false, saveI_b = false, useRPC_b = false;
float alphaRPC_f = 10;
char CUDAmessage[100];
hipError_t status;
float *d_obtainedPhase;

// Constant memory declarations
__device__ __constant__ int c_data_w[1];
__device__ __constant__ float c_data_w_f[1];
__device__ __constant__ int c_half_w[1];
__device__ __constant__ float c_half_w_f[1];
__device__ __constant__ int c_N_pixels[1];
__device__ __constant__ float c_N_pixels_f[1];
__device__ __constant__ float c_SLMpitch_f[1];
__device__ __constant__ bool c_applyLUT_b[1];
__device__ __constant__ bool c_useAberrationCorr_b[1];
__device__ __constant__ bool c_usePolLUT_b[1];
__device__ __constant__ int c_N_PolLUTCoeff[1];
__device__ __constant__ bool c_useRPC_b[1];
__device__ __constant__ float c_alphaRPC_f[1];
__device__ __constant__ bool c_saveI_b[1];
__device__ __constant__ int c_log2data_w[1];
__device__ __constant__ float c_x[MAX_SPOTS];
__device__ __constant__ float c_y[MAX_SPOTS];
__device__ __constant__ float c_z[MAX_SPOTS];
__device__ __constant__ float c_desiredAmp[MAX_SPOTS];
__device__ __constant__ int c_N_spots[1];

// Timing
double get_clock() {
  struct timeval tv;
  int ok;
  ok = gettimeofday(&tv, NULL);
  if (ok < 0) {
    printf("gettimeofday error");
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// Generates a hologram
int generate_hologram(unsigned char *hologram,
                      float *x_spots,
                      float *y_spots,
                      float *z_spots,
                      float *i_spots,
                      int num_spots,
                      const int num_iterations,
                      float *intensity,
                      int method)
{
  if (num_spots > MAX_SPOTS)
    num_spots = MAX_SPOTS;
  else if (num_spots < 1)
    method = 100;
  else if (num_spots < 3)
    method = 0;

  memsize_spotsf = num_spots * sizeof(float);

  // Sets method to -1 if num_spots == 0.
  method = computeAndCopySpotData(i_spots, x_spots, y_spots, z_spots, num_spots, method);

  double t;

  switch (method) {
    case 0:
      // Generate hologram using "Lenses and Prisms"
      printf("Starting Lenses and Prisms...\n");
      t = get_clock();

      LensesAndPrisms<<<n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_CHECK_ERROR();

      if (saveI_b) {
        calculateIobtained<<<num_spots, SLM_SIZE>>>(d_pSLM_uc, d_Iobtained);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
        M_SAFE_CALL(hipMemcpy(intensity, d_Iobtained, num_spots*sizeof(float), hipMemcpyDeviceToHost));
      }
      M_SAFE_CALL(hipMemcpy(hologram, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = get_clock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      break;
    case 1:
      // Generate holgram using fresnel propagation
      printf("Starting Fresnel...\n");
      t = get_clock();

      // Uncomment this to start with pre-calculated hologram:
      //hipMemcpy(d_pSLM_uc, hologram, memsize_SLMuc, hipMemcpyHostToDevice);
      //hipDeviceSynchronize();
      //uc2f<<<n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_f, d_pSLM_uc, N_pixels);

      for (int l = 0; l < num_iterations; l++) {
        printf("Iteration %d\n", l);

        // Propagate to the spot positions
        PropagateToSpotPositions_Fresnel<<<num_spots, SLM_SIZE>>>(d_pSLM_f, d_spotRe_f, d_spotIm_f);
        M_CHECK_ERROR();
        hipDeviceSynchronize();

        // Propagate to the SLM plane
        PropagateToSLM_Fresnel<<<n_blocks_Phi, BLOCK_SIZE >>>(d_spotRe_f, d_spotIm_f, d_pSLM_f, d_weights, l, d_pSLMstart_f, d_Iobtained, (l==(num_iterations-1)), d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
      }

      if (saveI_b)
        M_SAFE_CALL(hipMemcpy(intensity, d_Iobtained, num_spots*(num_iterations)*sizeof(float), hipMemcpyDeviceToHost));
      else
        M_SAFE_CALL(hipMemcpy(intensity, d_weights, num_spots*(num_iterations)*sizeof(float), hipMemcpyDeviceToHost));
      M_SAFE_CALL(hipMemcpy(hologram, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = get_clock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      printf("Time/iteration = %12.8lf seconds\n", t/((double) num_iterations));
      break;
    case 100:
      // Apply corrections to pre-calculated hologram
      M_SAFE_CALL(hipMemcpy(d_pSLM_uc, hologram, memsize_SLMuc, hipMemcpyHostToDevice));
      ApplyCorrections<<<n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
      M_SAFE_CALL(hipMemcpy(hologram, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));
      break;
    default:
      break;
  }

  // Handle CUDA errors
  status = hipGetLastError();
  return status;
}

// Set correction parameters
int corrections(int UseAberrationCorr, float *h_AberrationCorr, int UseLUTPol, int PolOrder, float *h_LUTPolCoeff, int saveAmplitudes, float alpha, int UseLUT, unsigned char *h_LUT_uc)
{
  UseAberrationCorr_b = (bool) UseAberrationCorr;
  hipMemcpyToSymbol(HIP_SYMBOL(c_useAberrationCorr_b), &UseAberrationCorr_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  UsePolLUT_b = (bool) UseLUTPol;
  hipMemcpyToSymbol(HIP_SYMBOL(c_usePolLUT_b), &UsePolLUT_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  saveI_b = (bool) saveAmplitudes;
  hipMemcpyToSymbol(HIP_SYMBOL(c_saveI_b), &saveI_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  ApplyLUT_b = (bool) UseLUT;
  hipMemcpyToSymbol(HIP_SYMBOL(c_applyLUT_b), &ApplyLUT_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  alphaRPC_f = alpha*2.0f*M_PI;
  if (alpha < 1.0f)
    useRPC_b = true;
  else
    useRPC_b = false;
  hipMemcpyToSymbol(HIP_SYMBOL(c_alphaRPC_f), &alphaRPC_f, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_useRPC_b), &useRPC_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  int Ncoeff[5] = {20, 35, 56, 84, 120};

  if ((3 <= PolOrder) && (PolOrder <= 7)) {
    N_PolLUTCoeff = Ncoeff[PolOrder - 3];
    printf("%d\n", N_PolLUTCoeff);
  } else {
    printf("Polynomial order out of range\n -coerced to 3\n");
    N_PolLUTCoeff = Ncoeff[0];
  }
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_PolLUTCoeff), &N_PolLUTCoeff, sizeof(int), 0, hipMemcpyHostToDevice);

  if(UseAberrationCorr_b)
  {
    if (d_AberrationCorr_f == NULL)   //Allocate memory only if not already allocated
      hipMalloc((void**)&d_AberrationCorr_f, memsize_SLM_f);
    UseAberrationCorr_b = !hipMemcpy(d_AberrationCorr_f, h_AberrationCorr, memsize_SLM_f, hipMemcpyHostToDevice);
  }
  else if (d_AberrationCorr_f != NULL)  //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_AberrationCorr_f);
    d_AberrationCorr_f = NULL;
  }
  if(UsePolLUT_b)
  {
    if (d_LUTPolCoeff_f == NULL)          //Allocate memory only if not already allocated
      hipMalloc((void**)&d_LUTPolCoeff_f, 120*sizeof(float));
    UsePolLUT_b = !hipMemcpy(d_LUTPolCoeff_f, h_LUTPolCoeff, N_PolLUTCoeff*sizeof(float), hipMemcpyHostToDevice);
  }
  else if (d_LUTPolCoeff_f!=NULL) //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_LUTPolCoeff_f);
    d_LUTPolCoeff_f = NULL;
  }

  if(ApplyLUT_b&&(!UsePolLUT_b))
  {
    if (d_LUT_uc == NULL)         //Allocate memory only if not already allocated
      hipMalloc((void**)&d_LUT_uc, 256*sizeof(unsigned char));
    UseLUT = !hipMemcpy(d_LUT_uc, h_LUT_uc, 256*sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  else if (d_LUT_uc!=NULL)  //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_LUT_uc);
    d_LUT_uc = NULL;
  }
  M_CHECK_ERROR();

  status = hipGetLastError();
  return status;
}

// Allocate GPU memory and start up SLM
int setup(float *h_init_phases)
{
  UseAberrationCorr_b = false;
  UsePolLUT_b = false;
  saveI_b = true;
  ApplyLUT_b = false;

  // Make sure there's a GPU
  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != 0) {
    printf("No CUDA compatible GPU found\n");
    exit(1);
  } else {
    M_SAFE_CALL(hipSetDevice(0));
  }

  int MaxIterations = 1000;
  data_w = SLM_SIZE;
  hipMemcpyToSymbol(HIP_SYMBOL(c_data_w), &data_w, sizeof(int), 0, hipMemcpyHostToDevice);
  float data_w_f = (float)data_w;
  hipMemcpyToSymbol(HIP_SYMBOL(c_data_w_f), &data_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
  int half_w = (int)(data_w/2);
  hipMemcpyToSymbol(HIP_SYMBOL(c_half_w), &half_w, sizeof(int), 0, hipMemcpyHostToDevice);
  float half_w_f = (float)data_w/2.0f;
  hipMemcpyToSymbol(HIP_SYMBOL(c_half_w_f), &half_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
  N_pixels = data_w * data_w;
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels), &N_pixels, sizeof(int), 0, hipMemcpyHostToDevice);
  float N_pixels_f = (float)N_pixels;
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels_f), &N_pixels_f, sizeof(float), 0, hipMemcpyHostToDevice);
  int logN = (int)(log2(data_w_f));
  hipMemcpyToSymbol(HIP_SYMBOL(c_log2data_w), &logN, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_useRPC_b), &useRPC_b, sizeof(bool), 0, hipMemcpyHostToDevice);
  float SLMpitch_f = 1.0f/data_w_f;
  hipMemcpyToSymbol(HIP_SYMBOL(c_SLMpitch_f), &SLMpitch_f, sizeof(float), 0, hipMemcpyHostToDevice);

  memsize_spotsf = MAX_SPOTS * sizeof(float);
  memsize_SLM_f = N_pixels * sizeof(float);
  memsize_SLMuc = N_pixels * sizeof(unsigned char);
  n_blocks_Phi = (N_pixels/BLOCK_SIZE + (N_pixels%BLOCK_SIZE == 0 ? 0:1));

  // Memory allocations for all methods
  M_SAFE_CALL(hipMalloc((void**)&d_x, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_y, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_z, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_I, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_desiredAmp, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_weights, MAX_SPOTS*(MaxIterations+1)*sizeof(float)));
  M_SAFE_CALL(hipMalloc((void**)&d_Iobtained, MAX_SPOTS*MaxIterations*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_obtainedPhase, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotRe_f, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotIm_f, memsize_spotsf ));

  int data_w_pow2 = pow(2, ceil(log((float)data_w)/log(2.0f)));
  M_SAFE_CALL(hipMalloc((void**)&d_pSLM_f, data_w_pow2*data_w_pow2*sizeof(float)));//the size of d_pSLM_f must be a power of 2 for the summation algorithm to work
  M_SAFE_CALL(hipMemset(d_pSLM_f, 0, data_w_pow2*data_w_pow2*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_pSLMstart_f, memsize_SLM_f));
  M_SAFE_CALL(hipMalloc((void**)&d_pSLM_uc, memsize_SLMuc));
  M_SAFE_CALL(hipMemset(d_pSLMstart_f, 0, N_pixels*sizeof(float)));

  M_SAFE_CALL(hipMemcpy(d_pSLM_f, h_init_phases, N_pixels*sizeof(float), hipMemcpyHostToDevice));

  status = hipGetLastError();
  return status;
}

// Free GPU memory and shut down SLM
int finish()
{
  M_SAFE_CALL(hipFree(d_x));
  M_SAFE_CALL(hipFree(d_y));
  M_SAFE_CALL(hipFree(d_z));
  M_SAFE_CALL(hipFree(d_I));

  M_SAFE_CALL(hipFree(d_weights));
  M_SAFE_CALL(hipFree(d_Iobtained));
  M_SAFE_CALL(hipFree(d_pSLM_f));
  M_SAFE_CALL(hipFree(d_pSLMstart_f));
  M_SAFE_CALL(hipFree(d_pSLM_uc));

  if (ApplyLUT_b) {
    hipFree(d_LUT_uc);
    d_LUT_uc = NULL;
  }

  if (UseAberrationCorr_b) {
    hipFree(d_AberrationCorr_f);
    d_AberrationCorr_f = NULL;
  }

  if (UsePolLUT_b) {
    hipFree(d_LUTPolCoeff_f);
    d_LUTPolCoeff_f = NULL;
  }

  hipDeviceReset();
  status = hipGetLastError();
  return status;
}

// Device functions

__device__ float uc2phase(float uc)
{
  return (float) (uc * 2.0f * M_PI/256.0f - M_PI);
}

__device__ unsigned char phase2uc(float phase2pi)
{
  return (unsigned char) floor((phase2pi + M_PI) * 256.0f / (2.0f * M_PI));
}

__device__ int phase2int32(float phase2pi)
{
  return (int) floor((phase2pi + M_PI) * 256.0f / (2.0f * M_PI));
}

__device__ float ApplyAberrationCorrection(float pSpot, float correction)
{
  pSpot = pSpot - correction;   //apply correction
  return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI))); //apply mod([-pi, pi], pSpot)
}

__device__ int getXint(int index)
{
#ifdef SLMPOW2
  int X_int = index & (c_data_w[0] - 1);
#else
  float X_int= index % c_data_w[0];
#endif
  return X_int;
}

__device__ int getYint(int index, int X_int)
{
#ifdef SLMPOW2
  int Y_int = (index - X_int) >> c_log2data_w[0];
#else
  int Y_int = (float) (floor((float) index/c_data_w_f[0]));
#endif
  return Y_int;
}

__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c)
{
  float phase255 = 0.0f;
  switch (c_N_PolLUTCoeff[0]) {
    case 120:
      phase255 += s_c[84]*X*X*X*X*X*X*X;
      phase255 += s_c[85]*X*X*X*X*X*X*Y;
      phase255 += s_c[86]*X*X*X*X*X*X*phase2pi;
      phase255 += s_c[87]*X*X*X*X*X*Y*Y;
      phase255 += s_c[88]*X*X*X*X*X*Y*phase2pi;
      phase255 += s_c[89]*X*X*X*X*X*phase2pi*phase2pi;
      phase255 += s_c[90]*X*X*X*X*Y*Y*Y;
      phase255 += s_c[91]*X*X*X*X*Y*Y*phase2pi;
      phase255 += s_c[92]*X*X*X*X*Y*phase2pi*phase2pi;
      phase255 += s_c[93]*X*X*X*X*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[94]*X*X*X*Y*Y*Y*Y;
      phase255 += s_c[95]*X*X*X*Y*Y*Y*phase2pi;
      phase255 += s_c[96]*X*X*X*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[97]*X*X*X*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[98]*X*X*X*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[99]*X*X*Y*Y*Y*Y*Y;
      phase255 += s_c[100]*X*X*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[101]*X*X*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[102]*X*X*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[103]*X*X*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[104]*X*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[105]*X*Y*Y*Y*Y*Y*Y;
      phase255 += s_c[106]*X*Y*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[107]*X*Y*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[108]*X*Y*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[109]*X*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[110]*X*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[111]*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[112]*Y*Y*Y*Y*Y*Y*Y;
      phase255 += s_c[113]*Y*Y*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[114]*Y*Y*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[115]*Y*Y*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[116]*Y*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[117]*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[118]*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[119]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
    case 84:
      phase255 += s_c[56]*X*X*X*X*X*X;
      phase255 += s_c[57]*X*X*X*X*X*Y;
      phase255 += s_c[58]*X*X*X*X*X*phase2pi;
      phase255 += s_c[59]*X*X*X*X*Y*Y;
      phase255 += s_c[60]*X*X*X*X*Y*phase2pi;
      phase255 += s_c[61]*X*X*X*X*phase2pi*phase2pi;
      phase255 += s_c[62]*X*X*X*Y*Y*Y;
      phase255 += s_c[63]*X*X*X*Y*Y*phase2pi;
      phase255 += s_c[64]*X*X*X*Y*phase2pi*phase2pi;
      phase255 += s_c[65]*X*X*X*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[66]*X*X*Y*Y*Y*Y;
      phase255 += s_c[67]*X*X*Y*Y*Y*phase2pi;
      phase255 += s_c[68]*X*X*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[69]*X*X*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[70]*X*X*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[71]*X*Y*Y*Y*Y*Y;
      phase255 += s_c[72]*X*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[73]*X*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[74]*X*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[75]*X*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[76]*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[77]*Y*Y*Y*Y*Y*Y;
      phase255 += s_c[78]*Y*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[79]*Y*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[80]*Y*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[81]*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[82]*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[83]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
    case 56:
      phase255 += s_c[35]*X*X*X*X*X;
      phase255 += s_c[36]*X*X*X*X*Y;
      phase255 += s_c[37]*X*X*X*X*phase2pi;
      phase255 += s_c[38]*X*X*X*Y*Y;
      phase255 += s_c[39]*X*X*X*Y*phase2pi;
      phase255 += s_c[40]*X*X*X*phase2pi*phase2pi;
      phase255 += s_c[41]*X*X*Y*Y*Y;
      phase255 += s_c[42]*X*X*Y*Y*phase2pi;
      phase255 += s_c[43]*X*X*Y*phase2pi*phase2pi;
      phase255 += s_c[44]*X*X*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[45]*X*Y*Y*Y*Y;
      phase255 += s_c[46]*X*Y*Y*Y*phase2pi;
      phase255 += s_c[47]*X*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[48]*X*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[49]*X*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[50]*Y*Y*Y*Y*Y;
      phase255 += s_c[51]*Y*Y*Y*Y*phase2pi;
      phase255 += s_c[52]*Y*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[53]*Y*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[54]*Y*phase2pi*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[55]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi;
    case 35:
      phase255 += s_c[20]*X*X*X*X;
      phase255 += s_c[21]*X*X*X*Y;
      phase255 += s_c[22]*X*X*X*phase2pi;
      phase255 += s_c[23]*X*X*Y*Y;
      phase255 += s_c[24]*X*X*Y*phase2pi;
      phase255 += s_c[25]*X*X*phase2pi*phase2pi;
      phase255 += s_c[26]*X*Y*Y*Y;
      phase255 += s_c[27]*X*Y*Y*phase2pi;
      phase255 += s_c[28]*X*Y*phase2pi*phase2pi;
      phase255 += s_c[29]*X*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[30]*Y*Y*Y*Y;
      phase255 += s_c[31]*Y*Y*Y*phase2pi;
      phase255 += s_c[32]*Y*Y*phase2pi*phase2pi;
      phase255 += s_c[33]*Y*phase2pi*phase2pi*phase2pi;
      phase255 += s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi;
    case 20:
      phase255 += s_c[0];
      phase255 += s_c[1]*X;
      phase255 += s_c[2]*Y;
      phase255 += s_c[3]*phase2pi;
      phase255 += s_c[4]*X*X;
      phase255 += s_c[5]*X*Y;
      phase255 += s_c[6]*X*phase2pi;
      phase255 += s_c[7]*Y*Y;
      phase255 += s_c[8]*Y*phase2pi;
      phase255 += s_c[9]*phase2pi*phase2pi;
      phase255 += s_c[10]*X*X*X;
      phase255 += s_c[11]*X*X*Y;
      phase255 += s_c[12]*X*X*phase2pi;
      phase255 += s_c[13]*X*Y*Y;
      phase255 += s_c[14]*X*Y*phase2pi;
      phase255 += s_c[15]*X*phase2pi*phase2pi;
      phase255 += s_c[16]*Y*Y*Y;
      phase255 += s_c[17]*Y*Y*phase2pi;
      phase255 += s_c[18]*Y*phase2pi*phase2pi;
      phase255 += s_c[19]*phase2pi*phase2pi*phase2pi;
      break;
    default:
      phase255 = 0.0f;
      break;
  }
  if (phase255 < 0.0f)
    phase255 = 0.0f;
  return (unsigned char)(phase255);
}

// Performs intra-warp reduction
__device__ void warpReduceC(volatile float *s_Vre, volatile float *s_Vim, int tid)
{
  s_Vre[tid] += s_Vre[tid + 32];
  s_Vim[tid] += s_Vim[tid + 32];

  s_Vre[tid] += s_Vre[tid + 16];
  s_Vim[tid] += s_Vim[tid + 16];

  s_Vre[tid] += s_Vre[tid + 8];
  s_Vim[tid] += s_Vim[tid + 8];

  s_Vre[tid] += s_Vre[tid + 4];
  s_Vim[tid] += s_Vim[tid + 4];

  s_Vre[tid] += s_Vre[tid + 2];
  s_Vim[tid] += s_Vim[tid + 2];

  s_Vre[tid] += s_Vre[tid + 1];
  s_Vim[tid] += s_Vim[tid + 1];
}

inline int computeAndCopySpotData(float *h_I, float *x, float *y, float *z, int N_spots, int method)
{
  //float Isum = 0.0f;
  //for (int i = 0; i<N_spots; i++)
  //  Isum += h_I[i];
  for (int j = 0; j < N_spots; j++) {
    float sincx_rec = (x[j] == 0) ? 1.0f : ((M_PI * x[j]/SLMsizef)/sinf(M_PI * x[j]/SLMsizef));
    float sincy_rec = (y[j] == 0) ? 1.0f : ((M_PI * y[j]/SLMsizef)/sinf(M_PI * y[j]/SLMsizef));
    h_desiredAmp[j] = (h_I[j] <= 0.0f) ? 1.0f : (sincx_rec * sincy_rec * sqrtf(h_I[j]/100) * SLMsizef * SLMsizef);
  }
  hipMemcpyToSymbol(HIP_SYMBOL(c_x), x, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_y), y, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_z), z, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_desiredAmp), h_desiredAmp, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_spots), &N_spots, sizeof(int), 0, hipMemcpyHostToDevice);

  if (N_spots == 0)
    method = -1;

  return method;
}

// Apply corrections to precalculated hologram
__global__ void ApplyCorrections(unsigned char *g_pSLM_uc, unsigned char *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float pSLM2pi_f = uc2phase(g_pSLM_uc[idx]);
  if (c_useAberrationCorr_b[0])
    pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);

  if (c_usePolLUT_b[0])
  {
    int X_int = getXint(idx);
    int Y_int = getYint(idx, X_int);
    float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
    float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);
    __shared__ float s_LUTcoeff[120];
    if (tid < c_N_PolLUTCoeff[0])
      s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
    __syncthreads();
    g_pSLM_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff);
  }
  else if (c_applyLUT_b[0])
  {
    __shared__ unsigned char s_LUT[256];
    if (tid < 256)
      s_LUT[tid] = g_LUT[tid];
    __syncthreads();
    g_pSLM_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
  }
  else
    g_pSLM_uc[idx] = phase2uc(pSLM2pi_f);
}

// Calculate hologram using "Lenses and Prisms"
__global__ void LensesAndPrisms(unsigned char *g_SLMuc, unsigned char *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  if (idx < c_N_pixels[0])
  {
    //get pixel coordinates
    int X_int = getXint(idx);
    int Y_int = getYint(idx, X_int);
    float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
    float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);

    float phase2pi;
    float SLMre = 0.0f;
    float SLMim = 0.0f;

    for (int ii=0; ii<c_N_spots[0]; ++ii)
    {
      //add variable phases to function call
      phase2pi = M_PI * c_z[ii] * (X*X + Y*Y) + 2.0f * M_PI * (X * (c_x[ii]) + Y * (c_y[ii]) );
      SLMre = SLMre + c_desiredAmp[ii] * cosf(phase2pi);
      SLMim = SLMim + c_desiredAmp[ii] * sinf(phase2pi);
    }
    phase2pi = atan2f(SLMim, SLMre);  // [-pi,pi]

    if (c_useAberrationCorr_b[0])
      phase2pi = ApplyAberrationCorrection(phase2pi, d_AberrationCorr_f[idx]);

    if (c_usePolLUT_b[0])
    {
      __shared__ float s_LUTcoeff[120];
      if (tid < c_N_PolLUTCoeff[0])
        s_LUTcoeff[tid] = d_LUTPolCoeff_f[tid];
      __syncthreads();
      g_SLMuc[idx] = applyPolLUT(phase2pi, X, Y, s_LUTcoeff);
    }
    else if (c_applyLUT_b[0])
    {
      __shared__ unsigned char s_LUT[256];
      if (tid < 256)
        s_LUT[tid] = g_LUT[tid];
      __syncthreads();
      g_SLMuc[idx] = s_LUT[phase2int32(phase2pi)];
    }
    else
      g_SLMuc[idx] = phase2uc(phase2pi);
  }
}

__global__ void calculateIobtained(unsigned char *g_pSLM_uc, float *g_Iobtained)
{
  int blockSize = c_data_w[0];
  int spot_number = blockIdx.x;
  int tid = threadIdx.x;
  int i = tid;

  __shared__ float s_Vre[SLM_SIZE];
  __shared__ float s_Vim[SLM_SIZE];

  s_Vre[tid] = 0.0f;
  s_Vim[tid] = 0.0f;

  float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
  float Y = - c_SLMpitch_f[0] * c_half_w_f[0];

  float pSLM_1;
  float p;
  while (i < c_N_pixels[0])
  {
    pSLM_1 = 2.0f*M_PI*(float)g_pSLM_uc[i]/255.0f - M_PI;
    p = pSLM_1 - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2 * (X * c_x[spot_number] + Y * c_y[spot_number]));

    s_Vre[tid] += cosf(p);
    s_Vim[tid] += sinf(p);

    i += blockSize;
    Y += c_SLMpitch_f[0];
  }
  /*__syncthreads();
  if (tid < 512)
  {
    s_Vre[tid] += s_Vre[tid + 512];
    s_Vim[tid] += s_Vim[tid + 512];
  } */
  __syncthreads();
  if ((tid < 256)&&(SLM_SIZE>256))
  {
    s_Vre[tid] += s_Vre[tid + 256];
    s_Vim[tid] += s_Vim[tid + 256];
  }
  __syncthreads();

  if (tid < 128)
  {
    s_Vre[tid] += s_Vre[tid + 128];
    s_Vim[tid] += s_Vim[tid + 128];
  }
  __syncthreads();

  if (tid < 64)
  {
    s_Vre[tid] += s_Vre[tid + 64];
    s_Vim[tid] += s_Vim[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduceC(s_Vre, s_Vim, tid);

  if (tid == 0)
  {
    float spotRe_f = s_Vre[0] / c_N_pixels_f[0];      //512!
    float spotIm_f = s_Vim[0] / c_N_pixels_f[0];
    float amp = hypotf(spotRe_f, spotIm_f);
    g_Iobtained[spot_number] = amp*amp;
  }
}
__global__ void calculateIandPhase(unsigned char *g_pSLM_uc, float *g_Iobtained, float *g_Pobtained)
{
  int blockSize = c_data_w[0];
  int spot_number = blockIdx.x;
  int tid = threadIdx.x;
  int i = tid;

  __shared__ float s_Vre[SLM_SIZE];
  __shared__ float s_Vim[SLM_SIZE];

  s_Vre[tid] = 0.0f;
  s_Vim[tid] = 0.0f;

  float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
  float Y = - c_SLMpitch_f[0] * c_half_w_f[0];

  float pSLM_1;
  float p;
  while (i < c_N_pixels[0])
  {
    pSLM_1 = 2.0f*M_PI*(float)g_pSLM_uc[i]/255.0f - M_PI;
    p = pSLM_1 - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2 * (X * c_x[spot_number] + Y * c_y[spot_number]));

    s_Vre[tid] += cosf(p+2*M_PI*c_z[spot_number]);
    s_Vim[tid] += sinf(p+2*M_PI*c_z[spot_number]);

    i += blockSize;
    Y += c_SLMpitch_f[0];
  }
  /*__syncthreads();
  if (tid < 512)
  {
    s_Vre[tid] += s_Vre[tid + 512];
    s_Vim[tid] += s_Vim[tid + 512];
  } */
  __syncthreads();
  if ((tid < 256)&&(SLM_SIZE>256))
  {
    s_Vre[tid] += s_Vre[tid + 256];
    s_Vim[tid] += s_Vim[tid + 256];
  }
  __syncthreads();

  if (tid < 128)
  {
    s_Vre[tid] += s_Vre[tid + 128];
    s_Vim[tid] += s_Vim[tid + 128];
  }
  __syncthreads();

  if (tid < 64)
  {
    s_Vre[tid] += s_Vre[tid + 64];
    s_Vim[tid] += s_Vim[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduceC(s_Vre, s_Vim, tid);

  if (tid == 0)
  {
    float spotRe_f = s_Vre[0] / c_N_pixels_f[0];      //512!
    float spotIm_f = s_Vim[0] / c_N_pixels_f[0];
    float amp = hypotf(spotRe_f, spotIm_f);
    g_Pobtained[spot_number] = atan2f(spotIm_f , spotRe_f);
    g_Iobtained[spot_number] = amp*amp;
  }
}

// Functions for GS with Fresnel propagation

// Propagate from the SLM to the spot positions using Fresnel summation
// works only for blocksize = SLMsize
__global__ void PropagateToSpotPositions_Fresnel(float *g_pSLM2pi, float *g_spotRe_f, float *g_spotIm_f)
{
  __shared__ float s_Vre[SLM_SIZE];
  __shared__ float s_Vim[SLM_SIZE];

  int spot_number = blockIdx.x;
  int tid = threadIdx.x;
  int i = tid;

  s_Vre[tid] = 0.0f;
  s_Vim[tid] = 0.0f;
  int blockSize = blockDim.x;
  float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
  float Y = - c_SLMpitch_f[0] * c_half_w_f[0];
  float p;

  while (i < c_N_pixels[0]) {
    p = g_pSLM2pi[i] - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2.0f * (X * c_x[spot_number] + Y * c_y[spot_number]));

    s_Vre[tid] += cosf(p);
    s_Vim[tid] += sinf(p);

    i += blockSize;
    Y += c_SLMpitch_f[0];
  }
  /*__syncthreads();

  if (tid < 512)
  {
    s_Vre[tid] += s_Vre[tid + 512];
    s_Vim[tid] += s_Vim[tid + 512];
  } */
  __syncthreads();

  if ((tid < 256)&&(SLM_SIZE>256)) {
    s_Vre[tid] += s_Vre[tid + 256];
    s_Vim[tid] += s_Vim[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    s_Vre[tid] += s_Vre[tid + 128];
    s_Vim[tid] += s_Vim[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    s_Vre[tid] += s_Vre[tid + 64];
    s_Vim[tid] += s_Vim[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduceC(s_Vre, s_Vim, tid);

  if (tid == 0) {
    g_spotRe_f[spot_number] = s_Vre[0];// / c_N_pixels_f[0];
    g_spotIm_f[spot_number] = s_Vim[0];// / c_N_pixels_f[0];
  }
}

// Obtain phases in SLM plane
__global__ void PropagateToSLM_Fresnel(float *g_spotRe_f, float *g_spotIm_f, float *g_pSLM2pi, float *g_weights, int iteration, float *g_pSLMstart, float *g_Iobtained, bool getpSLM255, unsigned char *g_pSLM255_uc, unsigned char *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
  __shared__ float s_aSpot[MAX_SPOTS], s_aSpotsMean, s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  float reSLM = 0.0f, imSLM = 0.0f, pSLM2pi_f = 0.0f;

  if (idx < c_N_pixels[0]) {
    if (tid < c_N_spots[0]) {
      float spotRe_f = g_spotRe_f[tid];
      float spotIm_f = g_spotIm_f[tid];
      s_pSpot[tid] = atan2f(spotIm_f, spotRe_f);
      s_aSpot[tid] = hypotf(spotRe_f, spotIm_f)/c_desiredAmp[tid];
      if (iteration != 0) {
        s_weight[tid] = g_weights[tid + iteration*c_N_spots[0]];
      } else {
        s_aSpot[tid] = (s_aSpot[tid]<0.5f) ? 0.5f : s_aSpot[tid];
        s_weight[tid] = c_desiredAmp[tid];
      }
    }
    __syncthreads();

    //compute weights
    if (tid == 0) {
      float s_aSpot_sum = 0.0f;
      for (int jj = 0; jj < c_N_spots[0]; jj++) {
        s_aSpot_sum += s_aSpot[jj];
      }
      s_aSpotsMean = s_aSpot_sum / (float)c_N_spots[0];
    }
    __syncthreads();

    if (tid < c_N_spots[0]) {
      s_weight[tid] = s_weight[tid] * s_aSpotsMean / s_aSpot[tid];
      if (!getpSLM255)                      //Copy weights to use as initial value next run
        g_weights[tid + c_N_spots[0]*(iteration+1)] = s_weight[tid];
      //else
      //  g_weights[tid] = s_weight[tid];             //Transferring weights to next run may give diverging weights
      if (c_saveI_b[0])
        g_Iobtained[tid + c_N_spots[0]*iteration] = s_aSpot[tid]*s_aSpot[tid];      //may be excluded, used for monitoring only
    }
    __syncthreads();

    //get pixel coordinates
    int X_int = getXint(idx);
    int Y_int = getYint(idx, X_int);
    float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
    float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);

    //compute SLM pSpot by summing contribution from all spots
    for (int k = 0; k < c_N_spots[0]; k++) {
      float delta = M_PI * c_z[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * c_x[k] + Y * c_y[k]);
      reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
      imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
    }
    pSLM2pi_f = atan2f(imSLM, reSLM);

    if (c_useRPC_b[0]) {      //Apply RPC (restricted Phase Change)
      float pSLMstart = g_pSLMstart[idx];
      if (fabs(pSLM2pi_f - pSLMstart) > c_alphaRPC_f[0])
        pSLM2pi_f = pSLMstart;
      if (getpSLM255)
        g_pSLMstart[idx] = pSLM2pi_f;
    }

    if (getpSLM255) {         //Compute final SLM phases and write to global memory...
      if (c_useAberrationCorr_b[0])
        pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);

      if (c_usePolLUT_b[0]) {
        __shared__ float s_LUTcoeff[120];
        if (tid < c_N_PolLUTCoeff[0])
          s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff);
      } else if (c_applyLUT_b[0]) {
        __shared__ unsigned char s_LUT[256];
        if (tid < 256)
          s_LUT[tid] = g_LUT[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
      } else {
        g_pSLM255_uc[idx] = phase2uc(pSLM2pi_f);
      }
    }
    g_pSLM2pi[idx] = pSLM2pi_f; //...or write intermediate pSpot to global memory
  }
}

// Convert from unsigned char [0, 255] to float [-pi, pi]
__global__ void uc2f(float *f, unsigned char *uc, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    f[idx] = uc[idx]*2.0f*M_PI/256.0f - M_PI;
  }
}

// Custom debug functions
inline void mSafeCall(hipError_t status, int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do {
    if (status != hipSuccess) {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      if (status != HIPFFT_SUCCESS)
        exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess) {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  } while (0);
#endif
  return;
}

inline void mCufftSafeCall(hipfftResult_t status, int line, char *file)
{
#ifdef M_CUDA_DEBUG
  if(status != HIPFFT_SUCCESS)
  {
    char CUDAmessage[200] = "CUFFT error, CUDA says:\n ";
    switch (status) {
            case HIPFFT_INVALID_PLAN:   strcat(CUDAmessage,"HIPFFT_INVALID_PLAN\n");break;
            case HIPFFT_ALLOC_FAILED:   strcat(CUDAmessage,"HIPFFT_ALLOC_FAILED\n");break;
            case HIPFFT_INVALID_TYPE:   strcat(CUDAmessage,"HIPFFT_INVALID_TYPE\n");break;
            case HIPFFT_INVALID_VALUE:  strcat(CUDAmessage,"HIPFFT_INVALID_VALUE\n");break;
            case HIPFFT_INTERNAL_ERROR: strcat(CUDAmessage,"HIPFFT_INTERNAL_ERROR\n");break;
            case HIPFFT_EXEC_FAILED:    strcat(CUDAmessage,"HIPFFT_EXEC_FAILED\n");break;
            case HIPFFT_SETUP_FAILED:   strcat(CUDAmessage,"HIPFFT_SETUP_FAILED\n");break;
            case HIPFFT_INVALID_SIZE:   strcat(CUDAmessage,"HIPFFT_INVALID_SIZE\n");break;
            //case HIPFFT_UNALIGNED_DATA: strcat(CUDAmessage,"HIPFFT_UNALIGNED_DATA\n");break;
            default: strcat(CUDAmessage,"CUFFT Unknown error code\n");

    }
    sprintf(CUDAmessage,  "%son line: %d\nin file: %s", CUDAmessage, line, file);
    printf("%s", CUDAmessage);
    exit(-1);
  }
  hipDeviceSynchronize();
  hipError_t status2 = hipGetLastError();
  if(status2!=hipSuccess)
  {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status2));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
  }
#endif
  return;
}
inline void mCheckError(int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do
  {
    hipError_t status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  }while(0);
#endif
  return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    float *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (float*)malloc(length * sizeof (float));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(float), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %f", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataCC(hipfftComplex *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 25;
    hipfftComplex *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (hipfftComplex*)malloc(length * sizeof (hipfftComplex));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s re: %f im: %f", MessageString, h_data[ii].x, h_data[ii].y);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataUC(unsigned char *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    unsigned char *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (unsigned char*)malloc(length * sizeof (unsigned char));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(unsigned char), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %hhu", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    int *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (int*)malloc(length * sizeof (int));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(int), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %d", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

// Calculate amplitudes in positions given by x, y, and z from a given hologram
int get_amp_and_phase(float *x_spots, float *y_spots, float *z_spots, float *h_pSLM_uc, int N_spots_all, int data_w, float *h_I_obt, float *h_Phase_obt)
{
  float *d_Iobtained_all;
  float *d_Pobtained_all;
  hipMalloc((void**)&d_Iobtained_all, N_spots_all*sizeof(float) );
  hipMalloc((void**)&d_Pobtained_all, N_spots_all*sizeof(float) );
  hipMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, hipMemcpyHostToDevice);
  int offset = 0;
  int N_spots_rem = N_spots_all;
  int N_spots_this;
  while (N_spots_rem > 0)
  {
    N_spots_this = (N_spots_rem > MAX_SPOTS) ? MAX_SPOTS : N_spots_rem;
    hipMemcpyToSymbol(HIP_SYMBOL(c_x), x_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_y), y_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_z), z_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    calculateIandPhase<<<N_spots_this, 512>>>(d_pSLM_uc, d_Iobtained_all+offset, d_Pobtained_all+offset);
    //calculateIobtained(unsigned char *g_pSLM_uc, float *g_Iobtained)
    hipDeviceSynchronize();

    N_spots_rem -= MAX_SPOTS;
    offset += MAX_SPOTS;
  }
  hipMemcpy(h_I_obt, d_Iobtained_all, N_spots_all*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_Phase_obt, d_Pobtained_all, N_spots_all*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_Iobtained_all);
  hipFree(d_Pobtained_all);

  status = hipGetLastError();

  return status;
}
//compute amps for constant total int
/*void computeAmps(float *h_I, float *h_desiredAmp, float *x, float *y, int N_spots, float e_desired)
{
  float SLMsize = (float)SLM_SIZE;
  float Isum = 0.0f;
  for (int i = 0; i<N_spots; i++)
    Isum += h_I[i];
  for (int j = 0; j<N_spots; j++)
  {
    float sincx_rec = 1.0f;//= (x==0)? 1.0f:((M_PI*x[j]/SLMsize)/sinf(M_PI*x[j]/SLMsize));
    float sincy_rec = 1.0f;//(y==0)? 1.0f:((M_PI*y[j]/SLMsize)/sinf(M_PI*y[j]/SLMsize));
    h_desiredAmp[j] = (h_I[j] <= 0.0f) ? 1.0f:(sincx_rec * sincy_rec * sqrtf(e_desired*h_I[j]/Isum)*SLMsize*SLMsize);
  }
}*/

/*
// Obtain phases in SLM plane (ALTERNATIVE VERSION)
// works only for blocksize 512 and max 512 spots
__global__ void PropagateToSLM_Fresnel(float *g_x,
                float *g_y,
                float *g_z,
                float *g_I,
                float *g_spotRe_f,
                float *g_spotIm_f,
                float *g_pSLM2pi,
                int N_pixels,
                int N_spots,
                float *g_weights,
                int iteration,
                float *g_pSLMstart,
                float RPC,
                float *g_amps,
                bool getpSLM255,
                unsigned char *g_pSLM255_uc,
                unsigned char *g_LUT,
                bool ApplyLUT_b,
                bool UseAberrationCorr_b,
                float *g_AberrationCorr_f,
                bool UsePolLUT_b,
                float *g_LUTPolCoeff_f,
                int N_PolCoeff)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;
  __shared__ float s_aSpot[MAX_SPOTS], s_aSpotsMean, s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
  __shared__ float s_xm[MAX_SPOTS];
  __shared__ float s_ym[MAX_SPOTS];
  __shared__ float s_zm[MAX_SPOTS];
  float reSLM = 0.0f, imSLM = 0.0f, pSLM2pi_f = 0.0f;
  __shared__ float s_weights_sum;
  if (idx<N_pixels)
  {
    //float N = 512;
    //int logN = (int)log2(N);

    if (tid<c_N_spots[0])
    {
      float spotRe_f = g_spotRe_f[tid];
      float spotIm_f = g_spotIm_f[tid];
      s_pSpot[tid] = atan2f(spotIm_f, spotRe_f);

      s_weight[tid] = g_weights[tid + iteration*c_N_spots[0]] * sqrtf(g_I[tid])/hypotf(spotRe_f, spotIm_f);

      s_xm[tid] = g_x[tid];
      s_ym[tid] = g_y[tid];
      s_zm[tid] = g_z[tid];

      __syncthreads();
      if  (tid==0)
      {
        s_weights_sum = 0.0f;
        for (int jj=0; jj<c_N_spots[0];jj++)
        {
          s_weights_sum += s_weight[jj];
        }
      }
      __syncthreads();
      s_weight[tid] /= s_weights_sum;
      s_weight[tid] = (s_weight[tid]<0.0001f) ? 0.0001f : s_weight[tid];
      //////////////////////////////////////////////////////////////////////////////////
      if (getpSLM255)                     //Copy weights to use as initial value next run
        g_weights[tid] = s_weight[tid];
      else
        g_weights[tid + c_N_spots[0]*(iteration+1)] = s_weight[tid];
      //g_amps[tid + c_N_spots[0]*iteration] = s_aSpot[tid];      //may be excluded, used for monitoring only
    }
    __syncthreads();
    //get pixel coordinates
    float X = c_SLMpitch_f[0] * ((float)threadIdx.x - 256.0f);        //512!
    float Y = c_SLMpitch_f[0] * ((float)blockIdx.x - 256.0f);

    //compute SLM pSpot by summing contribution from all spots
    for (int k=0; k<c_N_spots[0]; k++)
    {
      float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * s_xm[k] + Y * s_ym[k]);
      reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
      imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
    }
    pSLM2pi_f = atan2f(imSLM, reSLM);


    if (RPC < (2.0f*M_PI))      //Apply RPC (restricted Phase Change)
    {
      float pSLMstart = g_pSLMstart[idx];
      if (fabs(pSLM2pi_f - pSLMstart) > RPC)
        pSLM2pi_f = pSLMstart;
      if (getpSLM255)
        g_pSLMstart[idx] = pSLM2pi_f;
    }

    if (getpSLM255)         //Compute final SLM phases and write to global memory...
    {
      if (UseAberrationCorr_b)
        pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);

      if (UsePolLUT_b)
      {
        __shared__ float s_LUTcoeff[120];
        if (tid < N_PolCoeff)
          s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff);
      }
      else if (ApplyLUT_b)
      {
        __shared__ unsigned char s_LUT[256];
        if (tid < 256)
          s_LUT[tid] = g_LUT[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
      }
      else
        g_pSLM255_uc[idx] = phase2uc(pSLM2pi_f);
    }
    else
      g_pSLM2pi[idx] = pSLM2pi_f; //...or write intermediate pSpot to global memory
  }
}*/

int main()
{
  srand(1);
  const int numPixels = SLM_SIZE * SLM_SIZE; // 512^2
  const int N = 4; // Four spots
  const int method = 1; // 0 => Direct, 1 => Fresnel, 2 => FFT
  const int iterations = 10; // 10 iterations for convergence

  // These form a quadrant across four planes
  float x[] = {-128.0, -128.0, 127.0, 127.0};
  float y[] = {127.0, -128.0, 127.0, -128.0};
  float z[] = {1.0, 2.0, 3.0, 4.0};
  float I[] = {0.12, 0.34, 0.56, 0.78};

  float *polLUT = (float *) malloc(120 * sizeof(float));
  for (int i = 0; i < 120; i++) {
    polLUT[i] = random() / ((float) RAND_MAX);
  }

  unsigned char *hologram = (unsigned char *) malloc(numPixels * sizeof(unsigned char));
  float *init_phases = (float *) malloc(numPixels * sizeof(float)); // [-pi, pi]
  for (int i = 0; i < numPixels; i++) {
    hologram[i] = 0.0f;
    init_phases[i] = (2.0 * M_PI * (random() / ((float) RAND_MAX))) - M_PI;
  }

  float *amps = (float *) malloc(N * iterations * sizeof(float));
  for (int i = 0; i < N * iterations; i++) {
    amps[i] = 0.0f;
  }

  if (setup(init_phases) != 0) {
    printf("Init failed.\n");
    exit(1);
  }

  if (corrections(0, NULL, 1, 7, polLUT, 1, 1, 0, NULL) != 0) {
    printf("Correction setup failed.\n");
    exit(1);
  }

  double t = get_clock();

  if (generate_hologram(hologram, x, y, z, I, N, iterations, amps, method) != 0) {
    printf("Computation failed.\n");
    exit(1);
  }

  t = get_clock() - t;

  if (finish() != 0) {
    printf("Cleanup failed.\n");
    exit(1);
  }

  printf("Total time = %12.8lf seconds\n", t);

  // Save output
  FILE *hfile = fopen("orig_hologram.dat", "w");
  for (int i = 0; i < numPixels; i++) {
    fprintf(hfile, "%hhu\n", hologram[i]);
  }

  FILE *afile = fopen("orig_amps.dat", "w");
  for (int i = 0; i < N * iterations; i++) {
    fprintf(afile, "%f\n", amps[i]);
  }

  fclose(hfile);
  fclose(afile);

  return 0;
}

