/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

// Activates a number of custom debug macros
#define M_CUDA_DEBUG

// Includes

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 64 // Even 64 is pushing it
#define MAX_POL 120
#define MAX_UCHAR 256

#define BLOCK_SIZE 256

// FIXME: This shouldn't be hardcoded
#define SLM_SIZE 256

// Use bitwise modulo operations if the SLM size is a power of 2
#if (((SLM_SIZE - 1) & (SLM_SIZE)) == 0)
#define SLMPOW2
#endif

// Forward declaration
void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n);

// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, const char *file);
inline void mCheckError(int line, const char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataUC(unsigned char *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

// Hologram info
unsigned char *d_hologram; // [0, 255]
float *d_hologramPhase;    // [-pi, pi]
float *d_prevHologramPhase;
int slmDim;
float slmPitch;
int numPixels;
int numIterations;
int hologramMemSize;

// Spot info
float *d_x;
float *d_y;
float *d_z;
float *d_desiredAmp;
float *d_spotRe;
float *d_spotIm;
float *d_phase;
float *d_weights;
float *d_obtainedI;
int numSpots;
bool saveSpotI;
int weightMemSize;

// Aberration correction
bool useAC;
float *d_aberrationCoeffs;

// Spatially Varying Phase Response
bool useSVPR;
int polOrder;
int numPolCoeffs;
float *d_polCoeffs;

// phase-to-uc LUT
bool useLUT;
unsigned char *d_lut;

// Restricted Phase Change
bool useRPC;
float alpha;

// Error checking
char CUDAmessage[100];
hipError_t status;

/****************************** Device functions ******************************/

// Convert from unsigned char [0, 255] to phase (float) [-pi, pi]
__device__ inline float uc2phase(const unsigned char uc)
{
  return (((float) uc) * 2.0f * M_PI/256.0f - M_PI);
}

// Convert from phase (float) [-pi, pi] to unsigned char [0, 255]
__device__ inline unsigned char phase2uc(const float phase)
{
  return (unsigned char) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Convert from phase (float) [-pi, pi] to int
__device__ inline int phase2int(const float phase)
{
  return (int) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Apply wavefront distortion/aberration correction
__device__ inline float applyAberrationCorrection(float pSpot, const float correction)
{
  // First apply correction, then apply mod([-pi, pi], pSpot)
  pSpot = pSpot - correction;
  return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI)));
}

// Get x coordinate from global thread ID
__device__ inline int getXIdx(const int index, const int slmDim)
{
#ifdef SLMPOW2
  int idx = index & (slmDim - 1);
#else
  int idx = index % slmDim;
#endif
  return idx;
}

// Get y coordinate from global thread ID and x coordinate
__device__ inline int getYIdx(const int index, const int xIdx, const float slmPitch)
{
#ifdef SLMPOW2
  int idx = (index - xIdx) >> 9; // FIXME
#else
  int idx = floor(((float) (index - xIdx)) * slmPitch);
#endif
  return idx;
}

// Get pixel coordinates in [-0.5, 0.5]
__device__ inline float getPixelCoords(const int index, const int slmDim, const float slmPitch)
{
  return (slmPitch * ((float) (index - (slmDim >> 1))));
}

// Compute phase from pixel position and spot position
__device__ inline float computePhase(const float x,
                                     const float y,
                                     const float spotx,
                                     const float spoty,
                                     const float spotz)
{
  // TODO: use equation from paper?
  return (M_PI * (spotz * (x*x + y*y) + 2.0f * (x*spotx + y*spoty)));
}

// Apply SVPR
__device__ unsigned char applySVPR(const float phase,
                                   const float x,
                                   const float y,
                                   const float * const coeff,
                                   const int numPolCoeffs)
{
  float result = 0.0f;
  switch (numPolCoeffs) {
    case 120:
      result += coeff[84]*x*x*x*x*x*x*x;
      result += coeff[85]*x*x*x*x*x*x*y;
      result += coeff[86]*x*x*x*x*x*x*phase;
      result += coeff[87]*x*x*x*x*x*y*y;
      result += coeff[88]*x*x*x*x*x*y*phase;
      result += coeff[89]*x*x*x*x*x*phase*phase;
      result += coeff[90]*x*x*x*x*y*y*y;
      result += coeff[91]*x*x*x*x*y*y*phase;
      result += coeff[92]*x*x*x*x*y*phase*phase;
      result += coeff[93]*x*x*x*x*phase*phase*phase;
      result += coeff[94]*x*x*x*y*y*y*y;
      result += coeff[95]*x*x*x*y*y*y*phase;
      result += coeff[96]*x*x*x*y*y*phase*phase;
      result += coeff[97]*x*x*x*y*phase*phase*phase;
      result += coeff[98]*x*x*x*phase*phase*phase*phase;
      result += coeff[99]*x*x*y*y*y*y*y;
      result += coeff[100]*x*x*y*y*y*y*phase;
      result += coeff[101]*x*x*y*y*y*phase*phase;
      result += coeff[102]*x*x*y*y*phase*phase*phase;
      result += coeff[103]*x*x*y*phase*phase*phase*phase;
      result += coeff[104]*x*x*phase*phase*phase*phase*phase;
      result += coeff[105]*x*y*y*y*y*y*y;
      result += coeff[106]*x*y*y*y*y*y*phase;
      result += coeff[107]*x*y*y*y*y*phase*phase;
      result += coeff[108]*x*y*y*y*phase*phase*phase;
      result += coeff[109]*x*y*y*phase*phase*phase*phase;
      result += coeff[110]*x*y*phase*phase*phase*phase*phase;
      result += coeff[111]*x*phase*phase*phase*phase*phase*phase;
      result += coeff[112]*y*y*y*y*y*y*y;
      result += coeff[113]*y*y*y*y*y*y*phase;
      result += coeff[114]*y*y*y*y*y*phase*phase;
      result += coeff[115]*y*y*y*y*phase*phase*phase;
      result += coeff[116]*y*y*y*phase*phase*phase*phase;
      result += coeff[117]*y*y*phase*phase*phase*phase*phase;
      result += coeff[118]*y*phase*phase*phase*phase*phase*phase;
      result += coeff[119]*phase*phase*phase*phase*phase*phase*phase;
    case 84:
      result += coeff[56]*x*x*x*x*x*x;
      result += coeff[57]*x*x*x*x*x*y;
      result += coeff[58]*x*x*x*x*x*phase;
      result += coeff[59]*x*x*x*x*y*y;
      result += coeff[60]*x*x*x*x*y*phase;
      result += coeff[61]*x*x*x*x*phase*phase;
      result += coeff[62]*x*x*x*y*y*y;
      result += coeff[63]*x*x*x*y*y*phase;
      result += coeff[64]*x*x*x*y*phase*phase;
      result += coeff[65]*x*x*x*phase*phase*phase;
      result += coeff[66]*x*x*y*y*y*y;
      result += coeff[67]*x*x*y*y*y*phase;
      result += coeff[68]*x*x*y*y*phase*phase;
      result += coeff[69]*x*x*y*phase*phase*phase;
      result += coeff[70]*x*x*phase*phase*phase*phase;
      result += coeff[71]*x*y*y*y*y*y;
      result += coeff[72]*x*y*y*y*y*phase;
      result += coeff[73]*x*y*y*y*phase*phase;
      result += coeff[74]*x*y*y*phase*phase*phase;
      result += coeff[75]*x*y*phase*phase*phase*phase;
      result += coeff[76]*x*phase*phase*phase*phase*phase;
      result += coeff[77]*y*y*y*y*y*y;
      result += coeff[78]*y*y*y*y*y*phase;
      result += coeff[79]*y*y*y*y*phase*phase;
      result += coeff[80]*y*y*y*phase*phase*phase;
      result += coeff[81]*y*y*phase*phase*phase*phase;
      result += coeff[82]*y*phase*phase*phase*phase*phase;
      result += coeff[83]*phase*phase*phase*phase*phase*phase;
    case 56:
      result += coeff[35]*x*x*x*x*x;
      result += coeff[36]*x*x*x*x*y;
      result += coeff[37]*x*x*x*x*phase;
      result += coeff[38]*x*x*x*y*y;
      result += coeff[39]*x*x*x*y*phase;
      result += coeff[40]*x*x*x*phase*phase;
      result += coeff[41]*x*x*y*y*y;
      result += coeff[42]*x*x*y*y*phase;
      result += coeff[43]*x*x*y*phase*phase;
      result += coeff[44]*x*x*phase*phase*phase;
      result += coeff[45]*x*y*y*y*y;
      result += coeff[46]*x*y*y*y*phase;
      result += coeff[47]*x*y*y*phase*phase;
      result += coeff[48]*x*y*phase*phase*phase;
      result += coeff[49]*x*phase*phase*phase*phase;
      result += coeff[50]*y*y*y*y*y;
      result += coeff[51]*y*y*y*y*phase;
      result += coeff[52]*y*y*y*phase*phase;
      result += coeff[53]*y*y*phase*phase*phase;
      result += coeff[54]*y*phase*phase*phase*phase;
      result += coeff[55]*phase*phase*phase*phase*phase;
    case 35:
      result += coeff[20]*x*x*x*x;
      result += coeff[21]*x*x*x*y;
      result += coeff[22]*x*x*x*phase;
      result += coeff[23]*x*x*y*y;
      result += coeff[24]*x*x*y*phase;
      result += coeff[25]*x*x*phase*phase;
      result += coeff[26]*x*y*y*y;
      result += coeff[27]*x*y*y*phase;
      result += coeff[28]*x*y*phase*phase;
      result += coeff[29]*x*phase*phase*phase;
      result += coeff[30]*y*y*y*y;
      result += coeff[31]*y*y*y*phase;
      result += coeff[32]*y*y*phase*phase;
      result += coeff[33]*y*phase*phase*phase;
      result += coeff[34]*phase*phase*phase*phase;
    case 20:
      result += coeff[0];
      result += coeff[1]*x;
      result += coeff[2]*y;
      result += coeff[3]*phase;
      result += coeff[4]*x*x;
      result += coeff[5]*x*y;
      result += coeff[6]*x*phase;
      result += coeff[7]*y*y;
      result += coeff[8]*y*phase;
      result += coeff[9]*phase*phase;
      result += coeff[10]*x*x*x;
      result += coeff[11]*x*x*y;
      result += coeff[12]*x*x*phase;
      result += coeff[13]*x*y*y;
      result += coeff[14]*x*y*phase;
      result += coeff[15]*x*phase*phase;
      result += coeff[16]*y*y*y;
      result += coeff[17]*y*y*phase;
      result += coeff[18]*y*phase*phase;
      result += coeff[19]*phase*phase*phase;
      break;
    default:
      result = 0.0f;
      break;
  }

  if (result < 0.0f)
    result = 0.0f;

  return (unsigned char) result;
}

// Performs intra-warp reduction. The arrays are in shared memory.
__device__ void warpReduce(volatile float *vRe, volatile float *vIm, int tid)
{
  vRe[tid] += vRe[tid + 32];
  vIm[tid] += vIm[tid + 32];

  vRe[tid] += vRe[tid + 16];
  vIm[tid] += vIm[tid + 16];

  vRe[tid] += vRe[tid + 8];
  vIm[tid] += vIm[tid + 8];

  vRe[tid] += vRe[tid + 4];
  vIm[tid] += vIm[tid + 4];

  vRe[tid] += vRe[tid + 2];
  vIm[tid] += vIm[tid + 2];

  vRe[tid] += vRe[tid + 1];
  vIm[tid] += vIm[tid + 1];
}

// Apply corrections to precalculated hologram
__global__ void applyCorrections(// Hologram information
                                 unsigned char * const hologram,       // hologram to use
                                 const unsigned int slmDim,            // SLM's dimension
                                 const float slmPitch,                 // 1/slmDim
                                 // Correction information
                                 const bool useAC,                     // use aberration correction
                                 const float * const aberrationCoeffs, // correction coefficients
                                 const bool useSVPR,                   // use spatially varying phase response
                                 const int numPolCoeffs,               // number of polynomial coefficients
                                 const float * const polCoeffs,        // polynomial coefficients
                                 const bool useLUT,                    // use LUT for phase-to-uc conversion
                                 const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  const int tid = threadIdx.x;
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float pixelPhase = uc2phase(hologram[idx]);
  if (useAC)
    pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

  if (useSVPR) {
    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    __shared__ float coeff[MAX_POL];
    if (tid < numPolCoeffs)
      coeff[tid] = polCoeffs[tid];
    __syncthreads();

    hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
  } else if (useLUT) {
    __shared__ unsigned char lut_t[MAX_UCHAR];
    if (tid < MAX_UCHAR)
      lut_t[tid] = lut[tid];
    __syncthreads();

    hologram[idx] = lut_t[phase2int(pixelPhase)];
  } else {
    hologram[idx] = phase2uc(pixelPhase);
  }
}

// Calculate hologram using "Lenses and Prisms"
__global__ void lensesAndPrisms(// Hologram information
                                unsigned char * const hologram,       // hologram to use
                                const unsigned int slmDim,            // SLM's dimension
                                const float slmPitch,                 // 1/slmDim
                                const unsigned int numPixels,         // number of pixels in SLM
                                // Spot information
                                const float * const spotX,            // x coordinates of spots
                                const float * const spotY,            // y coordinates of spots
                                const float * const spotZ,            // z coordinates of spots
                                const float * const spotDesiredAmp,   // desired amplitudes of spots
                                const unsigned int numSpots,          // number of spots
                                // Correction information
                                const bool useAC,                     // use aberration correction
                                const float * const aberrationCoeffs, // correction coefficients
                                const bool useSVPR,                   // use spatially varying phase response
                                const int numPolCoeffs,               // number of polynomial coefficients
                                const float * const polCoeffs,        // polynomial coefficients
                                const bool useLUT,                    // use LUT for phase-to-uc conversion
                                const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  __shared__ float spotx[MAX_SPOTS];
  __shared__ float spoty[MAX_SPOTS];
  __shared__ float spotz[MAX_SPOTS];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  if (idx < numPixels) {
    if (tid < numSpots) {
      spotx[tid] = spotX[tid];
      spoty[tid] = spotY[tid];
      spotz[tid] = spotZ[tid];
    }
    __syncthreads();

    float pixelPhase;
    float vRe = 0.0f;
    float vIm = 0.0f;

    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    // Compute the pixel's phase by summing contributions from all spots
    for (int i = 0; i < numSpots; i++) {
      // TODO: Add variable phases to function call
      pixelPhase = computePhase(x, y, spotx[i], spoty[i], spotz[i]);
      vRe += spotDesiredAmp[i] * cosf(pixelPhase);
      vIm += spotDesiredAmp[i] * sinf(pixelPhase);
    }

    pixelPhase = atan2f(vIm, vRe); // [-pi, pi]
    if (useAC)
      pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

    if (useSVPR) {
      __shared__ float coeff[MAX_POL];
      if (tid < numPolCoeffs)
        coeff[tid] = polCoeffs[tid];
      __syncthreads();

      hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
    } else if (useLUT) {
      __shared__ unsigned char lut_t[MAX_UCHAR];
      if (tid < MAX_UCHAR)
        lut_t[tid] = lut[tid];
      __syncthreads();

      hologram[idx] = lut_t[phase2int(pixelPhase)];
    } else {
      hologram[idx] = phase2uc(pixelPhase);
    }
  }
}

__global__ void calculateI(// Hologram information
                           const unsigned char * const hologram, // hologram to use
                           const unsigned int slmDim,            // SLM's dimension
                           const float slmPitch,                 // 1/slmDim
                           const unsigned int numPixels,         // number of pixels in SLM
                           // Spot information
                           const float * const spotX,            // x coordinates of spots
                           const float * const spotY,            // y coordinates of spots
                           const float * const spotZ,            // z coordinates of spots
                           float * const spotI)                  // spot intensities
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  while (i < numPixels) {
    pixelPhase = uc2phase(hologram[i]) - computePhase(x, y, spotx, spoty, spotz);

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0] / ((float) numPixels);
    float im = vIm[0] / ((float) numPixels);
    spotI[spotNumber] = re*re + im*im;
  }
}

__global__ void calculateIAndPhase(// Hologram information
                                   const unsigned char * const hologram, // hologram to use
                                   const unsigned int slmDim,            // SLM's dimension
                                   const float slmPitch,                 // 1/slmDim
                                   const unsigned int numPixels,         // number of pixels in SLM
                                   // Spot information
                                   const float * const spotX,            // x coordinates of spots
                                   const float * const spotY,            // y coordinates of spots
                                   const float * const spotZ,            // z coordinates of spots
                                   float * const spotI,                  // spot intensities
                                   float * const spotP)                  // spot phases
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  while (i < numPixels) {
    pixelPhase = uc2phase(hologram[i]) - computePhase(x, y, spotx, spoty, spotz);
    pixelPhase += 2.0f * M_PI * spotZ[spotNumber];

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0] / ((float) numPixels);
    float im = vIm[0] / ((float) numPixels);
    spotI[spotNumber] = re*re + im*im;
    spotP[spotNumber] = atan2f(im, re);
  }
}

__device__ inline float normalize(int idx, int support){
	return ((float)idx - (support >> 1))/support;
}

// Propagate from the SLM to the spot positions using Fresnel summation
// FIXME: Works only for blocksize = SLMsize
__global__ void propagateToSpotPositions(// Hologram information
                                         const float * const hologramPhase,    // hologram's phase
                                         const unsigned int slmDim,            // SLM's dimension
                                         const float slmPitch,                 // 1/slmDim
                                         const unsigned int numPixels,         // number of pixels in SLM
                                         // Spot information
                                         const float * const spotX,            // x coordinates of spots
                                         const float * const spotY,            // y coordinates of spots
                                         const float * const spotZ,            // z coordinates of spots
                                         float * const spotRe,                 // real component of spot states
                                         float * const spotIm)                 // imaginary component of spot states
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  //float x = getPixelCoords(tid, slmDim, slmPitch);
  //float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  int y = 0;
  int x = tid;
  float normalX = normalize(x, (int)slmDim);
  while (i < numPixels) {
    float normalY = normalize(y, (int)slmDim);
    pixelPhase = hologramPhase[i] - computePhase(normalX, normalY, spotx, spoty, spotz);

    if (x == 12 && y == 12)
    	printf("%f\n", pixelPhase);

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    y++;
    i += blockSize;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    spotRe[spotNumber] = vRe[0];
    spotIm[spotNumber] = vIm[0];
  }
}

// Obtain phases in SLM plane
__global__ void propagateToSLM(// Hologram information
                               unsigned char * const hologram,       // output hologram
                               float * const hologramPhase,          // current hologram phase
                               float * const prevHologramPhase,      // previous hologram phase, used for RPC
                               const unsigned int slmDim,            // SLM's dimension
                               const float slmPitch,                 // 1/slmDim
                               const unsigned int numPixels,         // number of pixels in SLM
                               const int iteration,                  // GSW iteration number
                               const bool lastIteration,             // true if last iteration of GSW
                               // Spot information
                               const float * const spotX,            // x coordinates of spots
                               const float * const spotY,            // y coordinates of spots
                               const float * const spotZ,            // z coordinates of spots
                               const float * const spotDesiredAmp,   // desired amplitudes of spots
                               const float * const spotRe,           // real component of spot states
                               const float * const spotIm,           // imaginary component of spot states
                               float * const spotWeight,             // spot weights
                               float * const spotI,                  // intermediate spot intensities (debug)
                               const bool saveSpotI,                 // if true, spot intensities are saved to global memory (debug)
                               const unsigned int numSpots,          // number of spots
                               // Correction information
                               const bool useAC,                     // use aberration correction
                               const float * const aberrationCoeffs, // correction coefficients
                               const bool useSVPR,                   // use spatially varying phase response
                               const int numPolCoeffs,               // number of polynomial coefficients
                               const float * const polCoeffs,        // polynomial coefficients
                               const bool useLUT,                    // use LUT for phase-to-uc conversion
                               const unsigned char * const lut,      // LUT for phase-to-uc conversion
                               const bool useRPC,                    // use restricted phase change
                               const float alpha)                    // RPC threshold
{
  __shared__ float spotAMean;
  __shared__ float spotP[MAX_SPOTS];
  __shared__ float spotA[MAX_SPOTS];
  __shared__ float spotW[MAX_SPOTS];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  float pixelRe = 0.0f;
  float pixelIm = 0.0f;
  float pixelPhase = 0.0f;

  if (idx < numPixels) {
    // Load spot phases, amplitudes, and weights
    // FIXME: would be faster on CPU?
    if (tid < numSpots) {
      float re = spotRe[tid];
      float im = spotIm[tid];
      spotP[tid] = atan2f(im, re);
      spotA[tid] = hypotf(re, im)/spotDesiredAmp[tid];
      if (iteration != 0) {
        spotW[tid] = spotWeight[tid + iteration*numSpots];
      } else {
        spotA[tid] = (spotA[tid] < 0.5f) ? 0.5f : spotA[tid];
        spotW[tid] = spotDesiredAmp[tid];
      }
    }
    __syncthreads();

    // Compute mean spot amplitude
    // FIXME: parallel reduction?
    if (tid == 0) {
      float sum = 0.0f;
      for (int i = 0; i < numSpots; i++) {
        sum += spotA[i];
      }
      spotAMean = sum/((float) numSpots);
    }
    __syncthreads();

    // Update spot weights
    if (tid < numSpots) {
      spotW[tid] = spotW[tid] * spotAMean / spotA[tid];

      // Copy weights to use as initial value in next run
      if (!lastIteration)
        spotWeight[tid + numSpots*(iteration + 1)] = spotW[tid];

      // May be excluded, used for monitoring only
      if (saveSpotI)
        spotI[tid + numSpots*iteration] = spotA[tid]*spotA[tid];
    }
    __syncthreads();

    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    // Compute the pixel's phase by summing contributions from all spots
    // TODO: use scratchpad for spot x, y, z
    for (int i = 0; i < numSpots; i++) {
      float delta = computePhase(x, y, spotX[i], spotY[i], spotZ[i]);
      pixelRe += spotW[i] * cosf(spotP[i] + delta);
      pixelIm += spotW[i] * sinf(spotP[i] + delta);
    }

    pixelPhase = atan2f(pixelIm, pixelRe);
    if (useRPC) {
      float prevPhase = prevHologramPhase[idx];
      if (fabs(pixelPhase - prevPhase) > alpha)
        pixelPhase = prevPhase;

      if (lastIteration)
        prevHologramPhase[idx] = pixelPhase;
    }

    // This is the last iteration, compute and write the final hologram phases to global memory
    if (lastIteration) {
      if (useAC)
        pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

      if (useSVPR) {
        __shared__ float coeff[MAX_POL];
        if (tid < numPolCoeffs)
          coeff[tid] = polCoeffs[tid];
        __syncthreads();

        hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
      } else if (useLUT) {
        __shared__ unsigned char lut_t[MAX_UCHAR];
        if (tid < MAX_UCHAR)
          lut_t[tid] = lut[tid];
        __syncthreads();

        hologram[idx] = lut_t[phase2int(pixelPhase)];
      } else {
        hologram[idx] = phase2uc(pixelPhase);
      }
    } else { // Otherwise, write intermediate phases to global memory
      hologramPhase[idx] = pixelPhase;
    }
  }
}

// Convert from unsigned char [0, 255] to float [-pi, pi]
__global__ void uc2f(float *f, const unsigned char * const uc, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    f[idx] = uc[idx] * 2.0f * M_PI/256.0f - M_PI;
  }
}

// Custom debug functions
inline void mSafeCall(hipError_t status, int line, const char *file)
{
#ifdef M_CUDA_DEBUG
  do {
    if (status != hipSuccess) {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess) {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  } while (0);
#endif
  return;
}

inline void mCheckError(int line, const char *file)
{
#ifdef M_CUDA_DEBUG
  do
  {
    hipError_t status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  }while(0);
#endif
  return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    float *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (float*)malloc(length * sizeof (float));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(float), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %f", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataUC(unsigned char *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    unsigned char *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (unsigned char*)malloc(length * sizeof (unsigned char));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(unsigned char), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %hhu", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    int *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (int*)malloc(length * sizeof (int));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(int), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %d", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

/******************************* Host functions *******************************/

// Timing
double getClock() {
  struct timeval tv;
  int ok;
  ok = gettimeofday(&tv, NULL);
  if (ok < 0) {
    printf("gettimeofday error");
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// Allocate GPU memory and parameters
int setup(const float * const initPhases,       // initial pixel phases
          const float * const aberrationCoeffs, // aberration correction matrix
          const float * const polCoeffs,        // SVPR polynomial coefficients
          const unsigned char * const lut)      // phase-to-uc conversion LUT
{
  // Make sure there's a GPU that we can use
  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != 0) {
    printf("No CUDA compatible GPU found\n");
    exit(1);
  } else {
    M_SAFE_CALL(hipSetDevice(0));
  }

  /*** Hologram ***/

  hologramMemSize = numPixels * sizeof(unsigned char);
  const unsigned int hologramPhaseMemSize = numPixels * sizeof(float);
  M_SAFE_CALL(hipMalloc((void **) &d_hologram, hologramMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_hologramPhase, hologramPhaseMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_prevHologramPhase, hologramPhaseMemSize));
  M_SAFE_CALL(hipMemcpy(d_hologramPhase, initPhases, hologramPhaseMemSize, hipMemcpyHostToDevice));
  M_SAFE_CALL(hipMemcpy(d_prevHologramPhase, initPhases, hologramPhaseMemSize, hipMemcpyHostToDevice));

  /*** Spots ***/

  const unsigned int spotMemSize = numSpots * sizeof(float);
  weightMemSize = numSpots * numIterations * sizeof(float);
  M_SAFE_CALL(hipMalloc((void **) &d_x, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_y, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_z, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_desiredAmp, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_spotRe, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_spotIm, spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_weights, weightMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_obtainedI, weightMemSize));

  /*** Corrections ***/

  // Aberration correction
  if (useAC) {
    hipMalloc((void **) &d_aberrationCoeffs, hologramPhaseMemSize);
    M_SAFE_CALL(hipMemcpy(d_aberrationCoeffs, aberrationCoeffs, hologramPhaseMemSize, hipMemcpyHostToDevice));
  }

  // SVPR
  if (useSVPR) {
    int numCoeff[5] = {20, 35, 56, 84, 120};
    if ((3 <= polOrder) && (polOrder <= 7)) {
      numPolCoeffs = numCoeff[polOrder - 3];
    } else {
      printf("Polynomial order out of range. Coerced to 3.\n");
      numPolCoeffs = numCoeff[0];
    }

    hipMalloc((void **) &d_polCoeffs, numPolCoeffs * sizeof(float));
    M_SAFE_CALL(hipMemcpy(d_polCoeffs, polCoeffs, numPolCoeffs * sizeof(float), hipMemcpyHostToDevice));
  }

  // LUT
  if (useLUT && !useSVPR) {
    hipMalloc((void **) &d_lut, MAX_UCHAR * sizeof(unsigned char));
    M_SAFE_CALL(hipMemcpy(d_lut, lut, MAX_UCHAR * sizeof(unsigned char), hipMemcpyHostToDevice));
  }

  // RPC
  if (useRPC && alpha < (2.0f * M_PI))
    useRPC = true;
  else
    useRPC = false;

  status = hipGetLastError();
  return status;
}

// Free GPU memory
int finish()
{
  /*** Hologram ***/

  M_SAFE_CALL(hipFree(d_hologram));
  M_SAFE_CALL(hipFree(d_hologramPhase));
  M_SAFE_CALL(hipFree(d_prevHologramPhase));

  /*** Spots ***/

  M_SAFE_CALL(hipFree(d_x));
  M_SAFE_CALL(hipFree(d_y));
  M_SAFE_CALL(hipFree(d_z));
  M_SAFE_CALL(hipFree(d_desiredAmp));
  M_SAFE_CALL(hipFree(d_spotRe));
  M_SAFE_CALL(hipFree(d_spotIm));
  M_SAFE_CALL(hipFree(d_weights));
  M_SAFE_CALL(hipFree(d_obtainedI));

  if (useAC) {
    M_SAFE_CALL(hipFree(d_aberrationCoeffs));
  }

  if (useSVPR) {
    M_SAFE_CALL(hipFree(d_polCoeffs));
  }

  if (useLUT) {
    M_SAFE_CALL(hipFree(d_lut));
  }

  hipDeviceReset();
  status = hipGetLastError();
  return status;
}

// Generate a hologram
int generateHologram(unsigned char * const hologram, // hologram to send to SLM
                     const float * const spotX,      // x coordinates of spots/traps
                     const float * const spotY,      // y coordinates of spots/traps
                     const float * const spotZ,      // z coordinates of spots/traps
                     const float * const spotI,      // relative intensities of spots/traps
                     const int numSpots,             // number of spots/traps
                     const int numIterations,        // number of iterations to run GSW
                     float * const interAmps,        // intermediate amplitudes (debug)
                     int method)                     // method to use for generating hologram
{
  if (numSpots < 1)
    method = 100;
  else if (numSpots < 3)
    method = 0;

  computeAndCopySpotData(spotX, spotY, spotZ, spotI, numSpots);
  double t;
  int numBlocks;

  switch (method) {
    case 0:
      // Generate hologram using "Lenses and Prisms"
      printf("Starting Lenses and Prisms...\n");
      t = getClock();

      numBlocks = (numPixels/BLOCK_SIZE + (numPixels % BLOCK_SIZE ? 1 : 0));
      lensesAndPrisms<<<numBlocks, BLOCK_SIZE>>>(d_hologram,
                                                 slmDim,
                                                 slmPitch,
                                                 numPixels,
                                                 d_x,
                                                 d_y,
                                                 d_z,
                                                 d_desiredAmp,
                                                 numSpots,
                                                 useAC,
                                                 d_aberrationCoeffs,
                                                 useSVPR,
                                                 numPolCoeffs,
                                                 d_polCoeffs,
                                                 useLUT,
                                                 d_lut);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_CHECK_ERROR();

      if (saveSpotI) {
        calculateI<<<numSpots, SLM_SIZE>>>(d_hologram,
                                           slmDim,
                                           slmPitch,
                                           numPixels,
                                           d_x,
                                           d_y,
                                           d_z,
                                           d_obtainedI);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
        M_SAFE_CALL(hipMemcpy(interAmps, d_obtainedI, numSpots*sizeof(float), hipMemcpyDeviceToHost));
      }
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, hologramMemSize, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      break;
    case 1:
      // Generate holgram using fresnel propagation
      printf("Starting Fresnel...\n");
      t = getClock();
      numBlocks = (numPixels/BLOCK_SIZE + (numPixels % BLOCK_SIZE ? 1 : 0));

      // Uncomment this to start with pre-calculated hologram
      //cudaMemcpy(d_hologram, hologram, hologramMemSize, cudaMemcpyHostToDevice);
      //cudaDeviceSynchronize();
      //uc2f<<<numBlocks, BLOCK_SIZE >>>(d_hologramPhase, d_hologram, numPixels);

      for (int l = 0; l < numIterations; l++) {
        printf("Iteration %d\n", l);
        propagateToSpotPositions<<<numSpots, SLM_SIZE>>>(d_hologramPhase,
                                                         slmDim,
                                                         slmPitch,
                                                         numPixels,
                                                         d_x,
                                                         d_y,
                                                         d_z,
                                                         d_spotRe,
                                                         d_spotIm);
        M_CHECK_ERROR();
        hipDeviceSynchronize();

        propagateToSLM<<<numBlocks, BLOCK_SIZE>>>(d_hologram,
                                                  d_hologramPhase,
                                                  d_prevHologramPhase,
                                                  slmDim,
                                                  slmPitch,
                                                  numPixels,
                                                  l,
                                                  (l == (numIterations - 1)),
                                                  d_x,
                                                  d_y,
                                                  d_z,
                                                  d_desiredAmp,
                                                  d_spotRe,
                                                  d_spotIm,
                                                  d_weights,
                                                  d_obtainedI,
                                                  saveSpotI,
                                                  numSpots,
                                                  useAC,
                                                  d_aberrationCoeffs,
                                                  useSVPR,
                                                  numPolCoeffs,
                                                  d_polCoeffs,
                                                  useLUT,
                                                  d_lut,
                                                  useRPC,
                                                  alpha);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
/*                // debuggin output
        float* peekSpot = (float*)malloc(32 * sizeof(float));
        cudaMemcpy(peekSpot, d_hologram, 32 * sizeof(float), cudaMemcpyDeviceToHost);
        for (int i = 0; i < 32; ++i){
          printf("%f ", peekSpot[i]);
        }
        printf("\n");
        M_CHECK_ERROR();
        cudaDeviceSynchronize();*/
      }

      if (saveSpotI)
        M_SAFE_CALL(hipMemcpy(interAmps, d_obtainedI, weightMemSize, hipMemcpyDeviceToHost));
      else
        M_SAFE_CALL(hipMemcpy(interAmps, d_weights, weightMemSize, hipMemcpyDeviceToHost));
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, hologramMemSize, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      printf("Time/iteration = %12.8lf seconds\n", t/((double) numIterations));
      break;
    case 100:
      // Apply corrections to pre-calculated hologram
      numBlocks = (numPixels/BLOCK_SIZE + (numPixels % BLOCK_SIZE ? 1 : 0));
      M_SAFE_CALL(hipMemcpy(d_hologram, hologram, hologramMemSize, hipMemcpyHostToDevice));
      applyCorrections<<<numBlocks, BLOCK_SIZE>>>(d_hologram,
                                                  slmDim,
                                                  slmPitch,
                                                  useAC,
                                                  d_aberrationCoeffs,
                                                  useSVPR,
                                                  numPolCoeffs,
                                                  d_polCoeffs,
                                                  useLUT,
                                                  d_lut);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, hologramMemSize, hipMemcpyDeviceToHost));
      break;
    default:
      break;
  }

  // Handle CUDA errors
  status = hipGetLastError();
  return status;
}

// Calculate amplitude and phase at positions (x, y, z) from a given hologram
int getAmpAndPhase(const float * spotX,                  // x coordinates of spots/traps
                   const float * spotY,                  // y coordinates of spots/traps
                   const float * spotZ,                  // z coordinates of spots/traps
                   const int numSpots,                   // number of spots/traps
                   const unsigned char * const hologram, // hologram to use
                   float *amp,                           // amplitude at (x, y, z)
                   float *phase)                         // phase at (x, y, z)
{
  float *d_amp;
  float *d_phase;
  hipMalloc((void **) &d_amp, numSpots * sizeof(float));
  hipMalloc((void **) &d_phase, numSpots * sizeof(float));
  hipMemcpy(d_hologram, hologram, hologramMemSize, hipMemcpyHostToDevice);

  int offset = 0;
  int numSpotsRem = numSpots;
  int numSpotsThis;

  while (numSpotsRem > 0) {
    numSpotsThis = (numSpotsRem > MAX_SPOTS) ? MAX_SPOTS : numSpotsRem;
    hipMemcpy(d_x, spotX + offset, numSpotsThis * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, spotY + offset, numSpotsThis * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, spotZ + offset, numSpotsThis * sizeof(float), hipMemcpyHostToDevice);
    calculateIAndPhase<<<numSpotsThis, SLM_SIZE>>>(d_hologram,
                                                   slmDim,
                                                   slmPitch,
                                                   numPixels,
                                                   d_x,
                                                   d_y,
                                                   d_z,
                                                   d_amp + offset,
                                                   d_phase + offset);
    hipDeviceSynchronize();

    numSpotsRem -= MAX_SPOTS;
    offset += numSpotsThis;
  }

  hipMemcpy(amp, d_amp, numSpots * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(phase, d_phase, numSpots * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_amp);
  hipFree(d_phase);

  status = hipGetLastError();
  return status;
}

void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n)
{
  // An alternate way is to use sum instead of 100 in the formula below, but
  // I'm not sure what the difference is
  /*
  float sum = 0.0f;
  for (int i = 0; i < n; i++)
    sum += intensity[i];
  */

  const float slmDimf = (float) slmDim;
  float *desiredAmp = (float *) malloc(n * sizeof(float));

  for (int i = 0; i < n; i++) {
    float sincxRec = (x[i] == 0) ? 1.0f : ((M_PI * x[i]/slmDimf) / sinf(M_PI * x[i]/slmDimf));
    float sincyRec = (y[i] == 0) ? 1.0f : ((M_PI * y[i]/slmDimf) / sinf(M_PI * y[i]/slmDimf));
    desiredAmp[i] = (intensity[i] <= 0.0f) ? 1.0f : (sincxRec * sincyRec * sqrtf(intensity[i]/100) * slmDimf * slmDimf);
  }

  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_desiredAmp, desiredAmp, n * sizeof(float), hipMemcpyHostToDevice);
  free(desiredAmp);
}

int main(int argc, char *argv[])
{
  srand(1);
  const int method = atoi(argv[1]); // 0 => Direct, 1 => Fresnel, 100 => Corrections

  slmDim = SLM_SIZE;
  slmPitch = 1.0f / ((float) slmDim);
  numPixels = slmDim * slmDim;
  numIterations = 1; // 10 iterations for convergence
  numSpots = 4;

  // Spots/traps. These form a quadrant across four planes.
  const float x[] = {-128.0f, -128.0f, 127.0f, 127.0f};
  const float y[] = {127.0f, -128.0f, 127.0f, -128.0f};
  const float z[] = {1.0f, 2.0f, 3.0f, 4.0f};
  const float I[] = {0.12f, 0.34f, 0.56f, 0.78f};

  // Correction parameters
  saveSpotI = true;
  useAC = false;
  const float * const aberrationCoeffs = NULL;

  useSVPR = false;
  polOrder = 5;
  float * const polCoeffs = (float *) malloc(MAX_POL * sizeof(float));
  for (int i = 0; i < MAX_POL; i++) {
    polCoeffs[i] = 0.0f;
  }

  useLUT = false;
  const unsigned char * const lut = NULL;

  useRPC = false;
  alpha = 2.0f * M_PI * 0.123f;

  unsigned char *hologram = (unsigned char *) malloc(numPixels * sizeof(unsigned char));
  float *amps = (float *) malloc(numSpots * numIterations * sizeof(float));
  float * const initPhases = (float *) malloc(numPixels * sizeof(float)); // [-pi, pi]
  for (int i = 0; i < numPixels; i++) {
    hologram[i] = random() % 256;
    initPhases[i] = (2.0 * M_PI * (random() / ((float) RAND_MAX))) - M_PI;
  }

  if (setup(initPhases, aberrationCoeffs, polCoeffs, lut) != 0) {
    printf("Init failed.\n");
    exit(1);
  }

  // Save initial hologram
  FILE *ifile = fopen("my_init_hologram.dat", "w");
  for (int i = 0; i < numPixels; i++) {
    fprintf(ifile, "%hhu\n", hologram[i]);
  }

  double t = getClock();

  if (generateHologram(hologram, x, y, z, I, numSpots, numIterations, amps, method) != 0) {
    printf("Computation failed.\n");
    exit(1);
  }

  t = getClock() - t;

  if (finish() != 0) {
    printf("Cleanup failed.\n");
    exit(1);
  }

  printf("Total time = %12.8lf seconds\n", t);

  // Save hologram
  FILE *hfile = fopen("my_output_hologram.dat", "w");
  for (int i = 0; i < numPixels; i++) {
    fprintf(hfile, "%hhu\n", hologram[i]);
  }

  // Save amplitudes
  FILE *afile = fopen("my_amps.dat", "w");
  for (int i = 0; i < numSpots * numIterations; i++) {
    fprintf(afile, "%f\n", amps[i]);
  }

  fclose(ifile);
  fclose(hfile);
  fclose(afile);

  free(polCoeffs);
  free(hologram);
  free(initPhases);
  free(amps);

  return 0;
}

