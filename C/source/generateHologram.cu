#include "hip/hip_runtime.h"
/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

// Activates a number of custom debug macros
#define M_CUDA_DEBUG

// Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

// Number of spots/traps/depth planes - even 64 is pushing it
#define MAX_SPOTS 64

// FIXME: This shouldn't be hardcoded
#define SLM_SIZE 512

// Use bitwise modulo operations if the SLM size is a power of 2
#if (((SLM_SIZE - 1) & (SLM_SIZE)) == 0)
#define SLMPOW2
#endif

#define BLOCK_SIZE 256
#define MAX_POL 120
#define MAX_UCHAR 256

// Forward declaration
void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n);

// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, char *file);
inline void mCheckError(int line, char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataUC(unsigned char *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

// Globals
// Spot coordinates and intensity
float *d_x, *d_y, *d_z, *d_I;

// s
float *d_pSLM_f;                //the optimized pSpot pattern, float [-pi, pi]
float *d_weights, *d_Iobtained, *d_desiredAmp;    //used h_weights and calculated amplitudes for each spot and each iteration
float *d_pSLMstart_f;             //Initial pSpot pattern [-pi, pi]
float *d_spotRe_f, *d_spotIm_f;
float SLMsizef = (float) SLM_SIZE;
int n_blocks_Phi;
int memsize_SLM_f;
int memsize_SLMuc;
int memsize_spotsf;
int data_w;
int N_pixels;
float SLMpitch_f;
float h_desiredAmp[MAX_SPOTS];
float *d_obtainedPhase;

// The optimized pSpot pattern, unsigned char, the one sent to the SLM [0, 255]
unsigned char *d_hologram;

// Aberration correction
bool useAC = false;
float *d_aberrationCoeffs = NULL;

// Spatially Varying Phase Response
bool useSVPR = false;
int polOrder = 0;
int numPolCoeffs = 0;
float *d_polCoeffs = NULL;

// phase-to-uc LUT
bool useLUT = false;
unsigned char *d_lut = NULL;

// Restricted Phase Change
bool useRPC = false;
float alpha = 10;

// Save spot intensities
bool saveSpotI = false;

// Error checking
char CUDAmessage[100];
hipError_t status;

// Constant memory declarations
__device__ __constant__ int c_log2data_w[1];
__device__ __constant__ float c_x[MAX_SPOTS];
__device__ __constant__ float c_y[MAX_SPOTS];
__device__ __constant__ float c_z[MAX_SPOTS];
__device__ __constant__ float c_desiredAmp[MAX_SPOTS];

/****************************** Device functions ******************************/

// Convert from unsigned char [0, 255] to phase (float) [-pi, pi]
__device__ inline float uc2phase(const unsigned char uc)
{
  return (((float) uc) * 2.0f * M_PI/256.0f - M_PI);
}

// Convert from phase (float) [-pi, pi] to unsigned char [0, 255]
__device__ inline unsigned char phase2uc(const float phase)
{
  return (unsigned char) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Convert from phase (float) [-pi, pi] to int
__device__ inline int phase2int(const float phase)
{
  return (int) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Apply wavefront distortion/aberration correction
__device__ inline float applyAberrationCorrection(float pSpot, const float correction)
{
  pSpot = pSpot - correction; // Apply correction
  return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI))); // Apply mod([-pi, pi], pSpot)
}

// Get x coordinate from global thread ID
__device__ inline int getXIdx(const int index, const int slmDim)
{
#ifdef SLMPOW2
  int idx = index & (slmDim - 1);
#else
  int idx = index % slmDim;
#endif
  return idx;
}

// Get y coordinate from global thread ID and x coordinate
__device__ inline int getYIdx(const int index, const int xIdx, const int slmDim, const float slmPitch)
{
#ifdef SLMPOW2
  int idx = (index - xIdx) >> c_log2data_w[0]; // FIXME
#else
  int idx = floor(((float) (index - xIdx)) * slmPitch);
#endif
  return idx;
}

// Get pixel coordinates in [-0.5, 0.5]
__device__ inline float getPixelCoords(const int index, const int slmDim, const float slmPitch)
{
  return (((float) (index - (slmDim << 1))) * slmPitch);
}

// Compute phase from pixel position and spot position
__device__ inline float computePhase(const float x,
                                     const float y,
                                     const float spotx,
                                     const float spoty,
                                     const float spotz)
{
  return (M_PI * (spotz * (x*x + y*y) + 2.0f * (x*spotx + y*spoty)));
}

// Apply SVPR
__device__ unsigned char applySVPR(const float phase,
                                   const float x,
                                   const float y,
                                   const float * const coeff,
                                   const int numPolCoeffs)
{
  float result = 0.0f;
  switch (numPolCoeffs) {
    case 120:
      result += coeff[84]*x*x*x*x*x*x*x;
      result += coeff[85]*x*x*x*x*x*x*y;
      result += coeff[86]*x*x*x*x*x*x*phase;
      result += coeff[87]*x*x*x*x*x*y*y;
      result += coeff[88]*x*x*x*x*x*y*phase;
      result += coeff[89]*x*x*x*x*x*phase*phase;
      result += coeff[90]*x*x*x*x*y*y*y;
      result += coeff[91]*x*x*x*x*y*y*phase;
      result += coeff[92]*x*x*x*x*y*phase*phase;
      result += coeff[93]*x*x*x*x*phase*phase*phase;
      result += coeff[94]*x*x*x*y*y*y*y;
      result += coeff[95]*x*x*x*y*y*y*phase;
      result += coeff[96]*x*x*x*y*y*phase*phase;
      result += coeff[97]*x*x*x*y*phase*phase*phase;
      result += coeff[98]*x*x*x*phase*phase*phase*phase;
      result += coeff[99]*x*x*y*y*y*y*y;
      result += coeff[100]*x*x*y*y*y*y*phase;
      result += coeff[101]*x*x*y*y*y*phase*phase;
      result += coeff[102]*x*x*y*y*phase*phase*phase;
      result += coeff[103]*x*x*y*phase*phase*phase*phase;
      result += coeff[104]*x*x*phase*phase*phase*phase*phase;
      result += coeff[105]*x*y*y*y*y*y*y;
      result += coeff[106]*x*y*y*y*y*y*phase;
      result += coeff[107]*x*y*y*y*y*phase*phase;
      result += coeff[108]*x*y*y*y*phase*phase*phase;
      result += coeff[109]*x*y*y*phase*phase*phase*phase;
      result += coeff[110]*x*y*phase*phase*phase*phase*phase;
      result += coeff[111]*x*phase*phase*phase*phase*phase*phase;
      result += coeff[112]*y*y*y*y*y*y*y;
      result += coeff[113]*y*y*y*y*y*y*phase;
      result += coeff[114]*y*y*y*y*y*phase*phase;
      result += coeff[115]*y*y*y*y*phase*phase*phase;
      result += coeff[116]*y*y*y*phase*phase*phase*phase;
      result += coeff[117]*y*y*phase*phase*phase*phase*phase;
      result += coeff[118]*y*phase*phase*phase*phase*phase*phase;
      result += coeff[119]*phase*phase*phase*phase*phase*phase*phase;
    case 84:
      result += coeff[56]*x*x*x*x*x*x;
      result += coeff[57]*x*x*x*x*x*y;
      result += coeff[58]*x*x*x*x*x*phase;
      result += coeff[59]*x*x*x*x*y*y;
      result += coeff[60]*x*x*x*x*y*phase;
      result += coeff[61]*x*x*x*x*phase*phase;
      result += coeff[62]*x*x*x*y*y*y;
      result += coeff[63]*x*x*x*y*y*phase;
      result += coeff[64]*x*x*x*y*phase*phase;
      result += coeff[65]*x*x*x*phase*phase*phase;
      result += coeff[66]*x*x*y*y*y*y;
      result += coeff[67]*x*x*y*y*y*phase;
      result += coeff[68]*x*x*y*y*phase*phase;
      result += coeff[69]*x*x*y*phase*phase*phase;
      result += coeff[70]*x*x*phase*phase*phase*phase;
      result += coeff[71]*x*y*y*y*y*y;
      result += coeff[72]*x*y*y*y*y*phase;
      result += coeff[73]*x*y*y*y*phase*phase;
      result += coeff[74]*x*y*y*phase*phase*phase;
      result += coeff[75]*x*y*phase*phase*phase*phase;
      result += coeff[76]*x*phase*phase*phase*phase*phase;
      result += coeff[77]*y*y*y*y*y*y;
      result += coeff[78]*y*y*y*y*y*phase;
      result += coeff[79]*y*y*y*y*phase*phase;
      result += coeff[80]*y*y*y*phase*phase*phase;
      result += coeff[81]*y*y*phase*phase*phase*phase;
      result += coeff[82]*y*phase*phase*phase*phase*phase;
      result += coeff[83]*phase*phase*phase*phase*phase*phase;
    case 56:
      result += coeff[35]*x*x*x*x*x;
      result += coeff[36]*x*x*x*x*y;
      result += coeff[37]*x*x*x*x*phase;
      result += coeff[38]*x*x*x*y*y;
      result += coeff[39]*x*x*x*y*phase;
      result += coeff[40]*x*x*x*phase*phase;
      result += coeff[41]*x*x*y*y*y;
      result += coeff[42]*x*x*y*y*phase;
      result += coeff[43]*x*x*y*phase*phase;
      result += coeff[44]*x*x*phase*phase*phase;
      result += coeff[45]*x*y*y*y*y;
      result += coeff[46]*x*y*y*y*phase;
      result += coeff[47]*x*y*y*phase*phase;
      result += coeff[48]*x*y*phase*phase*phase;
      result += coeff[49]*x*phase*phase*phase*phase;
      result += coeff[50]*y*y*y*y*y;
      result += coeff[51]*y*y*y*y*phase;
      result += coeff[52]*y*y*y*phase*phase;
      result += coeff[53]*y*y*phase*phase*phase;
      result += coeff[54]*y*phase*phase*phase*phase;
      result += coeff[55]*phase*phase*phase*phase*phase;
    case 35:
      result += coeff[20]*x*x*x*x;
      result += coeff[21]*x*x*x*y;
      result += coeff[22]*x*x*x*phase;
      result += coeff[23]*x*x*y*y;
      result += coeff[24]*x*x*y*phase;
      result += coeff[25]*x*x*phase*phase;
      result += coeff[26]*x*y*y*y;
      result += coeff[27]*x*y*y*phase;
      result += coeff[28]*x*y*phase*phase;
      result += coeff[29]*x*phase*phase*phase;
      result += coeff[30]*y*y*y*y;
      result += coeff[31]*y*y*y*phase;
      result += coeff[32]*y*y*phase*phase;
      result += coeff[33]*y*phase*phase*phase;
      result += coeff[34]*phase*phase*phase*phase;
    case 20:
      result += coeff[0];
      result += coeff[1]*x;
      result += coeff[2]*y;
      result += coeff[3]*phase;
      result += coeff[4]*x*x;
      result += coeff[5]*x*y;
      result += coeff[6]*x*phase;
      result += coeff[7]*y*y;
      result += coeff[8]*y*phase;
      result += coeff[9]*phase*phase;
      result += coeff[10]*x*x*x;
      result += coeff[11]*x*x*y;
      result += coeff[12]*x*x*phase;
      result += coeff[13]*x*y*y;
      result += coeff[14]*x*y*phase;
      result += coeff[15]*x*phase*phase;
      result += coeff[16]*y*y*y;
      result += coeff[17]*y*y*phase;
      result += coeff[18]*y*phase*phase;
      result += coeff[19]*phase*phase*phase;
      break;
    default:
      result = 0.0f;
      break;
  }

  if (result < 0.0f)
    result = 0.0f;

  return (unsigned char) result;
}

// Performs intra-warp reduction. The arrays are in shared memory.
__device__ void warpReduce(volatile float *vRe, volatile float *vIm, int tid)
{
  vRe[tid] += vRe[tid + 32];
  vIm[tid] += vIm[tid + 32];

  vRe[tid] += vRe[tid + 16];
  vIm[tid] += vIm[tid + 16];

  vRe[tid] += vRe[tid + 8];
  vIm[tid] += vIm[tid + 8];

  vRe[tid] += vRe[tid + 4];
  vIm[tid] += vIm[tid + 4];

  vRe[tid] += vRe[tid + 2];
  vIm[tid] += vIm[tid + 2];

  vRe[tid] += vRe[tid + 1];
  vIm[tid] += vIm[tid + 1];
}

// Apply corrections to precalculated hologram
__global__ void applyCorrections(// Hologram information
                                 unsigned char * const hologram,       // hologram to use
                                 const unsigned int slmDim,            // SLM's dimension
                                 const float slmPitch,                 // 1/slmDim
                                 // Correction information
                                 const bool useAC,                     // use aberration correction
                                 const float * const aberrationCoeffs, // correction coefficients
                                 const bool useSVPR,                   // use spatially varying phase response
                                 const int numPolCoeffs,               // number of polynomial coefficients
                                 const float * const polCoeffs,        // polynomial coefficients
                                 const bool useLUT,                    // use LUT for phase-to-uc conversion
                                 const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  const int tid = threadIdx.x;
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float pixelPhase = uc2phase(hologram[idx]);
  if (useAC)
    pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

  if (useSVPR) {
    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmDim, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    __shared__ float coeff[MAX_POL];
    if (tid < numPolCoeffs)
      coeff[tid] = polCoeffs[tid];
    __syncthreads();

    hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
  } else if (useLUT) {
    __shared__ unsigned char lut_t[MAX_UCHAR];
    if (tid < MAX_UCHAR)
      lut_t[tid] = lut[tid];
    __syncthreads();

    hologram[idx] = lut_t[phase2int(pixelPhase)];
  } else {
    hologram[idx] = phase2uc(pixelPhase);
  }
}

// Calculate hologram using "Lenses and Prisms"
__global__ void lensesAndPrisms(// Hologram information
                                unsigned char * const hologram,       // hologram to use
                                const unsigned int slmDim,            // SLM's dimension
                                const float slmPitch,                 // 1/slmDim
                                const unsigned int numPixels,         // number of pixels in SLM
                                // Spot information
                                const float * const spotX,            // x coordinates of spots
                                const float * const spotY,            // y coordinates of spots
                                const float * const spotZ,            // z coordinates of spots
                                const float * const spotDesiredAmp,   // desired amplitudes of spots
                                const unsigned int numSpots,          // number of spots
                                // Correction information
                                const bool useAC,                     // use aberration correction
                                const float * const aberrationCoeffs, // correction coefficients
                                const bool useSVPR,                   // use spatially varying phase response
                                const int numPolCoeffs,               // number of polynomial coefficients
                                const float * const polCoeffs,        // polynomial coefficients
                                const bool useLUT,                    // use LUT for phase-to-uc conversion
                                const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  __shared__ float spotx[MAX_SPOTS];
  __shared__ float spoty[MAX_SPOTS];
  __shared__ float spotz[MAX_SPOTS];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  if (idx < numPixels) {
    if (tid < numSpots) {
      spotx[tid] = spotX[tid];
      spoty[tid] = spotY[tid];
      spotz[tid] = spotZ[tid];
    };
    __syncthreads();

    float pixelPhase;
    float vRe = 0.0f;
    float vIm = 0.0f;

    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmDim, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    // Compute the pixel's phase by summing contributions from all spots
    for (int i = 0; i < numSpots; i++) {
      // TODO: Add variable phases to function call
      pixelPhase = computePhase(x, y, spotx[i], spoty[i], spotz[i]);
      vRe += spotDesiredAmp[i] * cosf(pixelPhase);
      vIm += spotDesiredAmp[i] * sinf(pixelPhase);
    }

    pixelPhase = atan2f(vIm, vRe); // [-pi, pi]
    if (useAC)
      pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

    if (useSVPR) {
      __shared__ float coeff[MAX_POL];
      if (tid < numPolCoeffs)
        coeff[tid] = polCoeffs[tid];
      __syncthreads();

      hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
    } else if (useLUT) {
      __shared__ unsigned char lut_t[MAX_UCHAR];
      if (tid < MAX_UCHAR)
        lut_t[tid] = lut[tid];
      __syncthreads();

      hologram[idx] = lut_t[phase2int(pixelPhase)];
    } else {
      hologram[idx] = phase2uc(pixelPhase);
    }
  }
}

__global__ void calculateI(// Hologram information
                           const unsigned char * const hologram, // hologram to use
                           const unsigned int slmDim,            // SLM's dimension
                           const float slmPitch,                 // 1/slmDim
                           const unsigned int numPixels,         // number of pixels in SLM
                           // Spot information
                           const float * const spotX,            // x coordinates of spots
                           const float * const spotY,            // y coordinates of spots
                           const float * const spotZ,            // z coordinates of spots
                           float * const spotI)                  // spot intensities
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  while (i < numPixels) {
    pixelPhase = uc2phase(hologram[i]) - computePhase(x, y, spotx, spoty, spotz);

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0] / ((float) numPixels);
    float im = vIm[0] / ((float) numPixels);
    spotI[spotNumber] = re*re + im*im;
  }
}

__global__ void calculateIAndPhase(// Hologram information
                                   const unsigned char * const hologram, // hologram to use
                                   const unsigned int slmDim,            // SLM's dimension
                                   const float slmPitch,                 // 1/slmDim
                                   const unsigned int numPixels,         // number of pixels in SLM
                                   // Spot information
                                   const float * const spotX,            // x coordinates of spots
                                   const float * const spotY,            // y coordinates of spots
                                   const float * const spotZ,            // z coordinates of spots
                                   float * const spotI,                  // spot intensities
                                   float * const spotP)                  // spot phases
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  while (i < numPixels) {
    pixelPhase = uc2phase(hologram[i]) - computePhase(x, y, spotx, spoty, spotz);
    pixelPhase += 2.0f * M_PI * spotZ[spotNumber];

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0] / ((float) numPixels);
    float im = vIm[0] / ((float) numPixels);
    spotI[spotNumber] = re*re + im*im;
    spotP[spotNumber] = atan2f(im, re);
  }
}

// Propagate from the SLM to the spot positions using Fresnel summation
// FIXME: Works only for blocksize = SLMsize
__global__ void propagateToSpotPositions(// Hologram information
                                         const float * const hologramPhase,    // hologram's phase
                                         const unsigned int slmDim,            // SLM's dimension
                                         const float slmPitch,                 // 1/slmDim
                                         const unsigned int numPixels,         // number of pixels in SLM
                                         // Spot information
                                         const float * const spotX,            // x coordinates of spots
                                         const float * const spotY,            // y coordinates of spots
                                         const float * const spotZ,            // z coordinates of spots
                                         float * const spotRe,                 // real component of spot states
                                         float * const spotIm)                 // imaginary component of spot states
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  const float spotx = spotX[spotNumber];
  const float spoty = spotY[spotNumber];
  const float spotz = spotZ[spotNumber];

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float pixelPhase;
  int i = tid;
  while (i < numPixels) {
    pixelPhase = hologramPhase[i] - computePhase(x, y, spotx, spoty, spotz);

    vRe[tid] += cosf(pixelPhase);
    vIm[tid] += sinf(pixelPhase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    spotRe[spotNumber] = vRe[0];
    spotIm[spotNumber] = vIm[0];
  }
}

// Obtain phases in SLM plane
__global__ void propagateToSLM(// Hologram information
                               unsigned char * const hologram,       // output hologram
                               float * const hologramPhase,          // current hologram phase
                               float * const prevHologramPhase,      // previous hologram phase, used for RPC
                               const unsigned int slmDim,            // SLM's dimension
                               const float slmPitch,                 // 1/slmDim
                               const unsigned int numPixels,         // number of pixels in SLM
                               const int iteration,                  // GSW iteration number
                               const bool lastIteration,             // true if last iteration of GSW
                               // Spot information
                               const float * const spotX,            // x coordinates of spots
                               const float * const spotY,            // y coordinates of spots
                               const float * const spotZ,            // z coordinates of spots
                               const float * const spotDesiredAmp,   // desired amplitudes of spots
                               const float * const spotRe,           // real component of spot states
                               const float * const spotIm,           // imaginary component of spot states
                               float * const spotWeight,             // spot weights
                               float * const spotI,                  // spot intensities (debug)
                               const bool saveSpotI,                 // if true, spot intensities are saved to global memory (debug)
                               const unsigned int numSpots,          // number of spots
                               // Correction information
                               const bool useAC,                     // use aberration correction
                               const float * const aberrationCoeffs, // correction coefficients
                               const bool useSVPR,                   // use spatially varying phase response
                               const int numPolCoeffs,               // number of polynomial coefficients
                               const float * const polCoeffs,        // polynomial coefficients
                               const bool useLUT,                    // use LUT for phase-to-uc conversion
                               const unsigned char * const lut,      // LUT for phase-to-uc conversion
                               const bool useRPC,                    // use restricted phase change
                               const float alpha)                    // RPC threshold
{
  __shared__ float spotAMean;
  __shared__ float spotP[MAX_SPOTS];
  __shared__ float spotA[MAX_SPOTS];
  __shared__ float spotW[MAX_SPOTS];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  float pixelRe = 0.0f;
  float pixelIm = 0.0f;
  float pixelPhase = 0.0f;

  if (idx < numPixels) {
    // Load spot phases, amplitudes, and weights
    // FIXME: would be faster on CPU?
    if (tid < numSpots) {
      float re = spotRe[tid];
      float im = spotIm[tid];
      spotP[tid] = atan2f(im, re);
      if (iteration != 0) {
        spotA[tid] = hypotf(re, im)/spotDesiredAmp[tid];
        spotW[tid] = spotWeight[tid + iteration*numSpots];
      } else {
        spotA[tid] = (spotA[tid] < 0.5f) ? 0.5f : spotA[tid];
        spotW[tid] = spotDesiredAmp[tid];
      }
    }
    __syncthreads();

    // Compute mean spot amplitude
    // FIXME: parallel reduction?
    if (tid == 0) {
      float sum = 0.0f;
      for (int i = 0; i < numSpots; i++) {
        sum += spotA[i];
      }
      spotAMean = sum/((float) numSpots);
    }
    __syncthreads();

    // Update spot weights
    if (tid < numSpots) {
      spotW[tid] = spotW[tid] * spotAMean / spotA[tid];

      // Copy weights to use as initial value in next run
      if (!lastIteration)
        spotWeight[tid + numSpots*(iteration + 1)] = spotW[tid];

      // May be excluded, used for monitoring only
      if (saveSpotI)
        spotI[tid + numSpots*iteration] = spotA[tid]*spotA[tid];
    }
    __syncthreads();

    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmDim, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    // Compute the pixel's phase by summing contributions from all spots
    // TODO: use scratchpad for spot x, y, z
    for (int i = 0; i < numSpots; i++) {
      float delta = computePhase(x, y, spotX[i], spotY[i], spotZ[i]);
      pixelRe += spotW[i] * cosf(spotP[i] + delta);
      pixelIm += spotW[i] * sinf(spotP[i] + delta);
    }

    pixelPhase = atan2f(pixelIm, pixelRe);
    if (useRPC) {
      float prevPhase = prevHologramPhase[idx];
      if (fabs(pixelPhase - prevPhase) > alpha)
        pixelPhase = prevPhase;

      if (lastIteration)
        prevHologramPhase[idx] = pixelPhase;
    }

    // This is the last iteration, compute and write the final hologram phases to global memory
    if (lastIteration) {
      if (useAC)
        pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

      if (useSVPR) {
        __shared__ float coeff[MAX_POL];
        if (tid < numPolCoeffs)
          coeff[tid] = polCoeffs[tid];
        __syncthreads();

        hologram[idx] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
      } else if (useLUT) {
        __shared__ unsigned char lut_t[MAX_UCHAR];
        if (tid < MAX_UCHAR)
          lut_t[tid] = lut[tid];
        __syncthreads();

        hologram[idx] = lut_t[phase2int(pixelPhase)];
      } else {
        hologram[idx] = phase2uc(pixelPhase);
      }
    } else { // Otherwise, write intermediate phases to global memory
      hologramPhase[idx] = pixelPhase;
    }
  }
}

// Convert from unsigned char [0, 255] to float [-pi, pi]
__global__ void uc2f(float *f, const unsigned char * const uc, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    f[idx] = uc[idx] * 2.0f * M_PI/256.0f - M_PI;
  }
}

// Custom debug functions
inline void mSafeCall(hipError_t status, int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do {
    if (status != hipSuccess) {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess) {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  } while (0);
#endif
  return;
}

inline void mCheckError(int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do
  {
    hipError_t status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  }while(0);
#endif
  return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    float *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (float*)malloc(length * sizeof (float));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(float), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %f", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataUC(unsigned char *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    unsigned char *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (unsigned char*)malloc(length * sizeof (unsigned char));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(unsigned char), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %hhu", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    int *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (int*)malloc(length * sizeof (int));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(int), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %d", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

// compute amps for constant total int
/*void computeAmps(float *h_I, float *h_desiredAmp, float *x, float *y, int N_spots, float e_desired)
{
  float SLMsize = (float)SLM_SIZE;
  float Isum = 0.0f;
  for (int i = 0; i<N_spots; i++)
    Isum += h_I[i];
  for (int j = 0; j<N_spots; j++)
  {
    float sincx_rec = 1.0f;//= (x==0)? 1.0f:((M_PI*x[j]/SLMsize)/sinf(M_PI*x[j]/SLMsize));
    float sincy_rec = 1.0f;//(y==0)? 1.0f:((M_PI*y[j]/SLMsize)/sinf(M_PI*y[j]/SLMsize));
    h_desiredAmp[j] = (h_I[j] <= 0.0f) ? 1.0f:(sincx_rec * sincy_rec * sqrtf(e_desired*h_I[j]/Isum)*SLMsize*SLMsize);
  }
}*/

/******************************* Host functions *******************************/

// Timing
double getClock() {
  struct timeval tv;
  int ok;
  ok = gettimeofday(&tv, NULL);
  if (ok < 0) {
    printf("gettimeofday error");
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// Allocate GPU memory and parameters
int setup(const float * const initPhases,       // initial pixel phases
          const float * const aberrationCoeffs, // aberration correction matrix
          const float * const polCoeffs,        // SVPR polynomial coefficients
          const unsigned char * const lut)      // phase-to-uc conversion LUT
{
  // Make sure there's a GPU
  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != 0) {
    printf("No CUDA compatible GPU found\n");
    exit(1);
  } else {
    M_SAFE_CALL(hipSetDevice(0));
  }

  int MaxIterations = 1000;
  data_w = SLM_SIZE;
  float data_w_f = (float)data_w;
  N_pixels = data_w * data_w;
  int logN = (int)(log2(data_w_f));
  SLMpitch_f = 1.0f/data_w_f;

  memsize_spotsf = MAX_SPOTS * sizeof(float);
  memsize_SLM_f = N_pixels * sizeof(float);
  memsize_SLMuc = N_pixels * sizeof(unsigned char);
  n_blocks_Phi = (N_pixels/BLOCK_SIZE + (N_pixels%BLOCK_SIZE == 0 ? 0:1));

  // Memory allocations for all methods
  M_SAFE_CALL(hipMalloc((void**)&d_x, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_y, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_z, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_I, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_desiredAmp, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_weights, MAX_SPOTS*(MaxIterations+1)*sizeof(float)));
  M_SAFE_CALL(hipMalloc((void**)&d_Iobtained, MAX_SPOTS*MaxIterations*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_obtainedPhase, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotRe_f, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotIm_f, memsize_spotsf ));

  int data_w_pow2 = pow(2, ceil(log((float)data_w)/log(2.0f)));
  M_SAFE_CALL(hipMalloc((void**)&d_pSLM_f, data_w_pow2*data_w_pow2*sizeof(float)));//the size of d_pSLM_f must be a power of 2 for the summation algorithm to work
  M_SAFE_CALL(hipMemset(d_pSLM_f, 0, data_w_pow2*data_w_pow2*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_pSLMstart_f, memsize_SLM_f));
  M_SAFE_CALL(hipMalloc((void**)&d_hologram, memsize_SLMuc));
  M_SAFE_CALL(hipMemset(d_pSLMstart_f, 0, N_pixels*sizeof(float)));

  M_SAFE_CALL(hipMemcpy(d_pSLM_f, initPhases, N_pixels*sizeof(float), hipMemcpyHostToDevice));

  /*** Corrections stuff ***/

  // Aberration correction
  if (useAC) {
    hipMalloc((void**) &d_aberrationCoeffs, memsize_SLM_f);
    M_SAFE_CALL(hipMemcpy(d_aberrationCoeffs, aberrationCoeffs, memsize_SLM_f, hipMemcpyHostToDevice));
  }

  // SVPR
  if (useSVPR) {
    int numCoeff[5] = {20, 35, 56, 84, 120};
    if ((3 <= polOrder) && (polOrder <= 7)) {
      numPolCoeffs = numCoeff[polOrder - 3];
    } else {
      printf("Polynomial order out of range. Coerced to 3.\n");
      numPolCoeffs = numCoeff[0];
    }

    hipMalloc((void**) &d_polCoeffs, numPolCoeffs * sizeof(float));
    M_SAFE_CALL(hipMemcpy(d_polCoeffs, polCoeffs, numPolCoeffs * sizeof(float), hipMemcpyHostToDevice));
  }

  // LUT
  if (useLUT && !useSVPR) {
    hipMalloc((void**) &d_lut, MAX_UCHAR * sizeof(unsigned char));
    M_SAFE_CALL(hipMemcpy(d_lut, lut, MAX_UCHAR * sizeof(unsigned char), hipMemcpyHostToDevice));
  }

  // RPC
  if (useRPC && alpha < (2.0f * M_PI))
    useRPC = true;
  else
    useRPC = false;

  status = hipGetLastError();
  return status;
}

// Free GPU memory
int finish()
{
  M_SAFE_CALL(hipFree(d_x));
  M_SAFE_CALL(hipFree(d_y));
  M_SAFE_CALL(hipFree(d_z));
  M_SAFE_CALL(hipFree(d_I));

  M_SAFE_CALL(hipFree(d_weights));
  M_SAFE_CALL(hipFree(d_Iobtained));
  M_SAFE_CALL(hipFree(d_pSLM_f));
  M_SAFE_CALL(hipFree(d_pSLMstart_f));
  M_SAFE_CALL(hipFree(d_hologram));

  if (useLUT) {
    hipFree(d_lut);
  }

  if (useAC) {
    hipFree(d_aberrationCoeffs);
  }

  if (useSVPR) {
    hipFree(d_polCoeffs);
  }

  hipDeviceReset();
  status = hipGetLastError();
  return status;
}

// Generate a hologram
int generateHologram(unsigned char * const hologram, // hologram to send to SLM
                     const float * const spotX,      // x coordinates of spots/traps
                     const float * const spotY,      // y coordinates of spots/traps
                     const float * const spotZ,      // z coordinates of spots/traps
                     const float * const spotI,      // relative intensities of spots/traps
                     const int numSpots,             // number of spots/traps
                     const int numIterations,        // number of iterations to run GSW
                     float * const interAmps,        // intermediate amplitudes (debug)
                     int method)                     // method to use for generating hologram
{
  if (numSpots < 1)
    method = 100;
  else if (numSpots < 3)
    method = 0;

  computeAndCopySpotData(spotX, spotY, spotZ, spotI, numSpots);
  memsize_spotsf = numSpots * sizeof(float);
  double t;

  switch (method) {
    case 0:
      // Generate hologram using "Lenses and Prisms"
      printf("Starting Lenses and Prisms...\n");
      t = getClock();

      lensesAndPrisms<<<n_blocks_Phi, BLOCK_SIZE>>>(d_hologram,
                                                    data_w,
                                                    SLMpitch_f,
                                                    N_pixels,
                                                    spotX, // FIXME: device arrays
                                                    spotY, // FIXME: device arrays
                                                    spotZ, // FIXME: device arrays
                                                    spotI, // FIXME: device arrays + desired amps
                                                    numSpots,
                                                    useAC,
                                                    d_aberrationCoeffs,
                                                    useSVPR,
                                                    numPolCoeffs,
                                                    d_polCoeffs,
                                                    useLUT,
                                                    d_lut);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_CHECK_ERROR();

      if (saveSpotI) {
        calculateI<<<numSpots, SLM_SIZE>>>(d_hologram,
                                           data_w,
                                           SLMpitch_f,
                                           N_pixels,
                                           spotX,
                                           spotY,
                                           spotZ,
                                           d_Iobtained);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
        M_SAFE_CALL(hipMemcpy(interAmps, d_Iobtained, numSpots*sizeof(float), hipMemcpyDeviceToHost));
      }
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      break;
    case 1:
      // Generate holgram using fresnel propagation
      printf("Starting Fresnel...\n");
      t = getClock();

      // Uncomment this to start with pre-calculated hologram:
      //hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice);
      //hipDeviceSynchronize();
      //uc2f<<<n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_f, d_hologram, N_pixels);

      for (int l = 0; l < numIterations; l++) {
        printf("Iteration %d\n", l);
        propagateToSpotPositions<<<numSpots, SLM_SIZE>>>(d_pSLM_f,
                                                         data_w,
                                                         SLMpitch_f,
                                                         N_pixels,
                                                         spotX,
                                                         spotY,
                                                         spotZ,
                                                         d_spotRe_f,
                                                         d_spotIm_f);
        M_CHECK_ERROR();
        hipDeviceSynchronize();

        propagateToSLM<<<n_blocks_Phi, BLOCK_SIZE>>>(d_hologram,
                                                     d_pSLM_f,
                                                     d_pSLMstart_f,
                                                     data_w,
                                                     SLMpitch_f,
                                                     N_pixels,
                                                     l,
                                                     (l == (numIterations - 1)),
                                                     spotX,
                                                     spotY,
                                                     spotZ,
                                                     spotI, // FIXME: desired amps
                                                     d_spotRe_f,
                                                     d_spotIm_f,
                                                     d_weights,
                                                     d_Iobtained,
                                                     saveSpotI,
                                                     numSpots,
                                                     useAC,
                                                     d_aberrationCoeffs,
                                                     useSVPR,
                                                     numPolCoeffs,
                                                     d_polCoeffs,
                                                     useLUT,
                                                     d_lut,
                                                     useRPC,
                                                     alpha);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
      }

      if (saveSpotI)
        M_SAFE_CALL(hipMemcpy(interAmps, d_Iobtained, numSpots*(numIterations)*sizeof(float), hipMemcpyDeviceToHost));
      else
        M_SAFE_CALL(hipMemcpy(interAmps, d_weights, numSpots*(numIterations)*sizeof(float), hipMemcpyDeviceToHost));
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      printf("Time/iteration = %12.8lf seconds\n", t/((double) numIterations));
      break;
    case 100:
      // Apply corrections to pre-calculated hologram
      M_SAFE_CALL(hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice));
      applyCorrections<<<n_blocks_Phi, BLOCK_SIZE>>>(d_hologram,
                                                     SLM_SIZE,
                                                     1.0/SLM_SIZE,
                                                     useAC,
                                                     d_aberrationCoeffs,
                                                     useSVPR,
                                                     numPolCoeffs,
                                                     d_polCoeffs,
                                                     useLUT,
                                                     d_lut);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));
      break;
    default:
      break;
  }

  // Handle CUDA errors
  status = hipGetLastError();
  return status;
}

// Calculate amplitude and phase at positions (x, y, z) from a given hologram
int getAmpAndPhase(const float * spotX,                  // x coordinates of spots/traps
                   const float * spotY,                  // y coordinates of spots/traps
                   const float * spotZ,                  // z coordinates of spots/traps
                   const int numSpots,                   // number of spots/traps
                   const unsigned char * const hologram, // hologram to use
                   float *amp,                           // amplitude at (x, y, z)
                   float *phase)                         // phase at (x, y, z)
{
  float *d_amp;
  float *d_phase;
  hipMalloc((void**)&d_amp, numSpots * sizeof(float));
  hipMalloc((void**)&d_phase, numSpots * sizeof(float));
  hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice);

  int offset = 0;
  int num_spots_rem = numSpots;
  int num_spots_this;

  while (num_spots_rem > 0) {
    num_spots_this = (num_spots_rem > MAX_SPOTS) ? MAX_SPOTS : num_spots_rem;
    hipMemcpyToSymbol(HIP_SYMBOL(c_x), spotX + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_y), spotY + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_z), spotZ + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    // FIXME: device arrays for spots
    calculateIAndPhase<<<num_spots_this, SLM_SIZE>>>(d_hologram,
                                                     SLM_SIZE,
                                                     1/SLM_SIZE,
                                                     SLM_SIZE*SLM_SIZE,
                                                     spotX,
                                                     spotY,
                                                     spotZ,
                                                     d_amp+offset,
                                                     d_phase+offset);
    hipDeviceSynchronize();

    num_spots_rem -= MAX_SPOTS;
    offset += MAX_SPOTS;
  }

  hipMemcpy(amp, d_amp, numSpots*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(phase, d_phase, numSpots*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_amp);
  hipFree(d_phase);

  status = hipGetLastError();
  return status;
}

void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n)
{
  //float Isum = 0.0f;
  //for (int i = 0; i<n; i++)
  //  Isum += intensity[i];

  for (int j = 0; j < n; j++) {
    float sincx_rec = (x[j] == 0) ? 1.0f : ((M_PI * x[j]/SLMsizef) / sinf(M_PI * x[j]/SLMsizef));
    float sincy_rec = (y[j] == 0) ? 1.0f : ((M_PI * y[j]/SLMsizef) / sinf(M_PI * y[j]/SLMsizef));
    h_desiredAmp[j] = (intensity[j] <= 0.0f) ? 1.0f : (sincx_rec * sincy_rec * sqrtf(intensity[j]/100) * SLMsizef * SLMsizef);
  }

  hipMemcpyToSymbol(HIP_SYMBOL(c_x), x, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_y), y, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_z), z, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_desiredAmp), h_desiredAmp, n * sizeof(float), 0, hipMemcpyHostToDevice);
}

int main()
{
  srand(1);
  const int numPixels = SLM_SIZE * SLM_SIZE; // 512^2
  const int numSpots = 4; // Four spots
  const int method = 100; // 0 => Direct, 1 => Fresnel, 100 => Corrections
  const int iterations = 10; // 10 iterations for convergence

  // Spots/traps. These form a quadrant across four planes.
  const float x[] = {-128.0f, -128.0f, 127.0f, 127.0f};
  const float y[] = {127.0f, -128.0f, 127.0f, -128.0f};
  const float z[] = {1.0f, 2.0f, 3.0f, 4.0f};
  const float I[] = {0.12f, 0.34f, 0.56f, 0.78f};

  // Correction parameters
  useAC = false;
  useSVPR = false;
  useLUT = false;
  useRPC = false;
  saveSpotI = false;

  polOrder = 5;
  alpha = 2.0f * M_PI * 0.123f;

  const float * const aberrationCoeffs = NULL;
  const unsigned char * const lut = NULL;
  float * const polCoeffs = (float *) malloc(MAX_POL * sizeof(float));
  for (int i = 0; i < MAX_POL; i++) {
    polCoeffs[i] = random() / ((float) RAND_MAX);
  }

  unsigned char *hologram = (unsigned char *) malloc(numPixels * sizeof(unsigned char));
  float * const initPhases = (float *) malloc(numPixels * sizeof(float)); // [-pi, pi]
  for (int i = 0; i < numPixels; i++) {
    hologram[i] = 0.0f;
    initPhases[i] = (2.0 * M_PI * (random() / ((float) RAND_MAX))) - M_PI;
  }

  float *amps = (float *) malloc(numSpots * iterations * sizeof(float));
  for (int i = 0; i < numSpots * iterations; i++) {
    amps[i] = 0.0f;
  }

  if (setup(initPhases, aberrationCoeffs, polCoeffs, lut) != 0) {
    printf("Init failed.\n");
    exit(1);
  }

  double t = getClock();

  if (generateHologram(hologram, x, y, z, I, numSpots, iterations, amps, method) != 0) {
    printf("Computation failed.\n");
    exit(1);
  }

  t = getClock() - t;

  if (finish() != 0) {
    printf("Cleanup failed.\n");
    exit(1);
  }

  printf("Total time = %12.8lf seconds\n", t);

  // Save hologram
  FILE *hfile = fopen("orig_hologram.dat", "w");
  for (int i = 0; i < numPixels; i++) {
    fprintf(hfile, "%hhu\n", hologram[i]);
  }

  // Save amplitudes
  FILE *afile = fopen("orig_amps.dat", "w");
  for (int i = 0; i < numSpots * iterations; i++) {
    fprintf(afile, "%f\n", amps[i]);
  }

  fclose(hfile);
  fclose(afile);

  return 0;
}

