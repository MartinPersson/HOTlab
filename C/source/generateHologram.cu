#include "hip/hip_runtime.h"
/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

// Activates a number of custom debug macros
#define M_CUDA_DEBUG
//#define M_CORE_DEBUG

// Includes
#include "hologram.h"

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 64 // Even 64 is pushing it
#define MAX_POL 120
#define MAX_UCHAR 256

#define BLOCK_SIZE 1024
#define BLOCK_STRIDE 8

// FIXME: This shouldn't be hardcoded
#define SLM_HEIGHT 4096
#define SLM_WIDTH 4096
#define NUM_PIXELS (SLM_HEIGHT * SLM_WIDTH)
#define NUM_CHANNELS 3
#define NUM_SPOTS 16

void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n);


// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, const char *file);
inline void mCheckError(int line, const char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataUC(unsigned char *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

// Hologram info
unsigned char *d_hologram; // [0, 255]
float *d_hologramPhase;    // [-pi, pi]
float *d_prevHologramPhase;

const int slmWidth = SLM_WIDTH;
const int slmHeight = SLM_HEIGHT;
const float slmPitch = 1.0f / ((float) SLM_WIDTH);
const int numPixels = SLM_WIDTH * SLM_HEIGHT;
const int numIterations = 10;
const int numSpots = NUM_SPOTS;
const int numLocalSumPerUnit = ceil(1.0 * numPixels/(BLOCK_SIZE * BLOCK_STRIDE));
const int numLocalSum = numLocalSumPerUnit * NUM_CHANNELS * numSpots;
int hologramMemSize;

// Spot info
float *d_local_spotRe;
float *d_local_spotIm;
float *d_desiredAmp;
float *d_spotRe;
float *d_spotIm;
float *d_phase;
float *d_weights;
float *d_obtainedI;
bool saveSpotI;
int weightMemSize;

// Aberration correction
bool useAC;
float *d_aberrationCoeffs;

// Spatially Varying Phase Response
bool useSVPR;
int polOrder;
int numPolCoeffs;
float *d_polCoeffs;

// phase-to-uc LUT
bool useLUT;
unsigned char *d_lut;

// Restricted Phase Change
bool useRPC;
float alpha;

// Error checking
char CUDAmessage[100];
hipError_t status;

__constant__ float d_spotX[numSpots];
__constant__ float d_spotY[numSpots];
__constant__ float d_spotZ[numSpots];

/****************************** Device functions ******************************/

// Convert from unsigned char [0, 255] to phase (float) [-pi, pi]
__device__ inline float uc2phase(const unsigned char uc)
{
  return (((float) uc) * 2.0f * M_PI/256.0f - M_PI);
}

// Convert from phase (float) [-pi, pi] to unsigned char [0, 255]
__device__ inline unsigned char phase2uc(const float phase)
{
  return (unsigned char) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Convert from phase (float) [-pi, pi] to int
__device__ inline int phase2int(const float phase)
{
  return (int) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Apply wavefront distortion/aberration correction
__device__ inline float applyAberrationCorrection(float pSpot, const float correction)
{
  // First apply correction, then apply mod([-pi, pi], pSpot)
  pSpot = pSpot - correction;
  return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI)));
}

// Get x coordinate from global thread ID
__device__ inline int getXIdx(const int index, const int slmDim)
{
  return index % slmDim;
}

// Get y coordinate from global thread ID and x coordinate
__device__ inline int getYIdx(const int index, const int xIdx, const float slmPitch)
{
  return floor(((float) (index - xIdx)) * slmPitch);
}

// Get pixel coordinates in [-0.5, 0.5]
__device__ inline float getPixelCoords(const int index, const int slmDim, const float slmPitch)
{
  return (slmPitch * ((float) (index - (slmDim >> 1))));
}

// Compute phase from pixel position and spot position
__device__ inline float computePhase(const float x,
                                     const float y,
                                     const float spotx,
                                     const float spoty,
                                     const float spotz)
{
  // TODO: use equation from paper?
  return (M_PI * (spotz * (x*x + y*y) + 2.0f * (x*spotx + y*spoty)));
}

// Apply SVPR
__device__ unsigned char applySVPR(const float phase,
                                   const float x,
                                   const float y,
                                   const float * const coeff,
                                   const int numPolCoeffs)
{
  float result = 0.0f;
  switch (numPolCoeffs) {
    case 120:
      result += coeff[84]*x*x*x*x*x*x*x;
      result += coeff[85]*x*x*x*x*x*x*y;
      result += coeff[86]*x*x*x*x*x*x*phase;
      result += coeff[87]*x*x*x*x*x*y*y;
      result += coeff[88]*x*x*x*x*x*y*phase;
      result += coeff[89]*x*x*x*x*x*phase*phase;
      result += coeff[90]*x*x*x*x*y*y*y;
      result += coeff[91]*x*x*x*x*y*y*phase;
      result += coeff[92]*x*x*x*x*y*phase*phase;
      result += coeff[93]*x*x*x*x*phase*phase*phase;
      result += coeff[94]*x*x*x*y*y*y*y;
      result += coeff[95]*x*x*x*y*y*y*phase;
      result += coeff[96]*x*x*x*y*y*phase*phase;
      result += coeff[97]*x*x*x*y*phase*phase*phase;
      result += coeff[98]*x*x*x*phase*phase*phase*phase;
      result += coeff[99]*x*x*y*y*y*y*y;
      result += coeff[100]*x*x*y*y*y*y*phase;
      result += coeff[101]*x*x*y*y*y*phase*phase;
      result += coeff[102]*x*x*y*y*phase*phase*phase;
      result += coeff[103]*x*x*y*phase*phase*phase*phase;
      result += coeff[104]*x*x*phase*phase*phase*phase*phase;
      result += coeff[105]*x*y*y*y*y*y*y;
      result += coeff[106]*x*y*y*y*y*y*phase;
      result += coeff[107]*x*y*y*y*y*phase*phase;
      result += coeff[108]*x*y*y*y*phase*phase*phase;
      result += coeff[109]*x*y*y*phase*phase*phase*phase;
      result += coeff[110]*x*y*phase*phase*phase*phase*phase;
      result += coeff[111]*x*phase*phase*phase*phase*phase*phase;
      result += coeff[112]*y*y*y*y*y*y*y;
      result += coeff[113]*y*y*y*y*y*y*phase;
      result += coeff[114]*y*y*y*y*y*phase*phase;
      result += coeff[115]*y*y*y*y*phase*phase*phase;
      result += coeff[116]*y*y*y*phase*phase*phase*phase;
      result += coeff[117]*y*y*phase*phase*phase*phase*phase;
      result += coeff[118]*y*phase*phase*phase*phase*phase*phase;
      result += coeff[119]*phase*phase*phase*phase*phase*phase*phase;
    case 84:
      result += coeff[56]*x*x*x*x*x*x;
      result += coeff[57]*x*x*x*x*x*y;
      result += coeff[58]*x*x*x*x*x*phase;
      result += coeff[59]*x*x*x*x*y*y;
      result += coeff[60]*x*x*x*x*y*phase;
      result += coeff[61]*x*x*x*x*phase*phase;
      result += coeff[62]*x*x*x*y*y*y;
      result += coeff[63]*x*x*x*y*y*phase;
      result += coeff[64]*x*x*x*y*phase*phase;
      result += coeff[65]*x*x*x*phase*phase*phase;
      result += coeff[66]*x*x*y*y*y*y;
      result += coeff[67]*x*x*y*y*y*phase;
      result += coeff[68]*x*x*y*y*phase*phase;
      result += coeff[69]*x*x*y*phase*phase*phase;
      result += coeff[70]*x*x*phase*phase*phase*phase;
      result += coeff[71]*x*y*y*y*y*y;
      result += coeff[72]*x*y*y*y*y*phase;
      result += coeff[73]*x*y*y*y*phase*phase;
      result += coeff[74]*x*y*y*phase*phase*phase;
      result += coeff[75]*x*y*phase*phase*phase*phase;
      result += coeff[76]*x*phase*phase*phase*phase*phase;
      result += coeff[77]*y*y*y*y*y*y;
      result += coeff[78]*y*y*y*y*y*phase;
      result += coeff[79]*y*y*y*y*phase*phase;
      result += coeff[80]*y*y*y*phase*phase*phase;
      result += coeff[81]*y*y*phase*phase*phase*phase;
      result += coeff[82]*y*phase*phase*phase*phase*phase;
      result += coeff[83]*phase*phase*phase*phase*phase*phase;
    case 56:
      result += coeff[35]*x*x*x*x*x;
      result += coeff[36]*x*x*x*x*y;
      result += coeff[37]*x*x*x*x*phase;
      result += coeff[38]*x*x*x*y*y;
      result += coeff[39]*x*x*x*y*phase;
      result += coeff[40]*x*x*x*phase*phase;
      result += coeff[41]*x*x*y*y*y;
      result += coeff[42]*x*x*y*y*phase;
      result += coeff[43]*x*x*y*phase*phase;
      result += coeff[44]*x*x*phase*phase*phase;
      result += coeff[45]*x*y*y*y*y;
      result += coeff[46]*x*y*y*y*phase;
      result += coeff[47]*x*y*y*phase*phase;
      result += coeff[48]*x*y*phase*phase*phase;
      result += coeff[49]*x*phase*phase*phase*phase;
      result += coeff[50]*y*y*y*y*y;
      result += coeff[51]*y*y*y*y*phase;
      result += coeff[52]*y*y*y*phase*phase;
      result += coeff[53]*y*y*phase*phase*phase;
      result += coeff[54]*y*phase*phase*phase*phase;
      result += coeff[55]*phase*phase*phase*phase*phase;
    case 35:
      result += coeff[20]*x*x*x*x;
      result += coeff[21]*x*x*x*y;
      result += coeff[22]*x*x*x*phase;
      result += coeff[23]*x*x*y*y;
      result += coeff[24]*x*x*y*phase;
      result += coeff[25]*x*x*phase*phase;
      result += coeff[26]*x*y*y*y;
      result += coeff[27]*x*y*y*phase;
      result += coeff[28]*x*y*phase*phase;
      result += coeff[29]*x*phase*phase*phase;
      result += coeff[30]*y*y*y*y;
      result += coeff[31]*y*y*y*phase;
      result += coeff[32]*y*y*phase*phase;
      result += coeff[33]*y*phase*phase*phase;
      result += coeff[34]*phase*phase*phase*phase;
    case 20:
      result += coeff[0];
      result += coeff[1]*x;
      result += coeff[2]*y;
      result += coeff[3]*phase;
      result += coeff[4]*x*x;
      result += coeff[5]*x*y;
      result += coeff[6]*x*phase;
      result += coeff[7]*y*y;
      result += coeff[8]*y*phase;
      result += coeff[9]*phase*phase;
      result += coeff[10]*x*x*x;
      result += coeff[11]*x*x*y;
      result += coeff[12]*x*x*phase;
      result += coeff[13]*x*y*y;
      result += coeff[14]*x*y*phase;
      result += coeff[15]*x*phase*phase;
      result += coeff[16]*y*y*y;
      result += coeff[17]*y*y*phase;
      result += coeff[18]*y*phase*phase;
      result += coeff[19]*phase*phase*phase;
      break;
    default:
      result = 0.0f;
      break;
  }

  if (result < 0.0f)
    result = 0.0f;

  return (unsigned char) result;
}

// Performs intra-warp reduction. The arrays are in shared memory.
__device__ void warpReduce(volatile float *vRe, volatile float *vIm, int tid)
{
  vRe[tid] += vRe[tid + 32];
  vIm[tid] += vIm[tid + 32];

  vRe[tid] += vRe[tid + 16];
  vIm[tid] += vIm[tid + 16];

  vRe[tid] += vRe[tid + 8];
  vIm[tid] += vIm[tid + 8];

  vRe[tid] += vRe[tid + 4];
  vIm[tid] += vIm[tid + 4];

  vRe[tid] += vRe[tid + 2];
  vIm[tid] += vIm[tid + 2];

  vRe[tid] += vRe[tid + 1];
  vIm[tid] += vIm[tid + 1];
}

// Propagate from the SLM to the spot positions using Fresnel summation
// FIXME: Works only for blocksize = SLMsize
__device__ inline float normalizeCoordinate(int idx, int total){
	return ((float)idx - (total >> 1))/total;
}
__global__ void propagateToSpotPositions(// Hologram information
                                         const float * const hologramPhase,    // hologram's phase
                                         const int slmWidth,
                                         const int slmHeight,
                                         // Spot information
                                         float * const spotRe,                 // real component of spot states
                                         float * const spotIm)                 // imaginary component of spot states
{
	__shared__ float vRe[BLOCK_SIZE];
	__shared__ float vIm[BLOCK_SIZE];

  	const int numPixels = slmWidth * slmHeight;
	int pixelIdx = blockIdx.x * BLOCK_SIZE * BLOCK_STRIDE + threadIdx.x;
  	int channelOffset = numPixels * blockIdx.z;
	float spotx = d_spotX[blockIdx.y];
	float spoty = d_spotY[blockIdx.y];
	float spotz = d_spotZ[blockIdx.y];
	int tid = threadIdx.x;
	vRe[tid] = 0.0f;
	vIm[tid] = 0.0f;

	for (int i = 0; i < BLOCK_STRIDE; ++i){
		// calculate normalized [-0.5, 0.5] pixel coordinate
		int coordX = pixelIdx % slmWidth;
		int coordY = pixelIdx / slmWidth;
		float normalX = normalizeCoordinate(coordX, slmWidth);
		float normalY = normalizeCoordinate(coordY, slmHeight);
		float pixelPhase = hologramPhase[pixelIdx + channelOffset] - computePhase(normalX, normalY, spotx, spoty, spotz);
		int validPixel = pixelIdx < numPixels;
		vRe[tid] += cosf(pixelPhase) * validPixel;
		vIm[tid] += sinf(pixelPhase) * validPixel;
		pixelIdx += BLOCK_SIZE;
	}
	if (BLOCK_SIZE > 1024){
    __syncthreads();
		if (tid < 1024){
			vRe[tid] += vRe[tid + 1024];
	  	vIm[tid] += vIm[tid + 1024];
		}
	}
	if (BLOCK_SIZE > 512){
    __syncthreads();
		if (tid < 512){
			vRe[tid] += vRe[tid + 512];
	  	vIm[tid] += vIm[tid + 512];
		}
	}
	if (BLOCK_SIZE > 256){
    __syncthreads();
		if (tid < 256) {
			vRe[tid] += vRe[tid + 256];
			vIm[tid] += vIm[tid + 256];
		}
	}

	__syncthreads();
	if (tid < 128) {
		vRe[tid] += vRe[tid + 128];
		vIm[tid] += vIm[tid + 128];
	}
	__syncthreads();
	if (tid < 64) {
		vRe[tid] += vRe[tid + 64];
		vIm[tid] += vIm[tid + 64];
	}
	__syncthreads();
	if (tid < 32)
		warpReduce(vRe, vIm, tid);

	// put back the local sum to global memory
	int localSumIdx = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
	if (tid == 0) {
		spotRe[localSumIdx] = vRe[0];
		spotIm[localSumIdx] = vIm[0];
	}
}

__global__ void propagateToSpotSum(float* local_spotRe,
                                     float* local_spotIm,
                                     const int numLocalSumPerUnit,
                                     const int numSpots,
                                     float* spotRe,
                                     float* spotIm
                                    )
{
  __shared__ float temp_spotRe[BLOCK_SIZE];
  __shared__ float temp_spotIm[BLOCK_SIZE];
  const int gidx = blockIdx.x + blockIdx.y * numSpots;
  const int tid = threadIdx.x;
  int sumIdx = numLocalSumPerUnit * gidx + tid;
  const int sumIdxEnd = numLocalSumPerUnit * gidx + numLocalSumPerUnit;
  temp_spotRe[tid] = 0;
  temp_spotIm[tid] = 0;
  // sum into a BLOCK_SIZE array
  while (sumIdx < sumIdxEnd)
  {
    temp_spotRe[tid] += local_spotRe[sumIdx];
    temp_spotIm[tid] += local_spotIm[sumIdx];
    sumIdx += BLOCK_SIZE;
  }
  // Parrallel reduction
  if (BLOCK_SIZE > 1024){
    __syncthreads();
    if (tid < 1024){
      temp_spotRe[tid] += temp_spotRe[tid + 1024];
      temp_spotIm[tid] += temp_spotIm[tid + 1024];
    }
  }

  if (BLOCK_SIZE > 512){
    __syncthreads();
    if (tid < 512){
      temp_spotRe[tid] += temp_spotRe[tid + 512];
      temp_spotIm[tid] += temp_spotIm[tid + 512];
    }
  }
  if (BLOCK_SIZE > 256){
    __syncthreads();
    if (tid < 256) {
      temp_spotRe[tid] += temp_spotRe[tid + 256];
      temp_spotIm[tid] += temp_spotIm[tid + 256];
    }
  }
  __syncthreads();
  if (tid < 128) {
    temp_spotRe[tid] += temp_spotRe[tid + 128];
    temp_spotIm[tid] += temp_spotIm[tid + 128];
  }
  __syncthreads();
  if (tid < 64) {
    temp_spotRe[tid] += temp_spotRe[tid + 64];
    temp_spotIm[tid] += temp_spotIm[tid + 64];
  }
  __syncthreads();
  if (tid < 32)
    warpReduce(temp_spotRe, temp_spotIm, tid);

  if (tid == 0){
    spotRe[gidx] = temp_spotRe[0];
    spotIm[gidx] = temp_spotIm[0];
  }
};

// Obtain phases in SLM plane
__global__ void propagateToSLM(// Hologram information
                               unsigned char * const hologram,       // output hologram
                               float * const hologramPhase,          // current hologram phase
                               float * const prevHologramPhase,      // previous hologram phase, used for RPC
                               const unsigned int slmDim,            // SLM's dimension
                               const float slmPitch,                 // 1/slmDim
                               const unsigned int numPixels,         // number of pixels in SLM
                               const int iteration,                  // GSW iteration number
                               const bool lastIteration,             // true if last iteration of GSW
                               // Spot information
                               const float * const spotDesiredAmp,   // desired amplitudes of spots
                               const float * const spotRe,           // real component of spot states
                               const float * const spotIm,           // imaginary component of spot states
                               float * const spotWeight,             // spot weights
                               float * const spotI,                  // intermediate spot intensities (debug)
                               const bool saveSpotI,                 // if true, spot intensities are saved to global memory (debug)
                               const unsigned int numSpots,          // number of spots
                               // Correction information
                               const bool useAC,                     // use aberration correction
                               const float * const aberrationCoeffs, // correction coefficients
                               const bool useSVPR,                   // use spatially varying phase response
                               const int numPolCoeffs,               // number of polynomial coefficients
                               const float * const polCoeffs,        // polynomial coefficients
                               const bool useLUT,                    // use LUT for phase-to-uc conversion
                               const unsigned char * const lut,      // LUT for phase-to-uc conversion
                               const bool useRPC,                    // use restricted phase change
                               const float alpha)                    // RPC threshold
{
  __shared__ float spotAMean;
  __shared__ float spotP[MAX_SPOTS];
  __shared__ float spotA[MAX_SPOTS];
  __shared__ float spotW[MAX_SPOTS];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;
  const int channelSpotOffset = numSpots * blockIdx.y;
  const int channelPixelOffset = numPixels * blockIdx.y;

  float pixelRe = 0.0f;
  float pixelIm = 0.0f;
  float pixelPhase = 0.0f;

  if (idx < numPixels) {
    // Load spot phases, amplitudes, and weights
    // FIXME: would be faster on CPU?
    if (tid < numSpots) {
      float re = spotRe[tid + channelSpotOffset];
      float im = spotIm[tid + channelSpotOffset];
      spotP[tid] = atan2f(im, re);
      spotA[tid] = hypotf(re, im)/spotDesiredAmp[tid];
      if (iteration != 0) {
        spotW[tid] = spotWeight[tid + iteration*numSpots];
      } else {
        spotA[tid] = (spotA[tid] < 0.5f) ? 0.5f : spotA[tid];
        spotW[tid] = spotDesiredAmp[tid];
      }
    }
    __syncthreads();

    // Compute mean spot amplitude
    // FIXME: parallel reduction?
    if (tid == 0) {
      float sum = 0.0f;
      for (int i = 0; i < numSpots; i++) {
        sum += spotA[i];
      }
      spotAMean = sum/((float) numSpots);
    }
    __syncthreads();

    // Update spot weights
    if (tid < numSpots) {
      spotW[tid] = spotW[tid] * spotAMean / spotA[tid];

      // Copy weights to use as initial value in next run
      if (!lastIteration)
        spotWeight[tid + numSpots*(iteration + 1)] = spotW[tid];

      // May be excluded, used for monitoring only
      if (saveSpotI)
        spotI[tid + numSpots*iteration] = spotA[tid]*spotA[tid];
    }
    __syncthreads();

    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    // Compute the pixel's phase by summing contributions from all spots
    // TODO: use scratchpad for spot x, y, z
    for (int i = 0; i < numSpots; i++) {
      float delta = computePhase(x, y, d_spotX[i], d_spotY[i], d_spotZ[i]);
      pixelRe += spotW[i] * cosf(spotP[i] + delta);
      pixelIm += spotW[i] * sinf(spotP[i] + delta);
    }

    pixelPhase = atan2f(pixelIm, pixelRe);
    if (useRPC) {
      float prevPhase = prevHologramPhase[idx + channelPixelOffset];
      if (fabs(pixelPhase - prevPhase) > alpha)
        pixelPhase = prevPhase;

      if (lastIteration)
        prevHologramPhase[idx + channelPixelOffset] = pixelPhase;
    }

    // This is the last iteration, compute and write the final hologram phases to global memory
    if (lastIteration) {
      if (useAC)
        pixelPhase = applyAberrationCorrection(pixelPhase, aberrationCoeffs[idx]);

      if (useSVPR) {
        __shared__ float coeff[MAX_POL];
        if (tid < numPolCoeffs)
          coeff[tid] = polCoeffs[tid];
        __syncthreads();

        hologram[idx + channelPixelOffset] = applySVPR(pixelPhase, x, y, coeff, numPolCoeffs);
      } else if (useLUT) {
        __shared__ unsigned char lut_t[MAX_UCHAR];
        if (tid < MAX_UCHAR)
          lut_t[tid] = lut[tid];
        __syncthreads();

        hologram[idx + channelPixelOffset] = lut_t[phase2int(pixelPhase)];
      } else {
        hologram[idx + channelPixelOffset] = phase2uc(pixelPhase);
      }
    } else { // Otherwise, write intermediate phases to global memory
      hologramPhase[idx + channelPixelOffset] = pixelPhase;
    }
  }
}

// Convert from unsigned char [0, 255] to float [-pi, pi]
__global__ void uc2f(float *f, const unsigned char * const uc, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    f[idx] = uc[idx] * 2.0f * M_PI/256.0f - M_PI;
  }
}

// Custom debug functions
inline void mSafeCall(hipError_t status, int line, const char *file)
{
#ifdef M_CUDA_DEBUG
  do {
    if (status != hipSuccess) {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess) {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  } while (0);
#endif
  return;
}

inline void mCheckError(int line, const char *file)
{
#ifdef M_CUDA_DEBUG
  do
  {
    hipError_t status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  }while(0);
#endif
  return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    float *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (float*)malloc(length * sizeof (float));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(float), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %f", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataUC(unsigned char *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    unsigned char *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (unsigned char*)malloc(length * sizeof (unsigned char));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(unsigned char), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %hhu", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    int *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (int*)malloc(length * sizeof (int));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(int), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %d", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

/******************************* Host functions *******************************/

// Timing
double getClock() {
  struct timeval tv;
  int ok;
  ok = gettimeofday(&tv, NULL);
  if (ok < 0) {
    printf("gettimeofday error");
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// Allocate GPU memory and parameters
int setup(const float * const initPhases,       // initial pixel phases
          const float * const aberrationCoeffs, // aberration correction matrix
          const float * const polCoeffs,        // SVPR polynomial coefficients
          const unsigned char * const lut)      // phase-to-uc conversion LUT
{
  // Make sure there's a GPU that we can use
  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != 0) {
    printf("No CUDA compatible GPU found\n");
    exit(1);
  } else {
    M_SAFE_CALL(hipSetDevice(0));
  }

  /*** Hologram ***/
  hologramMemSize = numPixels * sizeof(unsigned char) * NUM_CHANNELS;
  const unsigned int hologramPhaseMemSize = numPixels * sizeof(float) * NUM_CHANNELS;
  M_SAFE_CALL(hipMalloc((void **) &d_hologram, hologramMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_hologramPhase, hologramPhaseMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_prevHologramPhase, hologramPhaseMemSize));
  M_SAFE_CALL(hipMemcpy(d_hologramPhase, initPhases, hologramPhaseMemSize, hipMemcpyHostToDevice));
  M_SAFE_CALL(hipMemcpy(d_prevHologramPhase, initPhases, hologramPhaseMemSize, hipMemcpyHostToDevice));

  /*** Spots ***/
  const unsigned int spotMemSize = numSpots * sizeof(float);
  // local sum
  M_SAFE_CALL(hipMalloc((void **) &d_local_spotRe,
                          numLocalSum * sizeof(float)));
  M_SAFE_CALL(hipMalloc((void **) &d_local_spotIm,
                          numLocalSum * sizeof(float)));
  // final sum
  weightMemSize = numSpots * numIterations * sizeof(float);
  M_SAFE_CALL(hipMalloc((void **) &d_desiredAmp, NUM_CHANNELS * spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_spotRe, NUM_CHANNELS * spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_spotIm, NUM_CHANNELS * spotMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_weights, NUM_CHANNELS * weightMemSize));
  M_SAFE_CALL(hipMalloc((void **) &d_obtainedI, NUM_CHANNELS * weightMemSize));

  /*** Corrections ***/

  // Aberration correction
  if (useAC) {
    hipMalloc((void **) &d_aberrationCoeffs, hologramPhaseMemSize);
    M_SAFE_CALL(hipMemcpy(d_aberrationCoeffs, aberrationCoeffs, hologramPhaseMemSize, hipMemcpyHostToDevice));
  }

  // SVPR
  if (useSVPR) {
    int numCoeff[5] = {20, 35, 56, 84, 120};
    if ((3 <= polOrder) && (polOrder <= 7)) {
      numPolCoeffs = numCoeff[polOrder - 3];
    } else {
      printf("Polynomial order out of range. Coerced to 3.\n");
      numPolCoeffs = numCoeff[0];
    }

    hipMalloc((void **) &d_polCoeffs, numPolCoeffs * sizeof(float));
    M_SAFE_CALL(hipMemcpy(d_polCoeffs, polCoeffs, numPolCoeffs * sizeof(float), hipMemcpyHostToDevice));
  }

  // LUT
  if (useLUT && !useSVPR) {
    hipMalloc((void **) &d_lut, MAX_UCHAR * sizeof(unsigned char));
    M_SAFE_CALL(hipMemcpy(d_lut, lut, MAX_UCHAR * sizeof(unsigned char), hipMemcpyHostToDevice));
  }
  // RPC
  if (useRPC && alpha < (2.0f * M_PI))
    useRPC = true;
  else
    useRPC = false;
  status = hipGetLastError();
  return status;
}

// Free GPU memory
int finish()
{
  /*** Hologram ***/
  M_SAFE_CALL(hipFree(d_hologram));
  M_SAFE_CALL(hipFree(d_hologramPhase));
  M_SAFE_CALL(hipFree(d_prevHologramPhase));
  /*** Spots ***/
  M_SAFE_CALL(hipFree(d_desiredAmp));
  M_SAFE_CALL(hipFree(d_spotRe));
  M_SAFE_CALL(hipFree(d_spotIm));
  M_SAFE_CALL(hipFree(d_weights));
  M_SAFE_CALL(hipFree(d_obtainedI));
  if (useAC) {
    M_SAFE_CALL(hipFree(d_aberrationCoeffs));
  }
  if (useSVPR) {
    M_SAFE_CALL(hipFree(d_polCoeffs));
  }
  if (useLUT) {
    M_SAFE_CALL(hipFree(d_lut));
  }
  hipDeviceReset();
  status = hipGetLastError();
  return status;
}

// Generate a hologram
int generateHologram(unsigned char * const hologram, // hologram to send to SLM
                     const float * const spotX,      // x coordinates of spots/traps
                     const float * const spotY,      // y coordinates of spots/traps
                     const float * const spotZ,      // z coordinates of spots/traps
                     const float * const spotI,      // relative intensities of spots/traps
                     const int numSpots,             // number of spots/traps
                     const int numIterations,        // number of iterations to run GSW
                     float * const interAmps)        // intermediate amplitudes (debug)
{
	computeAndCopySpotData(spotX, spotY, spotZ, spotI, numSpots);
	double t;
	dim3 toSpotGridDim(ceil(1.0 * NUM_PIXELS/(BLOCK_SIZE * BLOCK_STRIDE)),
    				              numSpots,
    				              NUM_CHANNELS);
	dim3 toSpotBlockDim(BLOCK_SIZE, 1, 1);
	dim3 toSpotSumGridDim(numSpots, NUM_CHANNELS, 1);
	dim3 toSpotSumBlockDim(BLOCK_SIZE, 1, 1);
  	dim3 toSLMGridDim(ceil(1.0 * numPixels/BLOCK_SIZE), NUM_CHANNELS, 1);
  	dim3 toSLMBlockDim(BLOCK_SIZE, 1, 1);

    printf("Starting Fresnel...\n");
    t = getClock();
    // Uncomment this to start with pre-calculated hologram
    //hipMemcpy(d_hologram, hologram, hologramMemSize, hipMemcpyHostToDevice);
    //hipDeviceSynchronize();
    //uc2f<<<numBlocks, BLOCK_SIZE >>>(d_hologramPhase, d_hologram, numPixels);

    for (int l = 0; l < numIterations; l++) {
       	printf("Iteration %d\n", l);
       	propagateToSpotPositions<<<toSpotGridDim, toSpotBlockDim>>>(d_hologramPhase,
       	                                                            slmWidth,
       	                                                            slmHeight,
       	                                                            d_local_spotRe,
       	                                                            d_local_spotIm);
       	M_CHECK_ERROR();
       	hipDeviceSynchronize();
       	// Second level parallel reduction
       	propagateToSpotSum<<<toSpotSumGridDim, toSpotSumBlockDim>>>(d_local_spotRe,
                                                                    d_local_spotIm,
                                                                    numLocalSumPerUnit,
                                                                    numSpots,
                                                                    d_spotRe,
                                                                    d_spotIm);
        M_CHECK_ERROR();
        hipDeviceSynchronize();

    	propagateToSLM<<<toSLMGridDim, toSLMBlockDim>>>(d_hologram,
    	                                                d_hologramPhase,
    	                                                d_prevHologramPhase,
    	                                                slmWidth,
    	                                                slmPitch,
    	                                                numPixels,
    	                                                l,
    	                                                (l == (numIterations - 1)),
    	                                                d_desiredAmp,
    	                                                d_spotRe,
    	                                                d_spotIm,
    	                                                d_weights,
    	                                                d_obtainedI,
    	                                                saveSpotI,
    	                                                numSpots,
    	                                                useAC,
    	                                                d_aberrationCoeffs,
    	                                                useSVPR,
    	                                                numPolCoeffs,
    	                                                d_polCoeffs,
    	                                                useLUT,
    	                                                d_lut,
    	                                                useRPC,
    	                                                alpha);
    	M_CHECK_ERROR();
    	hipDeviceSynchronize();
        // debuggin output
/*        float* peekSpot = (float*)malloc(32 * sizeof(float));
        hipMemcpy(peekSpot, d_hologram, 32 * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < 32; ++i){
          printf("%f ", peekSpot[i]);
        }
        printf("\n");
        M_CHECK_ERROR();
        hipDeviceSynchronize();*/
    }
    	
    if (saveSpotI)
    	M_SAFE_CALL(hipMemcpy(interAmps, d_obtainedI, weightMemSize, hipMemcpyDeviceToHost));
    else
   		M_SAFE_CALL(hipMemcpy(interAmps, d_weights, weightMemSize, hipMemcpyDeviceToHost));
    M_SAFE_CALL(hipMemcpy(hologram, d_hologram, hologramMemSize, hipMemcpyDeviceToHost));
	t = getClock() - t;
	printf("Total time = %12.8lf seconds\n", t);
	printf("Time/iteration = %12.8lf seconds\n", t/((double) numIterations));

	// Handle CUDA errors
	status = hipGetLastError();
	return status;
}

void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n)
{
	// An alternate way is to use sum instead of 100 in the formula below, but
	// I'm not sure what the difference is
	/*
	float sum = 0.0f;
	for (int i = 0; i < n; i++)
		sum += intensity[i];
	*/
	const float slmDimf = (float) slmWidth;
	float *desiredAmp = (float *) malloc(n * sizeof(float));

	for (int i = 0; i < n; i++) {
		float sincxRec = (x[i] == 0) ? 1.0f : ((M_PI * x[i]/slmDimf) / sinf(M_PI * x[i]/slmDimf));
		float sincyRec = (y[i] == 0) ? 1.0f : ((M_PI * y[i]/slmDimf) / sinf(M_PI * y[i]/slmDimf));
		desiredAmp[i] = (intensity[i] <= 0.0f) ? 1.0f : (sincxRec * sincyRec * sqrtf(intensity[i]/100) * slmDimf * slmDimf);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_spotX), x, numSpots * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_spotY), y, numSpots * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_spotZ), z, numSpots * sizeof(float));
	hipMemcpy(d_desiredAmp, desiredAmp, n * sizeof(float), hipMemcpyHostToDevice);
	free(desiredAmp);
}

unsigned char *hologram = (unsigned char *) malloc(numPixels * NUM_CHANNELS * sizeof(unsigned char));
float *amps = (float *) malloc(numSpots * numIterations * NUM_CHANNELS * sizeof(float));
float * const initPhases = (float *) malloc(numPixels * NUM_CHANNELS * sizeof(float)); // [-pi, pi];
float * const polCoeffs = (float *) malloc(MAX_POL * sizeof(float));

bool HLG_initailize()
{
	srand(1);
	// Correction parameters
	saveSpotI = false;
	useAC = false;
	const float * const aberrationCoeffs = NULL;
	useSVPR = true;
	polOrder = 5;
	for (int i = 0; i < MAX_POL; i++) {
		polCoeffs[i] = 0.0f;
	}
	useLUT = false;
	const unsigned char * const lut = NULL;
	useRPC = false;
	alpha = 2.0f * M_PI * 0.123f;
#ifdef M_CORE_DEBUG
  FILE *savedPhase = fopen("my_init_hologram.dat", "r");
  for (int i = 0; i < numPixels * NUM_CHANNELS; i++) {
    fscanf(savedPhase, "%hhu\n", &hologram[i]);
  }
#else
  for (int i = 0; i < numPixels * NUM_CHANNELS; i++) {
		hologram[i] = random() % 256;
		initPhases[i] = (2.0 * M_PI * (random() / ((float) RAND_MAX))) - M_PI;
	}
#endif

	if (setup(initPhases, aberrationCoeffs, polCoeffs, lut) != 0) {
		printf("Init failed.\n");
		exit(1);
	}

#ifndef M_CORE_DEBUG
	// Save initial hologram
	FILE *ifile = fopen("my_init_hologram.dat", "w");
	for (int i = 0; i < numPixels; i++) {
		fprintf(ifile, "%hhu\n", hologram[i]);
	}
  fclose(ifile);
#endif
  return true;
}

bool HLG_process(){
  // Randomize depth planes
  float x[NUM_SPOTS];
  float y[NUM_SPOTS];
  float z[NUM_SPOTS];
  float I[NUM_SPOTS];
  for (int spotIdx = 0; spotIdx < NUM_SPOTS; ++spotIdx){
    x[spotIdx] = (float)(random() & (0xFF)) - 128.0f;
    y[spotIdx] = (float)(random() & (0xFF)) - 128.0f;
    z[spotIdx] = (float)(random() & (0xFF)) - 128.0f;
    I[spotIdx] = (float)(random() & (0xFF)) / 256.0f;
  }
  if (generateHologram(hologram, x, y, z, I, numSpots, numIterations, amps) != 0) {
  	printf("Computation failed.\n");
    return false;
  }
  return true;
}

bool HLG_cleanup(){
  if (finish() != 0) {
    printf("Cleanup failed.\n");
    return false;
  }

	// Save hologram
	FILE *hfile = fopen("new_output_hologram.dat", "w");
	for (int i = 0; i < numPixels; i++) {
		fprintf(hfile, "%hhu\n", hologram[i]);
	}

	// Save amplitudes
	FILE *afile = fopen("new_amps.dat", "w");
	for (int i = 0; i < numSpots * numIterations; i++) {
		fprintf(afile, "%f\n", amps[i]);
	}
#ifndef M_CORE_DEBUG
	
#endif
	fclose(hfile);
	fclose(afile);
	free(polCoeffs);
	free(hologram);
	free(initPhases);
	free(amps);

	return true;
}

