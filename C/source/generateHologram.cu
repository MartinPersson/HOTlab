#include "hip/hip_runtime.h"
/*
   Hologram generating algorithms for CUDA Devices

   Copyright 2009, 2010, 2011, 2012 Martin Persson
   martin.persson@physics.gu.se

   This file is part of GenerateHologramCUDA.

   GenerateHologramCUDA is free software: you can redistribute it and/or
   modify it under the terms of the GNU Lesser General Public License as published
   by the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   GenerateHologramCUDA is distributed in the hope that it will be
   useful, but WITHOUT ANY WARRANTY; without even the implied warranty
   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/

// Activates a number of custom debug macros
#define M_CUDA_DEBUG

// Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hipfft/hipfft.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

// Number of spots/traps/depth planes - even 64 is pushing it
#define MAX_SPOTS 64

// FIXME: This shouldn't be hardcoded
#define SLM_SIZE 512

// Use bitwise modulo operations if the SLM size is a power of 2
#if (((SLM_SIZE - 1) & (SLM_SIZE)) == 0)
#define SLMPOW2
#endif

#define BLOCK_SIZE 256
#define MAX_POL 120
#define MAX_UCHAR 256

// Forward declaration
void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n);

// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, char *file);
inline void mCheckError(int line, char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataUC(unsigned char *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

// Global declaration
float *d_x, *d_y, *d_z, *d_I;         //trap coordinates and intensity in GPU memory
float *d_pSLM_f;                //the optimized pSpot pattern, float [-pi, pi]
float *d_weights, *d_Iobtained, *d_desiredAmp;    //used h_weights and calculated amplitudes for each spot and each iteration
float *d_pSLMstart_f;             //Initial pSpot pattern [-pi, pi]
float *d_spotRe_f, *d_spotIm_f;
float *d_AberrationCorr_f = NULL;
float *d_LUTPolCoeff_f = NULL;
float SLMsizef = (float) SLM_SIZE;
int N_PolLUTCoeff = 0;
int n_blocks_Phi, memsize_SLM_f, memsize_SLMuc, memsize_spotsf, data_w, N_pixels;
float h_desiredAmp[MAX_SPOTS];
unsigned char *d_hologram;           //The optimized pSpot pattern, unsigned char, the one sent to the SLM [0, 255]
unsigned char *d_LUT_uc = NULL;
bool ApplyLUT_b = false, UseAberrationCorr_b = false, UsePolLUT_b = false, saveI_b = false, useRPC_b = false;
float alphaRPC_f = 10;
char CUDAmessage[100];
hipError_t status;
float *d_obtainedPhase;

// Constant memory declarations
__device__ __constant__ int c_data_w[1];
__device__ __constant__ float c_data_w_f[1];
__device__ __constant__ float c_half_w_f[1];
__device__ __constant__ int c_N_pixels[1];
__device__ __constant__ float c_N_pixels_f[1];
__device__ __constant__ float c_SLMpitch_f[1];
__device__ __constant__ bool c_applyLUT_b[1];
__device__ __constant__ bool c_useAberrationCorr_b[1];
__device__ __constant__ bool c_usePolLUT_b[1];
__device__ __constant__ int c_N_PolLUTCoeff[1];
__device__ __constant__ bool c_useRPC_b[1];
__device__ __constant__ float c_alphaRPC_f[1];
__device__ __constant__ bool c_saveI_b[1];
__device__ __constant__ int c_log2data_w[1];
__device__ __constant__ float c_x[MAX_SPOTS];
__device__ __constant__ float c_y[MAX_SPOTS];
__device__ __constant__ float c_z[MAX_SPOTS];
__device__ __constant__ float c_desiredAmp[MAX_SPOTS];
__device__ __constant__ int c_N_spots[1];

/****************************** Device functions ******************************/

// Convert from unsigned char [0, 255] to phase (float) [-pi, pi]
__device__ inline float uc2phase(const unsigned char uc)
{
  return (((float) uc) * 2.0f * M_PI/256.0f - M_PI);
}

// Convert from phase (float) [-pi, pi] to unsigned char [0, 255]
__device__ inline unsigned char phase2uc(const float phase)
{
  return (unsigned char) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Convert from phase (float) [-pi, pi] to int
__device__ inline int phase2int(const float phase)
{
  return (int) floor((phase + M_PI) * 256.0f / (2.0f * M_PI));
}

// Apply wavefront distortion/aberration correction
__device__ inline float applyAberrationCorrection(float pSpot, const float correction)
{
  pSpot = pSpot - correction; // Apply correction
  return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI))); // Apply mod([-pi, pi], pSpot)
}

// Get x coordinate from global thread ID
__device__ inline int getXIdx(const int index, const int slmDim)
{
#ifdef SLMPOW2
  int idx = index & (slmDim - 1);
#else
  int idx = index % slmDim;
#endif
  return idx;
}

// Get y coordinate from global thread ID and x coordinate
__device__ inline int getYIdx(const int index, const int xIdx, const int slmDim, const float slmPitch)
{
#ifdef SLMPOW2
  int idx = (index - xIdx) >> c_log2data_w[0]; // FIXME
#else
  int idx = floor(((float) (index - xIdx)) * slmPitch);
#endif
  return idx;
}

// Get pixel coordinates in [-0.5, 0.5]
__device__ inline float getPixelCoords(const int index, const int slmDim, const float slmPitch)
{
  return (((float) (index - (slmDim << 1))) * slmPitch);
}

// Compute phase from pixel position and spot position
__device__ inline float computePhase(const float x,
                                     const float y,
                                     const float spotx,
                                     const float spoty,
                                     const float spotz)
{
  return (M_PI * (spotz * (x*x + y*y) + 2.0f * (x*spotx + y*spoty)));
}

// Apply SVPR
__device__ unsigned char applySVPR(float phase, float x, float y, float *coeff)
{
  float result = 0.0f;
  switch (c_N_PolLUTCoeff[0]) {
    case 120:
      result += coeff[84]*x*x*x*x*x*x*x;
      result += coeff[85]*x*x*x*x*x*x*y;
      result += coeff[86]*x*x*x*x*x*x*phase;
      result += coeff[87]*x*x*x*x*x*y*y;
      result += coeff[88]*x*x*x*x*x*y*phase;
      result += coeff[89]*x*x*x*x*x*phase*phase;
      result += coeff[90]*x*x*x*x*y*y*y;
      result += coeff[91]*x*x*x*x*y*y*phase;
      result += coeff[92]*x*x*x*x*y*phase*phase;
      result += coeff[93]*x*x*x*x*phase*phase*phase;
      result += coeff[94]*x*x*x*y*y*y*y;
      result += coeff[95]*x*x*x*y*y*y*phase;
      result += coeff[96]*x*x*x*y*y*phase*phase;
      result += coeff[97]*x*x*x*y*phase*phase*phase;
      result += coeff[98]*x*x*x*phase*phase*phase*phase;
      result += coeff[99]*x*x*y*y*y*y*y;
      result += coeff[100]*x*x*y*y*y*y*phase;
      result += coeff[101]*x*x*y*y*y*phase*phase;
      result += coeff[102]*x*x*y*y*phase*phase*phase;
      result += coeff[103]*x*x*y*phase*phase*phase*phase;
      result += coeff[104]*x*x*phase*phase*phase*phase*phase;
      result += coeff[105]*x*y*y*y*y*y*y;
      result += coeff[106]*x*y*y*y*y*y*phase;
      result += coeff[107]*x*y*y*y*y*phase*phase;
      result += coeff[108]*x*y*y*y*phase*phase*phase;
      result += coeff[109]*x*y*y*phase*phase*phase*phase;
      result += coeff[110]*x*y*phase*phase*phase*phase*phase;
      result += coeff[111]*x*phase*phase*phase*phase*phase*phase;
      result += coeff[112]*y*y*y*y*y*y*y;
      result += coeff[113]*y*y*y*y*y*y*phase;
      result += coeff[114]*y*y*y*y*y*phase*phase;
      result += coeff[115]*y*y*y*y*phase*phase*phase;
      result += coeff[116]*y*y*y*phase*phase*phase*phase;
      result += coeff[117]*y*y*phase*phase*phase*phase*phase;
      result += coeff[118]*y*phase*phase*phase*phase*phase*phase;
      result += coeff[119]*phase*phase*phase*phase*phase*phase*phase;
    case 84:
      result += coeff[56]*x*x*x*x*x*x;
      result += coeff[57]*x*x*x*x*x*y;
      result += coeff[58]*x*x*x*x*x*phase;
      result += coeff[59]*x*x*x*x*y*y;
      result += coeff[60]*x*x*x*x*y*phase;
      result += coeff[61]*x*x*x*x*phase*phase;
      result += coeff[62]*x*x*x*y*y*y;
      result += coeff[63]*x*x*x*y*y*phase;
      result += coeff[64]*x*x*x*y*phase*phase;
      result += coeff[65]*x*x*x*phase*phase*phase;
      result += coeff[66]*x*x*y*y*y*y;
      result += coeff[67]*x*x*y*y*y*phase;
      result += coeff[68]*x*x*y*y*phase*phase;
      result += coeff[69]*x*x*y*phase*phase*phase;
      result += coeff[70]*x*x*phase*phase*phase*phase;
      result += coeff[71]*x*y*y*y*y*y;
      result += coeff[72]*x*y*y*y*y*phase;
      result += coeff[73]*x*y*y*y*phase*phase;
      result += coeff[74]*x*y*y*phase*phase*phase;
      result += coeff[75]*x*y*phase*phase*phase*phase;
      result += coeff[76]*x*phase*phase*phase*phase*phase;
      result += coeff[77]*y*y*y*y*y*y;
      result += coeff[78]*y*y*y*y*y*phase;
      result += coeff[79]*y*y*y*y*phase*phase;
      result += coeff[80]*y*y*y*phase*phase*phase;
      result += coeff[81]*y*y*phase*phase*phase*phase;
      result += coeff[82]*y*phase*phase*phase*phase*phase;
      result += coeff[83]*phase*phase*phase*phase*phase*phase;
    case 56:
      result += coeff[35]*x*x*x*x*x;
      result += coeff[36]*x*x*x*x*y;
      result += coeff[37]*x*x*x*x*phase;
      result += coeff[38]*x*x*x*y*y;
      result += coeff[39]*x*x*x*y*phase;
      result += coeff[40]*x*x*x*phase*phase;
      result += coeff[41]*x*x*y*y*y;
      result += coeff[42]*x*x*y*y*phase;
      result += coeff[43]*x*x*y*phase*phase;
      result += coeff[44]*x*x*phase*phase*phase;
      result += coeff[45]*x*y*y*y*y;
      result += coeff[46]*x*y*y*y*phase;
      result += coeff[47]*x*y*y*phase*phase;
      result += coeff[48]*x*y*phase*phase*phase;
      result += coeff[49]*x*phase*phase*phase*phase;
      result += coeff[50]*y*y*y*y*y;
      result += coeff[51]*y*y*y*y*phase;
      result += coeff[52]*y*y*y*phase*phase;
      result += coeff[53]*y*y*phase*phase*phase;
      result += coeff[54]*y*phase*phase*phase*phase;
      result += coeff[55]*phase*phase*phase*phase*phase;
    case 35:
      result += coeff[20]*x*x*x*x;
      result += coeff[21]*x*x*x*y;
      result += coeff[22]*x*x*x*phase;
      result += coeff[23]*x*x*y*y;
      result += coeff[24]*x*x*y*phase;
      result += coeff[25]*x*x*phase*phase;
      result += coeff[26]*x*y*y*y;
      result += coeff[27]*x*y*y*phase;
      result += coeff[28]*x*y*phase*phase;
      result += coeff[29]*x*phase*phase*phase;
      result += coeff[30]*y*y*y*y;
      result += coeff[31]*y*y*y*phase;
      result += coeff[32]*y*y*phase*phase;
      result += coeff[33]*y*phase*phase*phase;
      result += coeff[34]*phase*phase*phase*phase;
    case 20:
      result += coeff[0];
      result += coeff[1]*x;
      result += coeff[2]*y;
      result += coeff[3]*phase;
      result += coeff[4]*x*x;
      result += coeff[5]*x*y;
      result += coeff[6]*x*phase;
      result += coeff[7]*y*y;
      result += coeff[8]*y*phase;
      result += coeff[9]*phase*phase;
      result += coeff[10]*x*x*x;
      result += coeff[11]*x*x*y;
      result += coeff[12]*x*x*phase;
      result += coeff[13]*x*y*y;
      result += coeff[14]*x*y*phase;
      result += coeff[15]*x*phase*phase;
      result += coeff[16]*y*y*y;
      result += coeff[17]*y*y*phase;
      result += coeff[18]*y*phase*phase;
      result += coeff[19]*phase*phase*phase;
      break;
    default:
      result = 0.0f;
      break;
  }

  if (result < 0.0f)
    result = 0.0f;

  return (unsigned char) result;
}

// Performs intra-warp reduction. The arrays are in shared memory.
__device__ void warpReduce(volatile float *vRe, volatile float *vIm, int tid)
{
  vRe[tid] += vRe[tid + 32];
  vIm[tid] += vIm[tid + 32];

  vRe[tid] += vRe[tid + 16];
  vIm[tid] += vIm[tid + 16];

  vRe[tid] += vRe[tid + 8];
  vIm[tid] += vIm[tid + 8];

  vRe[tid] += vRe[tid + 4];
  vIm[tid] += vIm[tid + 4];

  vRe[tid] += vRe[tid + 2];
  vIm[tid] += vIm[tid + 2];

  vRe[tid] += vRe[tid + 1];
  vIm[tid] += vIm[tid + 1];
}

// Apply corrections to precalculated hologram
__global__ void applyCorrections(unsigned char * const hologram,       // hologram to use
                                 const unsigned int slmDim,            // SLM's dimension
                                 const float slmPitch,                 // 1/slmDim
                                 const bool useAC,                     // use aberration correction
                                 const float * const aberrationCoeffs, // correction coefficients
                                 const bool useSVPR,                   // use spatially varying phase response
                                 const int numPolCoeffs,               // polynomial order for phase correction
                                 const float * const polCoeffs,        // polynomial coefficients
                                 const bool useLUT,                    // use LUT for phase-to-uc conversion
                                 const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  const int tid = threadIdx.x;
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float phase = uc2phase(hologram[idx]);
  if (useAC)
    phase = applyAberrationCorrection(phase, aberrationCoeffs[idx]);

  if (useSVPR) {
    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmDim, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    __shared__ float coeff[MAX_POL];
    if (tid < numPolCoeffs)
      coeff[tid] = polCoeffs[tid];
    __syncthreads();

    hologram[idx] = applySVPR(phase, x, y, coeff);
  } else if (useLUT) {
    __shared__ unsigned char lut_t[MAX_UCHAR];
    if (tid < MAX_UCHAR)
      lut_t[tid] = lut[tid];
    __syncthreads();

    hologram[idx] = lut_t[phase2int(phase)];
  } else {
    hologram[idx] = phase2uc(phase);
  }
}

// Calculate hologram using "Lenses and Prisms"
__global__ void lensesAndPrisms(unsigned char * const hologram,       // hologram to use
                                const unsigned int slmDim,            // SLM's dimension
                                const float slmPitch,                 // 1/slmDim
                                const unsigned int numPixels,         // number of pixels in SLM
                                const float * const spotX,            // x coordinates of spots/traps
                                const float * const spotY,            // y coordinates of spots/traps
                                const float * const spotZ,            // z coordinates of spots/traps
                                const float * const spotI,            // relative intensities of spots/traps
                                const unsigned int numSpots,          // number of spots/traps
                                const bool useAC,                     // use aberration correction
                                const float * const aberrationCoeffs, // correction coefficients
                                const bool useSVPR,                   // use spatially varying phase response
                                const int numPolCoeffs,               // polynomial order for phase correction
                                const float * const polCoeffs,        // polynomial coefficients
                                const bool useLUT,                    // use LUT for phase-to-uc conversion
                                const unsigned char * const lut)      // LUT for phase-to-uc conversion
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  if (idx < numPixels) {
    int xIdx = getXIdx(idx, slmDim);
    int yIdx = getYIdx(idx, xIdx, slmDim, slmPitch);
    float x = getPixelCoords(xIdx, slmDim, slmPitch);
    float y = getPixelCoords(yIdx, slmDim, slmPitch);

    float phase;
    float re = 0.0f;
    float im = 0.0f;

    for (int i = 0; i < numSpots; i++) {
      // Add variable phases to function call
      phase = computePhase(x, y, spotX[i], spotY[i], spotZ[i]);
      re += spotI[i] * cosf(phase);
      im += spotI[i] * sinf(phase);
    }

    phase = atan2f(im, re); // [-pi, pi]
    if (useAC)
      phase = applyAberrationCorrection(phase, aberrationCoeffs[idx]);

    if (useSVPR) {
      __shared__ float coeff[MAX_POL];
      if (tid < numPolCoeffs)
        coeff[tid] = polCoeffs[tid];
      __syncthreads();

      hologram[idx] = applySVPR(phase, x, y, coeff);
    } else if (useLUT) {
      __shared__ unsigned char lut_t[MAX_UCHAR];
      if (tid < MAX_UCHAR)
        lut_t[tid] = lut[tid];
      __syncthreads();

      hologram[idx] = lut_t[phase2int(phase)];
    } else {
      hologram[idx] = phase2uc(phase);
    }
  }
}

__global__ void calculateI(const unsigned char * const hologram, // hologram to use
                           const unsigned int slmDim,            // SLM's dimension
                           const float slmPitch,                 // 1/slmDim
                           const unsigned int numPixels,         // number of pixels in SLM
                           const float * const spotX,            // x coordinates of spots/traps
                           const float * const spotY,            // y coordinates of spots/traps
                           const float * const spotZ,            // z coordinates of spots/traps
                           float * const intensity)              // spot intensity
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float phase;
  int i = tid;
  while (i < numPixels) {
    phase = uc2phase(hologram[i]) - computePhase(x, y, spotX[spotNumber], spotY[spotNumber], spotZ[spotNumber]);

    vRe[tid] += cosf(phase);
    vIm[tid] += sinf(phase);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0]/numPixels;
    float im = vIm[0]/numPixels;
    intensity[spotNumber] = re*re + im*im;
  }
}

__global__ void calculateIAndPhase(const unsigned char * const hologram, // hologram to use
                                   const unsigned int slmDim,            // SLM's dimension
                                   const float slmPitch,                 // 1/slmDim
                                   const unsigned int numPixels,         // number of pixels in SLM
                                   const float * const spotX,            // x coordinates of spots/traps
                                   const float * const spotY,            // y coordinates of spots/traps
                                   const float * const spotZ,            // z coordinates of spots/traps
                                   float * const intensity,              // spot intensity
                                   float * const phase)                  // spot phase
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float p;
  int i = tid;
  while (i < numPixels) {
    p = uc2phase(hologram[i]) - computePhase(x, y, spotX[spotNumber], spotY[spotNumber], spotZ[spotNumber]);
    p += 2.0f * M_PI * spotZ[spotNumber];

    vRe[tid] += cosf(p);
    vIm[tid] += sinf(p);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    float re = vRe[0]/numPixels;
    float im = vIm[0]/numPixels;
    intensity[spotNumber] = re*re + im*im;
    phase[spotNumber] = atan2f(im, re);
  }
}

// Propagate from the SLM to the spot positions using Fresnel summation
// FIXME: Works only for blocksize = SLMsize
__global__ void propagateToSpotPositions(const float * const phase,    // hologram's phase
                                         const unsigned int slmDim,    // SLM's dimension
                                         const float slmPitch,         // 1/slmDim
                                         const unsigned int numPixels, // number of pixels in SLM
                                         const float * const spotX,    // x coordinates of spots/traps
                                         const float * const spotY,    // y coordinates of spots/traps
                                         const float * const spotZ,    // z coordinates of spots/traps
                                         float * const spotRe,         // real component of spot phase
                                         float * const spotIm)         // imaginary component of spot phase
{
  __shared__ float vRe[SLM_SIZE];
  __shared__ float vIm[SLM_SIZE];

  const int blockSize = blockDim.x;
  const int spotNumber = blockIdx.x;
  const int tid = threadIdx.x;

  vRe[tid] = 0.0f;
  vIm[tid] = 0.0f;

  float x = getPixelCoords(tid, slmDim, slmPitch);
  float y = -0.5f; // (0 - slmDim/2) / slmDim

  float p;
  int i = tid;
  while (i < numPixels) {
    p = phase[i] - computePhase(x, y, spotX[spotNumber], spotY[spotNumber], spotZ[spotNumber]);

    vRe[tid] += cosf(p);
    vIm[tid] += sinf(p);

    i += blockSize;
    y += slmPitch;
  }
  __syncthreads();

  if ((tid < 256) && (SLM_SIZE > 256)) {
    vRe[tid] += vRe[tid + 256];
    vIm[tid] += vIm[tid + 256];
  }
  __syncthreads();

  if (tid < 128) {
    vRe[tid] += vRe[tid + 128];
    vIm[tid] += vIm[tid + 128];
  }
  __syncthreads();

  if (tid < 64) {
    vRe[tid] += vRe[tid + 64];
    vIm[tid] += vIm[tid + 64];
  }
  __syncthreads();

  if (tid < 32)
    warpReduce(vRe, vIm, tid);

  if (tid == 0) {
    spotRe[spotNumber] = vRe[0];
    spotIm[spotNumber] = vIm[0];
  }
}

// Obtain phases in SLM plane
__global__ void propagateToSLM(const float * const spotRe,
                               const float * const spotIm,
                               float * const phase,
                               float * const weights,
                               const int iteration,
                               float * const g_pSLMstart,
                               float * const g_Iobtained,
                               const bool lastIter,
                               const unsigned int numPixels,         // number of pixels in SLM
                               const float * const spotX,            // x coordinates of spots/traps
                               const float * const spotY,            // y coordinates of spots/traps
                               const float * const spotZ,            // z coordinates of spots/traps
                               const float * const spotI,            // relative intensities of spots/traps
                               const unsigned int numSpots,          // number of spots/traps
                               unsigned char * const hologram,       // output hologram
                               const unsigned int slmDim,            // SLM's dimension
                               const float slmPitch,                 // 1/slmDim
                               const bool useAC,                     // use aberration correction
                               const float * const aberrationCoeffs, // correction coefficients
                               const bool useSVPR,                   // use spatially varying phase response
                               const int numPolCoeffs,               // polynomial order for phase correction
                               const float * const polCoeffs,        // polynomial coefficients
                               const bool useLUT,                    // use LUT for phase-to-uc conversion
                               const unsigned char * const lut,      // LUT for phase-to-uc conversion
                               const bool useRPC,                    // use restricted phase change
                               const float alpha)                    // RPC threshold
{
  __shared__ float s_aSpot[MAX_SPOTS], s_aSpotsMean, s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  float re = 0.0f, im = 0.0f, p = 0.0f;

  if (idx < c_N_pixels[0]) {
    if (tid < c_N_spots[0]) {
      float spotRe_f = spotRe[tid];
      float spotIm_f = spotIm[tid];
      s_pSpot[tid] = atan2f(spotIm_f, spotRe_f);
      s_aSpot[tid] = hypotf(spotRe_f, spotIm_f)/c_desiredAmp[tid];
      if (iteration != 0) {
        s_weight[tid] = weights[tid + iteration*c_N_spots[0]];
      } else {
        s_aSpot[tid] = (s_aSpot[tid]<0.5f) ? 0.5f : s_aSpot[tid];
        s_weight[tid] = c_desiredAmp[tid];
      }
    }
    __syncthreads();

    //compute weights
    if (tid == 0) {
      float s_aSpot_sum = 0.0f;
      for (int jj = 0; jj < c_N_spots[0]; jj++) {
        s_aSpot_sum += s_aSpot[jj];
      }
      s_aSpotsMean = s_aSpot_sum / (float)c_N_spots[0];
    }
    __syncthreads();

    if (tid < c_N_spots[0]) {
      s_weight[tid] = s_weight[tid] * s_aSpotsMean / s_aSpot[tid];
      if (!lastIter)                      //Copy weights to use as initial value next run
        weights[tid + c_N_spots[0]*(iteration+1)] = s_weight[tid];
      //else
      //  weights[tid] = s_weight[tid];             //Transferring weights to next run may give diverging weights
      if (c_saveI_b[0])
        g_Iobtained[tid + c_N_spots[0]*iteration] = s_aSpot[tid]*s_aSpot[tid];      //may be excluded, used for monitoring only
    }
    __syncthreads();

    //get pixel coordinates
    int X_int = getXIdx(idx, 512);
    int Y_int = getYIdx(idx, X_int, 512, 1/512);
    float x = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
    float y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);

    //compute SLM pSpot by summing contribution from all spots
    for (int k = 0; k < c_N_spots[0]; k++) {
      float delta = computePhase(x, y, spotX[k], spotY[k], spotZ[k]);
      re += s_weight[k] * cosf(s_pSpot[k] + delta);
      im += s_weight[k] * sinf(s_pSpot[k] + delta);
    }
    p = atan2f(im, re);

    if (useRPC) {
      float pSLMstart = g_pSLMstart[idx];
      if (fabs(p - pSLMstart) > alpha)
        p = pSLMstart;
      if (lastIter)
        g_pSLMstart[idx] = p;
    }

    // This is the last iteration, compute and write the final SLM phases to global memory
    if (lastIter) {
      if (useAC)
        p = applyAberrationCorrection(p, aberrationCoeffs[idx]);

      if (useSVPR) {
        __shared__ float coeff[MAX_POL];
        if (tid < numPolCoeffs)
          coeff[tid] = polCoeffs[tid];
        __syncthreads();

        hologram[idx] = applySVPR(p, x, y, coeff);
      } else if (useLUT) {
        __shared__ unsigned char lut_t[MAX_UCHAR];
        if (tid < MAX_UCHAR)
          lut_t[tid] = lut[tid];
        __syncthreads();

        hologram[idx] = lut_t[phase2int(p)];
      } else {
        hologram[idx] = phase2uc(p);
      }
    } else { // Otherwise, write intermediate phases to global memory
      phase[idx] = p;
    }
  }
}

// Convert from unsigned char [0, 255] to float [-pi, pi]
__global__ void uc2f(float *f, const unsigned char * const uc, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    f[idx] = uc[idx] * 2.0f * M_PI/256.0f - M_PI;
  }
}

// Custom debug functions
inline void mSafeCall(hipError_t status, int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do {
    if (status != hipSuccess) {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      if (status != HIPFFT_SUCCESS)
        exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess) {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  } while (0);
#endif
  return;
}

inline void mCufftSafeCall(hipfftResult_t status, int line, char *file)
{
#ifdef M_CUDA_DEBUG
  if(status != HIPFFT_SUCCESS)
  {
    char CUDAmessage[200] = "CUFFT error, CUDA says:\n ";
    switch (status) {
            case HIPFFT_INVALID_PLAN:   strcat(CUDAmessage,"HIPFFT_INVALID_PLAN\n");break;
            case HIPFFT_ALLOC_FAILED:   strcat(CUDAmessage,"HIPFFT_ALLOC_FAILED\n");break;
            case HIPFFT_INVALID_TYPE:   strcat(CUDAmessage,"HIPFFT_INVALID_TYPE\n");break;
            case HIPFFT_INVALID_VALUE:  strcat(CUDAmessage,"HIPFFT_INVALID_VALUE\n");break;
            case HIPFFT_INTERNAL_ERROR: strcat(CUDAmessage,"HIPFFT_INTERNAL_ERROR\n");break;
            case HIPFFT_EXEC_FAILED:    strcat(CUDAmessage,"HIPFFT_EXEC_FAILED\n");break;
            case HIPFFT_SETUP_FAILED:   strcat(CUDAmessage,"HIPFFT_SETUP_FAILED\n");break;
            case HIPFFT_INVALID_SIZE:   strcat(CUDAmessage,"HIPFFT_INVALID_SIZE\n");break;
            //case HIPFFT_UNALIGNED_DATA: strcat(CUDAmessage,"HIPFFT_UNALIGNED_DATA\n");break;
            default: strcat(CUDAmessage,"CUFFT Unknown error code\n");

    }
    sprintf(CUDAmessage,  "%son line: %d\nin file: %s", CUDAmessage, line, file);
    printf("%s", CUDAmessage);
    exit(-1);
  }
  hipDeviceSynchronize();
  hipError_t status2 = hipGetLastError();
  if(status2!=hipSuccess)
  {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status2));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
  }
#endif
  return;
}
inline void mCheckError(int line, char *file)
{
#ifdef M_CUDA_DEBUG
  do
  {
    hipError_t status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA says: ";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
    hipDeviceSynchronize();
    status = hipGetLastError();
    if(status!=hipSuccess)
    {
      char CUDAmessage[200] = "CUDA failed after sychronization:\n";
      strcat(CUDAmessage, hipGetErrorString(status));
      sprintf(CUDAmessage,  "%s\non line: %d\n", CUDAmessage, line);
      printf("%s", CUDAmessage);
      exit(-1);
    }
  }while(0);
#endif
  return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    float *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (float*)malloc(length * sizeof (float));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(float), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %f", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataCC(hipfftComplex *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 25;
    hipfftComplex *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (hipfftComplex*)malloc(length * sizeof (hipfftComplex));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s re: %f im: %f", MessageString, h_data[ii].x, h_data[ii].y);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataUC(unsigned char *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    unsigned char *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (unsigned char*)malloc(length * sizeof (unsigned char));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(unsigned char), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %hhu", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
  do
  {
    int maxlength = 50;
    int *h_data;
    length = (length<=maxlength) ? length : maxlength;
    char MessageString[1000];
    h_data = (int*)malloc(length * sizeof (int));
    M_SAFE_CALL(hipMemcpy(h_data, d_data, length*sizeof(int), hipMemcpyDeviceToHost));
    sprintf(MessageString,  "Line: %d\nData: ", line);
    for (int ii = 0;ii<length;++ii)
    {
      sprintf(MessageString,  "%s %d", MessageString, h_data[ii]);
    }
    printf("%s", MessageString);
    free(h_data);
  }while(0);
#endif
  return;
}

//compute amps for constant total int
/*void computeAmps(float *h_I, float *h_desiredAmp, float *x, float *y, int N_spots, float e_desired)
{
  float SLMsize = (float)SLM_SIZE;
  float Isum = 0.0f;
  for (int i = 0; i<N_spots; i++)
    Isum += h_I[i];
  for (int j = 0; j<N_spots; j++)
  {
    float sincx_rec = 1.0f;//= (x==0)? 1.0f:((M_PI*x[j]/SLMsize)/sinf(M_PI*x[j]/SLMsize));
    float sincy_rec = 1.0f;//(y==0)? 1.0f:((M_PI*y[j]/SLMsize)/sinf(M_PI*y[j]/SLMsize));
    h_desiredAmp[j] = (h_I[j] <= 0.0f) ? 1.0f:(sincx_rec * sincy_rec * sqrtf(e_desired*h_I[j]/Isum)*SLMsize*SLMsize);
  }
}*/

/*
// Obtain phases in SLM plane (ALTERNATIVE VERSION)
// works only for blocksize 512 and max 512 spots
__global__ void propagateToSLM(float *g_x,
                float *g_y,
                float *g_z,
                float *g_I,
                float *g_spotRe_f,
                float *g_spotIm_f,
                float *g_pSLM2pi,
                int N_pixels,
                int N_spots,
                float *g_weights,
                int iteration,
                float *g_pSLMstart,
                float RPC,
                float *g_amps,
                bool getpSLM255,
                unsigned char *g_pSLM255_uc,
                unsigned char *g_LUT,
                bool ApplyLUT_b,
                bool UseAberrationCorr_b,
                float *g_AberrationCorr_f,
                bool UsePolLUT_b,
                float *g_LUTPolCoeff_f,
                int N_PolCoeff)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;
  __shared__ float s_aSpot[MAX_SPOTS], s_aSpotsMean, s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
  __shared__ float s_xm[MAX_SPOTS];
  __shared__ float s_ym[MAX_SPOTS];
  __shared__ float s_zm[MAX_SPOTS];
  float reSLM = 0.0f, imSLM = 0.0f, pSLM2pi_f = 0.0f;
  __shared__ float s_weights_sum;
  if (idx<N_pixels)
  {
    //float N = 512;
    //int logN = (int)log2(N);

    if (tid<c_N_spots[0])
    {
      float spotRe_f = g_spotRe_f[tid];
      float spotIm_f = g_spotIm_f[tid];
      s_pSpot[tid] = atan2f(spotIm_f, spotRe_f);

      s_weight[tid] = g_weights[tid + iteration*c_N_spots[0]] * sqrtf(g_I[tid])/hypotf(spotRe_f, spotIm_f);

      s_xm[tid] = g_x[tid];
      s_ym[tid] = g_y[tid];
      s_zm[tid] = g_z[tid];

      __syncthreads();
      if  (tid==0)
      {
        s_weights_sum = 0.0f;
        for (int jj=0; jj<c_N_spots[0];jj++)
        {
          s_weights_sum += s_weight[jj];
        }
      }
      __syncthreads();
      s_weight[tid] /= s_weights_sum;
      s_weight[tid] = (s_weight[tid]<0.0001f) ? 0.0001f : s_weight[tid];
      //////////////////////////////////////////////////////////////////////////////////
      if (getpSLM255)                     //Copy weights to use as initial value next run
        g_weights[tid] = s_weight[tid];
      else
        g_weights[tid + c_N_spots[0]*(iteration+1)] = s_weight[tid];
      //g_amps[tid + c_N_spots[0]*iteration] = s_aSpot[tid];      //may be excluded, used for monitoring only
    }
    __syncthreads();
    //get pixel coordinates
    float X = c_SLMpitch_f[0] * ((float)threadIdx.x - 256.0f);        //512!
    float Y = c_SLMpitch_f[0] * ((float)blockIdx.x - 256.0f);

    //compute SLM pSpot by summing contribution from all spots
    for (int k=0; k<c_N_spots[0]; k++)
    {
      float delta = M_PI * s_zm[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * s_xm[k] + Y * s_ym[k]);
      reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
      imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
    }
    pSLM2pi_f = atan2f(imSLM, reSLM);


    if (RPC < (2.0f*M_PI))      //Apply RPC (restricted Phase Change)
    {
      float pSLMstart = g_pSLMstart[idx];
      if (fabs(pSLM2pi_f - pSLMstart) > RPC)
        pSLM2pi_f = pSLMstart;
      if (getpSLM255)
        g_pSLMstart[idx] = pSLM2pi_f;
    }

    if (getpSLM255)         //Compute final SLM phases and write to global memory...
    {
      if (UseAberrationCorr_b)
        pSLM2pi_f = applyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);

      if (UsePolLUT_b)
      {
        __shared__ float s_LUTcoeff[MAX_POL];
        if (tid < N_PolCoeff)
          s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = applySVPR(pSLM2pi_f, X, Y, s_LUTcoeff);
      }
      else if (ApplyLUT_b)
      {
        __shared__ unsigned char s_LUT[MAX_UCHAR];
        if (tid < MAX_UCHAR)
          s_LUT[tid] = g_LUT[tid];
        __syncthreads();
        g_pSLM255_uc[idx] = s_LUT[phase2int(pSLM2pi_f)];
      }
      else
        g_pSLM255_uc[idx] = phase2uc(pSLM2pi_f);
    }
    else
      g_pSLM2pi[idx] = pSLM2pi_f; //...or write intermediate pSpot to global memory
  }
}*/

/******************************* Host functions *******************************/

// Timing
double getClock() {
  struct timeval tv;
  int ok;
  ok = gettimeofday(&tv, NULL);
  if (ok < 0) {
    printf("gettimeofday error");
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// Allocate GPU memory and parameters
int setup(const float * const initPhases)
{
  UseAberrationCorr_b = false;
  UsePolLUT_b = false;
  saveI_b = true;
  ApplyLUT_b = false;

  // Make sure there's a GPU
  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != 0) {
    printf("No CUDA compatible GPU found\n");
    exit(1);
  } else {
    M_SAFE_CALL(hipSetDevice(0));
  }

  int MaxIterations = 1000;
  data_w = SLM_SIZE;
  hipMemcpyToSymbol(HIP_SYMBOL(c_data_w), &data_w, sizeof(int), 0, hipMemcpyHostToDevice);
  float data_w_f = (float)data_w;
  hipMemcpyToSymbol(HIP_SYMBOL(c_data_w_f), &data_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
  float half_w_f = (float)data_w/2.0f;
  hipMemcpyToSymbol(HIP_SYMBOL(c_half_w_f), &half_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
  N_pixels = data_w * data_w;
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels), &N_pixels, sizeof(int), 0, hipMemcpyHostToDevice);
  float N_pixels_f = (float)N_pixels;
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels_f), &N_pixels_f, sizeof(float), 0, hipMemcpyHostToDevice);
  int logN = (int)(log2(data_w_f));
  hipMemcpyToSymbol(HIP_SYMBOL(c_log2data_w), &logN, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_useRPC_b), &useRPC_b, sizeof(bool), 0, hipMemcpyHostToDevice);
  float SLMpitch_f = 1.0f/data_w_f;
  hipMemcpyToSymbol(HIP_SYMBOL(c_SLMpitch_f), &SLMpitch_f, sizeof(float), 0, hipMemcpyHostToDevice);

  memsize_spotsf = MAX_SPOTS * sizeof(float);
  memsize_SLM_f = N_pixels * sizeof(float);
  memsize_SLMuc = N_pixels * sizeof(unsigned char);
  n_blocks_Phi = (N_pixels/BLOCK_SIZE + (N_pixels%BLOCK_SIZE == 0 ? 0:1));

  // Memory allocations for all methods
  M_SAFE_CALL(hipMalloc((void**)&d_x, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_y, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_z, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_I, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_desiredAmp, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_weights, MAX_SPOTS*(MaxIterations+1)*sizeof(float)));
  M_SAFE_CALL(hipMalloc((void**)&d_Iobtained, MAX_SPOTS*MaxIterations*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_obtainedPhase, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotRe_f, memsize_spotsf ));
  M_SAFE_CALL(hipMalloc((void**)&d_spotIm_f, memsize_spotsf ));

  int data_w_pow2 = pow(2, ceil(log((float)data_w)/log(2.0f)));
  M_SAFE_CALL(hipMalloc((void**)&d_pSLM_f, data_w_pow2*data_w_pow2*sizeof(float)));//the size of d_pSLM_f must be a power of 2 for the summation algorithm to work
  M_SAFE_CALL(hipMemset(d_pSLM_f, 0, data_w_pow2*data_w_pow2*sizeof(float)));

  M_SAFE_CALL(hipMalloc((void**)&d_pSLMstart_f, memsize_SLM_f));
  M_SAFE_CALL(hipMalloc((void**)&d_hologram, memsize_SLMuc));
  M_SAFE_CALL(hipMemset(d_pSLMstart_f, 0, N_pixels*sizeof(float)));

  M_SAFE_CALL(hipMemcpy(d_pSLM_f, initPhases, N_pixels*sizeof(float), hipMemcpyHostToDevice));

  status = hipGetLastError();
  return status;
}

// Free GPU memory
int finish()
{
  M_SAFE_CALL(hipFree(d_x));
  M_SAFE_CALL(hipFree(d_y));
  M_SAFE_CALL(hipFree(d_z));
  M_SAFE_CALL(hipFree(d_I));

  M_SAFE_CALL(hipFree(d_weights));
  M_SAFE_CALL(hipFree(d_Iobtained));
  M_SAFE_CALL(hipFree(d_pSLM_f));
  M_SAFE_CALL(hipFree(d_pSLMstart_f));
  M_SAFE_CALL(hipFree(d_hologram));

  if (ApplyLUT_b) {
    hipFree(d_LUT_uc);
    d_LUT_uc = NULL;
  }

  if (UseAberrationCorr_b) {
    hipFree(d_AberrationCorr_f);
    d_AberrationCorr_f = NULL;
  }

  if (UsePolLUT_b) {
    hipFree(d_LUTPolCoeff_f);
    d_LUTPolCoeff_f = NULL;
  }

  hipDeviceReset();
  status = hipGetLastError();
  return status;
}

// Set correction parameters
int setCorrectionParameters(const bool useAC,                 // use wavefront aberration/distortion correction
                            const float *aberrationCoeffs,    // correction matrix
                            const bool useSVPR,               // use spatially varying phase response
                            const int polOrder,               // polynomial order for phase correction
                            const float * const polCoeffs,    // polynomial coefficients
                            const bool useLUT,                // use LUT for phase-to-uc conversion (mutually exclusive with svpr)
                            const unsigned char * const lut,  // LUT for phase-to-uc conversion
                            const bool useRPC,                // use restricted phase change
                            const float alpha,                // if < 1.0, RPC threshold = 2*pi*alpha
                            const bool saveAmps)              // save amplitudes in host array
{
  UseAberrationCorr_b = (bool) useAC;
  hipMemcpyToSymbol(HIP_SYMBOL(c_useAberrationCorr_b), &UseAberrationCorr_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  UsePolLUT_b = (bool) useSVPR;
  hipMemcpyToSymbol(HIP_SYMBOL(c_usePolLUT_b), &UsePolLUT_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  saveI_b = (bool) saveAmps;
  hipMemcpyToSymbol(HIP_SYMBOL(c_saveI_b), &saveI_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  ApplyLUT_b = (bool) useLUT;
  hipMemcpyToSymbol(HIP_SYMBOL(c_applyLUT_b), &ApplyLUT_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  alphaRPC_f = alpha * 2.0f * M_PI;
  if (useRPC && alpha < 1.0f)
    useRPC_b = true;
  else
    useRPC_b = false;
  hipMemcpyToSymbol(HIP_SYMBOL(c_alphaRPC_f), &alphaRPC_f, sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_useRPC_b), &useRPC_b, sizeof(bool), 0, hipMemcpyHostToDevice);

  int Ncoeff[5] = {20, 35, 56, 84, 120};

  if ((3 <= polOrder) && (polOrder <= 7)) {
    N_PolLUTCoeff = Ncoeff[polOrder - 3];
    printf("%d\n", N_PolLUTCoeff);
  } else {
    printf("Polynomial order out of range\n -coerced to 3\n");
    N_PolLUTCoeff = Ncoeff[0];
  }
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_PolLUTCoeff), &N_PolLUTCoeff, sizeof(int), 0, hipMemcpyHostToDevice);

  if(UseAberrationCorr_b)
  {
    if (d_AberrationCorr_f == NULL)   //Allocate memory only if not already allocated
      hipMalloc((void**)&d_AberrationCorr_f, memsize_SLM_f);
    UseAberrationCorr_b = !hipMemcpy(d_AberrationCorr_f, aberrationCoeffs, memsize_SLM_f, hipMemcpyHostToDevice);
  }
  else if (d_AberrationCorr_f != NULL)  //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_AberrationCorr_f);
    d_AberrationCorr_f = NULL;
  }
  if(UsePolLUT_b)
  {
    if (d_LUTPolCoeff_f == NULL)          //Allocate memory only if not already allocated
      hipMalloc((void**)&d_LUTPolCoeff_f, MAX_POL*sizeof(float));
    UsePolLUT_b = !hipMemcpy(d_LUTPolCoeff_f, polCoeffs, N_PolLUTCoeff*sizeof(float), hipMemcpyHostToDevice);
  }
  else if (d_LUTPolCoeff_f!=NULL) //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_LUTPolCoeff_f);
    d_LUTPolCoeff_f = NULL;
  }

  if(ApplyLUT_b&&(!UsePolLUT_b))
  {
    if (d_LUT_uc == NULL)         //Allocate memory only if not already allocated
      hipMalloc((void**)&d_LUT_uc, MAX_UCHAR*sizeof(unsigned char));
    ApplyLUT_b = !hipMemcpy(d_LUT_uc, lut, MAX_UCHAR*sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  else if (d_LUT_uc!=NULL)  //If memory is allocated: free memory and reset pointer to NULL
  {
    hipFree(d_LUT_uc);
    d_LUT_uc = NULL;
  }
  M_CHECK_ERROR();

  status = hipGetLastError();
  return status;
}

// Generate a hologram
int generateHologram(unsigned char * const hologram, // hologram to send to SLM
                     const float * const spotX,      // x coordinates of spots/traps
                     const float * const spotY,      // y coordinates of spots/traps
                     const float * const spotZ,      // z coordinates of spots/traps
                     const float * const spotI,      // relative intensities of spots/traps
                     const int numSpots,             // number of spots/traps
                     const int numIterations,        // number of iterations to run GSW
                     float * const interAmps,        // intermediate amplitudes (debug)
                     int method)                     // method to use for generating hologram
{
  if (numSpots < 1)
    method = 100;
  else if (numSpots < 3)
    method = 0;

  computeAndCopySpotData(spotX, spotY, spotZ, spotI, numSpots);
  memsize_spotsf = numSpots * sizeof(float);
  double t;

  switch (method) {
    case 0:
      // Generate hologram using "Lenses and Prisms"
      printf("Starting Lenses and Prisms...\n");
      t = getClock();

      lensesAndPrisms<<<n_blocks_Phi, BLOCK_SIZE>>>(d_hologram,
                                                    SLM_SIZE,
                                                    1.0/SLM_SIZE,
                                                    SLM_SIZE*SLM_SIZE,
                                                    spotX, // FIXME: device arrays
                                                    spotY, // FIXME: device arrays
                                                    spotZ, // FIXME: device arrays
                                                    spotI, // FIXME: device arrays + desired amps
                                                    numSpots,
                                                    UseAberrationCorr_b,
                                                    d_AberrationCorr_f,
                                                    UsePolLUT_b,
                                                    N_PolLUTCoeff,
                                                    d_LUTPolCoeff_f,
                                                    ApplyLUT_b,
                                                    d_LUT_uc);
      M_CHECK_ERROR();
      hipDeviceSynchronize();
      M_CHECK_ERROR();

      if (saveI_b) {
        calculateI<<<numSpots, SLM_SIZE>>>(d_hologram,
                                           SLM_SIZE,
                                           1.0/SLM_SIZE,
                                           SLM_SIZE*SLM_SIZE,
                                           spotX,
                                           spotY,
                                           spotZ,
                                           d_Iobtained);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
        M_SAFE_CALL(hipMemcpy(interAmps, d_Iobtained, numSpots*sizeof(float), hipMemcpyDeviceToHost));
      }
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      break;
    case 1:
      // Generate holgram using fresnel propagation
      printf("Starting Fresnel...\n");
      t = getClock();

      // Uncomment this to start with pre-calculated hologram:
      //hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice);
      //hipDeviceSynchronize();
      //uc2f<<<n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_f, d_hologram, N_pixels);

      for (int l = 0; l < numIterations; l++) {
        printf("Iteration %d\n", l);

        // Propagate to the spot positions
        propagateToSpotPositions<<<numSpots, SLM_SIZE>>>(d_pSLM_f,
                                                         SLM_SIZE,
                                                         1.0/SLM_SIZE,
                                                         SLM_SIZE*SLM_SIZE,
                                                         spotX,
                                                         spotY,
                                                         spotZ,
                                                         d_spotRe_f,
                                                         d_spotIm_f);
        M_CHECK_ERROR();
        hipDeviceSynchronize();

        // Propagate to the SLM plane
        propagateToSLM<<<n_blocks_Phi, BLOCK_SIZE >>>(d_spotRe_f, d_spotIm_f, d_pSLM_f, d_weights, l, d_pSLMstart_f, d_Iobtained, (l==(numIterations-1)), d_hologram, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
        M_CHECK_ERROR();
        hipDeviceSynchronize();
      }

      if (saveI_b)
        M_SAFE_CALL(hipMemcpy(interAmps, d_Iobtained, numSpots*(numIterations)*sizeof(float), hipMemcpyDeviceToHost));
      else
        M_SAFE_CALL(hipMemcpy(interAmps, d_weights, numSpots*(numIterations)*sizeof(float), hipMemcpyDeviceToHost));
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));

      t = getClock() - t;
      printf("Total time = %12.8lf seconds\n", t);
      printf("Time/iteration = %12.8lf seconds\n", t/((double) numIterations));
      break;
    case 100:
      // Apply corrections to pre-calculated hologram
      M_SAFE_CALL(hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice));
      applyCorrections<<<n_blocks_Phi, BLOCK_SIZE>>>(d_hologram,
                                                     SLM_SIZE,
                                                     1.0/SLM_SIZE,
                                                     UseAberrationCorr_b,
                                                     d_AberrationCorr_f,
                                                     UsePolLUT_b,
                                                     N_PolLUTCoeff,
                                                     d_LUTPolCoeff_f,
                                                     ApplyLUT_b,
                                                     d_LUT_uc);
      M_SAFE_CALL(hipMemcpy(hologram, d_hologram, memsize_SLMuc, hipMemcpyDeviceToHost));
      break;
    default:
      break;
  }

  // Handle CUDA errors
  status = hipGetLastError();
  return status;
}

// Calculate amplitude and phase at positions (x, y, z) from a given hologram
int getAmpAndPhase(const float * spotX,                  // x coordinates of spots/traps
                   const float * spotY,                  // y coordinates of spots/traps
                   const float * spotZ,                  // z coordinates of spots/traps
                   const int numSpots,                   // number of spots/traps
                   const unsigned char * const hologram, // hologram to use
                   float *amp,                           // amplitude at (x, y, z)
                   float *phase)                         // phase at (x, y, z)
{
  float *d_amp;
  float *d_phase;
  hipMalloc((void**)&d_amp, numSpots * sizeof(float));
  hipMalloc((void**)&d_phase, numSpots * sizeof(float));
  hipMemcpy(d_hologram, hologram, memsize_SLMuc, hipMemcpyHostToDevice);

  int offset = 0;
  int num_spots_rem = numSpots;
  int num_spots_this;

  while (num_spots_rem > 0) {
    num_spots_this = (num_spots_rem > MAX_SPOTS) ? MAX_SPOTS : num_spots_rem;
    hipMemcpyToSymbol(HIP_SYMBOL(c_x), spotX + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_y), spotY + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_z), spotZ + offset, num_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
    // FIXME: device arrays for spots
    calculateIAndPhase<<<num_spots_this, SLM_SIZE>>>(d_hologram,
                                                     SLM_SIZE,
                                                     1/SLM_SIZE,
                                                     SLM_SIZE*SLM_SIZE,
                                                     spotX,
                                                     spotY,
                                                     spotZ,
                                                     d_amp+offset,
                                                     d_phase+offset);
    hipDeviceSynchronize();

    num_spots_rem -= MAX_SPOTS;
    offset += MAX_SPOTS;
  }

  hipMemcpy(amp, d_amp, numSpots*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(phase, d_phase, numSpots*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_amp);
  hipFree(d_phase);

  status = hipGetLastError();
  return status;
}

void computeAndCopySpotData(const float * const x,
                            const float * const y,
                            const float * const z,
                            const float * const intensity,
                            const int n)
{
  //float Isum = 0.0f;
  //for (int i = 0; i<n; i++)
  //  Isum += intensity[i];

  for (int j = 0; j < n; j++) {
    float sincx_rec = (x[j] == 0) ? 1.0f : ((M_PI * x[j]/SLMsizef) / sinf(M_PI * x[j]/SLMsizef));
    float sincy_rec = (y[j] == 0) ? 1.0f : ((M_PI * y[j]/SLMsizef) / sinf(M_PI * y[j]/SLMsizef));
    h_desiredAmp[j] = (intensity[j] <= 0.0f) ? 1.0f : (sincx_rec * sincy_rec * sqrtf(intensity[j]/100) * SLMsizef * SLMsizef);
  }

  hipMemcpyToSymbol(HIP_SYMBOL(c_x), x, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_y), y, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_z), z, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_desiredAmp), h_desiredAmp, n * sizeof(float), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_N_spots), &n, sizeof(int), 0, hipMemcpyHostToDevice);
}

int main()
{
  srand(1);
  const int num_pixels = SLM_SIZE * SLM_SIZE; // 512^2
  const int num_spots = 4; // Four spots
  const int method = 100; // 0 => Direct, 1 => Fresnel, 100 => Corrections
  const int iterations = 10; // 10 iterations for convergence

  // Spots/traps. These form a quadrant across four planes.
  const float x[] = {-128.0f, -128.0f, 127.0f, 127.0f};
  const float y[] = {127.0f, -128.0f, 127.0f, -128.0f};
  const float z[] = {1.0f, 2.0f, 3.0f, 4.0f};
  const float I[] = {0.12f, 0.34f, 0.56f, 0.78f};

  // Correction parameters
  const int use_aberration_correction = 0;
  const float *aberration_coefficients = NULL;
  const int use_svpr = 0;
  const int pol_order = 5;
  const int pol_size = MAX_POL; // MAX_POL terms with order 7 polynomial
  const int use_lut = 0;
  const unsigned char *lut = NULL;
  int use_rpc = 0;
  float alpha = 0.0f;
  const int save_amplitudes = 1;

  float * const pol_coeffs = (float *) malloc(pol_size * sizeof(float));
  for (int i = 0; i < pol_size; i++) {
    pol_coeffs[i] = random() / ((float) RAND_MAX);
  }

  unsigned char *hologram = (unsigned char *) malloc(num_pixels * sizeof(unsigned char));
  float * const init_phases = (float *) malloc(num_pixels * sizeof(float)); // [-pi, pi]
  for (int i = 0; i < num_pixels; i++) {
    hologram[i] = 0.0f;
    init_phases[i] = (2.0 * M_PI * (random() / ((float) RAND_MAX))) - M_PI;
  }

  float *amps = (float *) malloc(num_spots * iterations * sizeof(float));
  for (int i = 0; i < num_spots * iterations; i++) {
    amps[i] = 0.0f;
  }

  if (setup(init_phases) != 0) {
    printf("Init failed.\n");
    exit(1);
  }

  if (setCorrectionParameters(use_aberration_correction, aberration_coefficients,
      use_svpr, pol_order, pol_coeffs, use_lut, lut, use_rpc, alpha, save_amplitudes) != 0) {
    printf("Correction setup failed.\n");
    exit(1);
  }

  double t = getClock();

  if (generateHologram(hologram, x, y, z, I, num_spots, iterations, amps, method) != 0) {
    printf("Computation failed.\n");
    exit(1);
  }

  t = getClock() - t;

  if (finish() != 0) {
    printf("Cleanup failed.\n");
    exit(1);
  }

  printf("Total time = %12.8lf seconds\n", t);

  // Save hologram
  FILE *hfile = fopen("orig_hologram.dat", "w");
  for (int i = 0; i < num_pixels; i++) {
    fprintf(hfile, "%hhu\n", hologram[i]);
  }

  // Save amplitudes
  FILE *afile = fopen("orig_amps.dat", "w");
  for (int i = 0; i < num_spots * iterations; i++) {
    fprintf(afile, "%f\n", amps[i]);
  }

  fclose(hfile);
  fclose(afile);

  return 0;
}

